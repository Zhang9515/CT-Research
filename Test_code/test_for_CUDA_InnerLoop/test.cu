#include "hip/hip_runtime.h"
#include "test.h"
// 2018/04/20 apply GPU acceleration
#define pow2(x) (1.0*(x)*(x))
__device__ const double PI = 3.141592653589793;

__global__ void BackProjection(const float *dev_R, float *dev_Display, bool * dev_signal)
{

	const unsigned int Tindex = threadIdx.x;
	const unsigned int Bindex = blockIdx.x;
	int index = Bindex * 256 + Tindex;
	__syncthreads();
	//for (int num = 0;num<256*16;num++)
	//{
	//	if (num == 0)
	//	{
			while (1) {
				if (!dev_signal[Tindex]) {
					dev_signal[Tindex] = true;
					dev_Display[Tindex] += /*dev_R[index]*/Bindex;
					dev_signal[Tindex] = false;
					__threadfence();
					break;
				}
			}
				

	//	}		
	//}

	//__threadfence();
	//dev_Display[Tindex] = 3;

}

// Helper function for using CUDA to add vectors in parallel.
hipError_t FDKpro(float *Display, const float *R)
{
	float* dev_Display = 0; bool *dev_signal = 0; float* dev_R = 0;
	int LR = 16 * 256; int LD = 256;

	const dim3 thread_cubic(256, 1, 1);
	const dim3 block_cubic(16, 1, 1);

	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?\n");
		mexPrintf("hipSetDevice failed!  Do you have a CUDA-capable GPU installed? %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	mexPrintf("call for space in GPU\n");

	// Allocate GPU buffers for three vectors (two input, one output).

	cudaStatus = hipMalloc((void**)&dev_R, LR * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_R hipMalloc failed!\n");
		mexPrintf("dev_R hipMalloc failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_Display, LD * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_Pdomain hipMalloc failed!\n");
		mexPrintf("dev_Pdomain hipMalloc failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}
	hipMemset(dev_Display, 0, sizeof(float));

	cudaStatus = hipMalloc((void**)&dev_signal, LD * sizeof(bool));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_Pdomain hipMalloc failed!\n");
		mexPrintf("dev_Pdomain hipMalloc failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}
	hipMemset(dev_signal, false, sizeof(bool));
	//mexPrintf("copy data in CPU to GPU\n");

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_R, R, LR * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy R failed!\n");
		mexPrintf("hipMemcpy R failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	//Backprojection
	BackProjection << <block_cubic, thread_cubic >> > (dev_R, dev_Display, dev_signal);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "BackProjection launch failed: %s\n", hipGetErrorString(cudaStatus));
		mexPrintf("BackProjection launch failed %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		mexPrintf("hipDeviceSynchronize returned error code %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	 //Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(Display, dev_Display, LD * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!\n");
		mexPrintf("hipMemcpy dev_Display failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

Error:
	hipFree(dev_R);
	hipFree(dev_Display);


	mexPrintf("Exit FDK\n");
	return cudaStatus;
}
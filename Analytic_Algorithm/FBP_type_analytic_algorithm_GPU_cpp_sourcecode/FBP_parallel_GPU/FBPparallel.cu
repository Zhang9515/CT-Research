#include "hip/hip_runtime.h"
#include "FBPparallel.h"
// 2019/03/28 apply GPU acceleration
#define pow2(x) (1.0*(x)*(x))
__device__ const double PI = 3.141592653589793;

__global__ void GFunction(float *dev_G, const float TInt, const int tstart, const int LT)
{

	const unsigned int Tindex = tstart + threadIdx.x;

	// S_L filter 
	if (Tindex == 0)
		dev_G[LT - 1] = 1.0 / pow2(PI*TInt);
	else
	{
		dev_G[(LT - 1) - Tindex] = -1.0 / pow2(TInt * PI) / (4 * pow2(Tindex) - 1);
		dev_G[(LT - 1) + Tindex] = -1.0 / pow2(TInt * PI) / (4 * pow2(Tindex) - 1);
	}
}

__global__ void Filter(float *dev_Rcov, float *dev_R, const float *dev_G, const float TInt, 
	const int LT, const int tstart, const int Thetastart, const int gstart, const int gend)
{
	const unsigned int Tindex = threadIdx.x + tstart;
	const unsigned int thetaIndex = blockIdx.x + Thetastart;
	const unsigned long base_id = thetaIndex * LT;
	const unsigned long thread_id = base_id + Tindex;

	double Rcovsum = 0;
	// convolution

	for (int g = gstart; g < gend; g++)
	{
		//convolution
		Rcovsum += dev_R[base_id + g] * dev_G[(LT - 1) + Tindex - g];
	}

	dev_Rcov[thread_id] += TInt * Rcovsum;
}

__global__ void BackProjection(const float *dev_Rcov, float *dev_Display, const double *dev_Size, const int T_length, const int S_length,
	const float Theta, const float *dev_Tdomain, const float TInt, const float ThetaScanInt, const float mint, 
	const float maxt, const int thetaIndex, const int LT, const int LTheta, const int T_start, const int S_start)
{
	// here makes clear an important note: t in lower case means the location on the detector, while T in captial case means a dimension of the axis.
	// only take effect in this function
	// initialize
	const double Resolution_t = 1.0 * dev_Size[0] / T_length;
	const double Resolution_s = 1.0 * dev_Size[1] / S_length;
	// rotation center
	double Center_t = dev_Size[0] / 2;
	double Center_s = dev_Size[1] / 2;
	// index 
	const unsigned int Tindex = T_start + threadIdx.x;
	const unsigned int Sindex = S_start + blockIdx.x;

	double image_t, image_s, dect_t;
	unsigned int tN1index, tN2index;
	double t_domain1, t_domain2;
	double t1, t2;
	double Display_pTheta = 0;
	double t;
	unsigned long thread_id;

	// this is a little different from code on MATLAB
	image_t = (Tindex + 0.5) * Resolution_t - Center_t;    // center of the image pixel in ground coordinate
	image_s = (Sindex + 0.5) * Resolution_s - Center_s;

	t = image_t * cos(Theta) + image_s * sin(Theta);   // rotate in ground coordinate
	//dect_t = -image_t * sin(Theta) + image_s * cos(Theta);   // rotate in ground coordinate
	
	if ((t >= mint) && (t < maxt))
	{
		tN1index = floor(fabs(t - dev_Tdomain[0]) / TInt);
		tN2index = tN1index + 1;

		t_domain1 = dev_Tdomain[tN1index]; t_domain2 = dev_Tdomain[tN2index];
	
		//bilinear interpolation
		t1 = fabs(t - t_domain1); t2 = fabs(t_domain2 - t);

		//Display_pTheta = 1;
		Display_pTheta = ( t2 * dev_Rcov[thetaIndex * LT + tN1index] + t1 * dev_Rcov[thetaIndex * LT + tN2index] )
			/ TInt * ThetaScanInt;
	}
	thread_id = Sindex * T_length + Tindex;
	dev_Display[thread_id] += Display_pTheta;

}

// HeLTer function for using CUDA to add vectors in parallel.
hipError_t FBPpara(float *Display, const float *R, const float *Tdomain, const float *ThetaScanRange, const int LTheta,
	const int LT, const double *Size, const int t_length, const int s_length)
{
	mexPrintf("Hello FBPpara!\n");
	float *dev_R = 0, *dev_Rcov = 0;
	float *dev_G = 0;
	float *dev_ThetaScanRange = 0, *dev_Tdomain = 0;
	double *dev_Size = 0;

	float *dev_Display = 0;
	float TInt = fabs(Tdomain[1] - Tdomain[0]);
	float ThetaScanInt = fabs(ThetaScanRange[1] - ThetaScanRange[0]);

	float maxt = MAX(Tdomain[0], Tdomain[LT - 1]);
	float mint = MIN(Tdomain[0], Tdomain[LT - 1]);
	//mexPrintf("%lf %lf %lf %lf \n", maxGama, minGama, maxXigama, minXigama);

	const long LDisplay = t_length * s_length;
	const long LR = LT * LTheta;
	const int LFilter = 2 * LT - 1;

	short thread_cubic_x = MIN(threadX, LT);
	short block_cubic_x = MIN(blockX, LTheta);
	short thread_cubic_Bp_x = MIN(threadX, t_length);
	short block_cubic_Bp_x = MIN(blockX, s_length);

	const dim3 thread_cubic(thread_cubic_x, 1, 1);
	const dim3 block_cubic(block_cubic_x, 1, 1);
	const dim3 thread_cubic_Bp(thread_cubic_Bp_x, 1, 1);
	const dim3 block_cubic_Bp(block_cubic_Bp_x, 1, 1);

	dim3 thread_cubic_residual(1, 1, 1);  // initial
	dim3 block_cubic_residual(1, 1, 1);  // initial
	dim3 thread_cubic_Bp_residual(1, 1, 1);  // initial
	dim3 block_cubic_Bp_residual(1, 1, 1);  // initial

	short LTResidual = LT % threadX;
	short LThetaResidual = LTheta % blockX;
	short tTime = LT / threadX;
	short ThetaTime = LTheta / blockX;
	short tstart = 0;
	short Thetastart = 0;
	short TlengthResidual = t_length % threadX;
	short SlengthResidual = s_length % blockX;
	short T_Time = t_length / threadX;
	short S_Time = s_length / blockX;
	short T_start = 0;
	short S_start = 0;
	short gstart = 0, gend = 0;
	short gtime = 1;
	//mexPrintf("tTime: %d ThetaTime: %d\n", tTime, ThetaTime);

	if (LTResidual != 0)
	{
		thread_cubic_residual.x = LTResidual;
	}
	if (LThetaResidual != 0)
	{
		block_cubic_residual.x = LThetaResidual;
	}
	if (TlengthResidual != 0)
	{
		thread_cubic_Bp_residual.x = TlengthResidual;
	}
	if (SlengthResidual != 0)
	{
		block_cubic_Bp_residual.x = SlengthResidual;
	}

	mexPrintf("thread_cubic_x: %d block_cubic_x: %d LTResidual: %d LThetaResidual: %d\n",
		thread_cubic_x, block_cubic_x, LTResidual, LThetaResidual);
	mexPrintf("thread_cubic_Bp_x: %d block_cubic_Bp_x: %d TlengthResidual: %d SlengthResidual: %d\n",
		thread_cubic_Bp_x, block_cubic_Bp_x, TlengthResidual, SlengthResidual);

	hipError_t cudaStatus;

	///////////////////////////////////////////////////////////////////////////////////////////////	
	mexPrintf("start cuda\n");

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?\n");
		mexPrintf("hipSetDevice failed!  Do you have a CUDA-capable GPU installed? %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	mexPrintf("call for space in GPU\n");

	// Allocate GPU buffers for three vectors (two input, one output).

	cudaStatus = hipMalloc((void**)&dev_R, LR * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_R hipMalloc failed!\n");
		mexPrintf("dev_R hipMalloc failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error1;
	}

	cudaStatus = hipMalloc((void**)&dev_Tdomain, LT * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_Tdomain hipMalloc failed!\n");
		mexPrintf("dev_Tdomain hipMalloc failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_ThetaScanRange, LTheta * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_ThetaScanRange hipMalloc failed!\n");
		mexPrintf("dev_ThetaScanRange hipMalloc failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	//mexPrintf("copy data in CPU to GPU\n");

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_R, R, LR * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy R failed!\n");
		mexPrintf("hipMemcpy R failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error1;
	}

	cudaStatus = hipMemcpy(dev_Tdomain, Tdomain, LT * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy Tdomain failed!\n");
		mexPrintf("hipMemcpy Tdomain failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_ThetaScanRange, ThetaScanRange, LTheta * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy ThetaScanRange failed!\n");
		mexPrintf("hipMemcpy ThetaScanRange failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	mexPrintf("start parallel computation\n");

	mexPrintf("Preweight and Filtering\n");
	// Preweight and filter 
	// to be adapted to GPU, we limited the number of threads in each grid (threadX * blockX * LTheta)
	// generate filter
	mexPrintf("Generate filter\n");
	cudaStatus = hipMalloc((void**)&dev_G, LFilter * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_G hipMalloc failed!\n");
		mexPrintf("dev_G hipMalloc failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error1;
	}
	hipMemset(dev_G, 0, sizeof(float));

	for (int numT = 0; numT < tTime; numT++)
	{
		tstart = numT * threadX;
		GFunction << <1, thread_cubic_x >> > (dev_G, TInt, tstart, LT);
		// Check for any errors launching the kernel
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "generate filter failed: %s\n", hipGetErrorString(cudaStatus));
			mexPrintf("generate filter launch failed %s\n", hipGetErrorString(cudaStatus));
			mexPrintf("Error happens at numT: %d\n", numT);
			goto Error;
		}
	}
	if (LTResidual != 0)
	{
		tstart = LT - LTResidual;
		GFunction << <1, thread_cubic_residual.x >> > (dev_G, TInt, tstart, LT);
		// Check for any errors launching the kernel
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "generate filter failed: %s\n", hipGetErrorString(cudaStatus));
			mexPrintf("generate filter launch failed %s\n", hipGetErrorString(cudaStatus));
			mexPrintf("Error happens at LTResidual: %d\n", LTResidual);
			goto Error;
		}
	}

	// store the result of convolution
	cudaStatus = hipMalloc((void**)&dev_Rcov, LR * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_Rcov hipMalloc failed!\n");
		mexPrintf("dev_Rcov hipMalloc failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}
	hipMemset(dev_Rcov, 0, sizeof(float));

	// convolution with limited scale
	gtime = ceil(1.0 * LT / Filterlengthlimit);
	mexPrintf("gtime: %d\n", gtime);

	for (int numT = 0; numT < tTime; numT++)
	{
		for (int numTheta = 0; numTheta < ThetaTime; numTheta++)
		{
			tstart = numT * threadX;
			Thetastart = numTheta * blockX;
			//mexPrintf("%d %d\n", tstart, Thetastart);
			for (int numg = 0; numg < gtime; numg++)
			{
				gstart = int(numg * Filterlengthlimit);
				if (numg != (gtime - 1))
				{
					gend = int((numg + 1)* Filterlengthlimit);
				}
				else
				{
					gend = LT;
				}
				/*mexPrintf("gstart: %d\n", gstart);
				mexPrintf("gend: %d\n", gend);*/
				Filter << <block_cubic, thread_cubic >> > (dev_Rcov, dev_R, dev_G, TInt, LT, tstart, Thetastart, gstart, gend);

				// Check for any errors launching the kernel
				cudaStatus = hipGetLastError();
				if (cudaStatus != hipSuccess) {
					fprintf(stderr, "Filter launch failed: %s\n", hipGetErrorString(cudaStatus));
					mexPrintf("Filter launch failed %s\n", hipGetErrorString(cudaStatus));
					mexPrintf("Error happens at tstart: %d Thetastart: %d numg: %d\n",
						tstart, Thetastart, numg);
					goto Error;
				}
			}
		}
	}

	if (LTResidual != 0)
	{
		tstart = LT - LTResidual;
		if (LThetaResidual != 0)
		{
			Thetastart = LTheta - LThetaResidual;
			//("%d %d\n", tstart, Thetastart);
			for (int numg = 0; numg < gtime; numg++)
			{
				gstart = int(numg * Filterlengthlimit);
				if (numg != (gtime - 1))
				{
					gend = int((numg + 1)* Filterlengthlimit);
				}
				else
				{
					gend = LT;
				}
				/*mexPrintf("gstart: %d\n", gstart);
				mexPrintf("gend: %d\n", gend);*/
				Filter << <block_cubic_residual, thread_cubic_residual >> > (dev_Rcov, dev_R, dev_G, TInt, LT, tstart, Thetastart, gstart, gend);
				// Check for any errors launching the kernel
				cudaStatus = hipGetLastError();
				if (cudaStatus != hipSuccess) {
					fprintf(stderr, "Filter launch failed: %s\n", hipGetErrorString(cudaStatus));
					mexPrintf("Filter launch failed %s\n", hipGetErrorString(cudaStatus));
					mexPrintf("Error happens at tstart: %d Thetastart: %d numg: %d\n",
						tstart, Thetastart, numg);
					goto Error;
				}
			}
		}

		for (int numTheta = 0; numTheta < ThetaTime; numTheta++)
		{
			Thetastart = numTheta * blockX;
			//("%d %d\n", tstart, Thetastart);
			for (int numg = 0; numg < gtime; numg++)
			{
				gstart = int(numg * Filterlengthlimit);
				if (numg != (gtime - 1))
				{
					gend = int((numg + 1)* Filterlengthlimit);
				}
				else
				{
					gend = LT;
				}
				/*mexPrintf("gstart: %d\n", gstart);
				mexPrintf("gend: %d\n", gend);*/
				Filter << <block_cubic, thread_cubic_residual >> > (dev_Rcov, dev_R, dev_G, TInt, LT, tstart, Thetastart, gstart, gend);
				// Check for any errors launching the kernel
				cudaStatus = hipGetLastError();
				if (cudaStatus != hipSuccess) {
					fprintf(stderr, "Filter launch failed: %s\n", hipGetErrorString(cudaStatus));
					mexPrintf("Filter launch failed %s\n", hipGetErrorString(cudaStatus));
					mexPrintf("Error happens at tstart: %d Thetastart: %d numg: %d\n",
						tstart, Thetastart, numg);
					goto Error;
				}
			}
		}
	}
	if (LThetaResidual != 0)
	{
		Thetastart = LTheta - LThetaResidual;
		for (int numT = 0; numT < tTime; numT++)
		{
			tstart = numT * threadX;
			//mexPrintf("%d %d\n", tstart, Thetastart);
			for (int numg = 0; numg < gtime; numg++)
			{
				gstart = int(numg * Filterlengthlimit);
				if (numg != (gtime - 1))
				{
					gend = int((numg + 1)* Filterlengthlimit);
				}
				else
				{
					gend = LT;
				}
				/*mexPrintf("gstart: %d\n", gstart);
				mexPrintf("gend: %d\n", gend);*/
				Filter << <block_cubic_residual, thread_cubic >> > (dev_Rcov, dev_R, dev_G, TInt, LT, tstart, Thetastart, gstart, gend);
				// Check for any errors launching the kernel
				cudaStatus = hipGetLastError();
				if (cudaStatus != hipSuccess) {
					fprintf(stderr, "Filter launch failed: %s\n", hipGetErrorString(cudaStatus));
					mexPrintf("Filter launch failed %s\n", hipGetErrorString(cudaStatus));
					mexPrintf("Error happens at tstart: %d Thetastart: %d numg: %d\n",
						tstart, Thetastart, numg);
					goto Error;
				}
			}
		}
	}

Error1:
	hipFree(dev_R);
	hipFree(dev_G);

	mexPrintf("backprojection\n");

	cudaStatus = hipMalloc((void**)&dev_Size, 2 * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_Size hipMalloc failed!\n");
		mexPrintf("dev_Size hipMalloc failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_Size, Size, 2 * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy Size failed!\n");
		mexPrintf("hipMemcpy Size failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_Display, LDisplay * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_Display hipMalloc failed!\n");
		mexPrintf("dev_Display hipMalloc failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}
	hipMemset(dev_Display, 0, sizeof(float));

	//Backprojection  
	for (int thetaIndex = 0; thetaIndex < LTheta; thetaIndex++)
	{
		for (int numT = 0; numT < T_Time; numT++)
		{
			for (int numS = 0; numS < S_Time; numS++)
			{
				T_start = numT * threadX;
				S_start = numS * blockX;
				BackProjection << <block_cubic_Bp, thread_cubic_Bp >> > (dev_Rcov, dev_Display, dev_Size, t_length, s_length,
					ThetaScanRange[thetaIndex], dev_Tdomain, TInt, ThetaScanInt, mint, maxt, thetaIndex, LT, LTheta, T_start, S_start);
				
				// Check for any errors launching the kernel
				cudaStatus = hipGetLastError();
				if (cudaStatus != hipSuccess) {
					fprintf(stderr, "BackProjection launch failed: %s\n", hipGetErrorString(cudaStatus));
					mexPrintf("BackProjection launch failed %s\n", hipGetErrorString(cudaStatus));
					mexPrintf("Error happens at thetaIndex: %d\n", thetaIndex);
					goto Error;
				}
			}
		}
		if (TlengthResidual != 0)
		{
			T_start = t_length - TlengthResidual;
			for (int numS = 0; numS < S_Time; numS++)
			{
				S_start = numS * blockX;
				BackProjection << <block_cubic_Bp, thread_cubic_Bp_residual >> > (dev_Rcov, dev_Display, dev_Size, t_length, s_length,
					ThetaScanRange[thetaIndex], dev_Tdomain, TInt, ThetaScanInt, mint, maxt, thetaIndex, LT, LTheta, T_start, S_start);
				// Check for any errors launching the kernel
				cudaStatus = hipGetLastError();
				if (cudaStatus != hipSuccess) {
					fprintf(stderr, "BackProjection launch failed: %s\n", hipGetErrorString(cudaStatus));
					mexPrintf("BackProjection launch failed %s\n", hipGetErrorString(cudaStatus));
					mexPrintf("Error happens at thetaIndex: %d\n", thetaIndex);
					goto Error;
				}
			}
			if (SlengthResidual != 0)
			{
				S_start = s_length - SlengthResidual;
				BackProjection << <block_cubic_Bp_residual, thread_cubic_Bp_residual >> > (dev_Rcov, dev_Display, dev_Size, t_length, s_length,
					ThetaScanRange[thetaIndex], dev_Tdomain, TInt, ThetaScanInt, mint, maxt, thetaIndex, LT, LTheta, T_start, S_start);
				// Check for any errors launching the kernel
				cudaStatus = hipGetLastError();
				if (cudaStatus != hipSuccess) {
					fprintf(stderr, "BackProjection launch failed: %s\n", hipGetErrorString(cudaStatus));
					mexPrintf("BackProjection launch failed %s\n", hipGetErrorString(cudaStatus));
					mexPrintf("Error happens at thetaIndex: %d\n", thetaIndex);
					goto Error;
				}
			}
		}
		if (SlengthResidual != 0)
		{
			S_start = s_length - SlengthResidual;
			for (int numT = 0; numT < T_Time; numT++)
			{
				T_start = numT * threadX;
				BackProjection << <block_cubic_Bp_residual, thread_cubic_Bp >> > (dev_Rcov, dev_Display, dev_Size, t_length, s_length,
					ThetaScanRange[thetaIndex], dev_Tdomain, TInt, ThetaScanInt, mint, maxt, thetaIndex, LT, LTheta, T_start, S_start);
				// Check for any errors launching the kernel
				cudaStatus = hipGetLastError();
				if (cudaStatus != hipSuccess) {
					fprintf(stderr, "BackProjection launch failed: %s\n", hipGetErrorString(cudaStatus));
					mexPrintf("BackProjection launch failed %s\n", hipGetErrorString(cudaStatus));
					mexPrintf("Error happens at thetaIndex: %d\n", thetaIndex);
					goto Error;
				}
			}
		}		
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		mexPrintf("hipDeviceSynchronize returned error code %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	//Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(Display, dev_Display, LDisplay * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!\n");
		mexPrintf("hipMemcpy dev_Display failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}
	//for debug{
	//cudaStatus = hipMemcpy(Rcov, dev_Rcov, LR * sizeof(float), hipMemcpyDeviceToHost);
	//if (cudaStatus != hipSuccess) {
	//	fprintf(stderr, "hipMemcpy failed!\n");
	//	mexPrintf("hipMemcpy dev_Display failed! %s\n", hipGetErrorString(cudaStatus));
	//	goto Error;
	//}
	//}end debug

Error:
	hipFree(dev_ThetaScanRange);
	hipFree(dev_Tdomain);
	hipFree(dev_Rcov);
	hipFree(dev_Display);
	hipFree(dev_Size);

	mexPrintf("Exit FBP\n");
	return cudaStatus;
}
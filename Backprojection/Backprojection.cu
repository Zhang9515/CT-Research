#include "hip/hip_runtime.h"
#include "Backprojection.h"
// 2018/11/16 apply GPU acceleration

__device__ const double PI = 3.141592653589793;

// thiss
__global__ void BackProjection(const float *dev_R, float *dev_Display, const double *dev_Size,
	const int t_length, const int s_length, const int z_length, const float Beta, const double Distance, 
	const float *dev_Pdomain, const float *dev_Xigamadomain, const float PInt, const float XigamaInt, 
	const float BetaScanInt, const float minP, const float maxP, const float minXigama, const float maxXigama, 
	const int betaIndex, const int LP, const int LXigama)
{
	const unsigned int Tindex = threadIdx.x;
	const unsigned int Sindex = blockIdx.x;
	const unsigned int Zindex = blockIdx.y;
	const unsigned long thread_id = Zindex * (gridDim.x * blockDim.x)
		+ Sindex * blockDim.x + Tindex;
	// initialize

	const double Resolution_t = 1.0 * dev_Size[0] / t_length;
	const double Resolution_s = 1.0 * dev_Size[1] / s_length;
	const double Resolution_z = 1.0 * dev_Size[2] / z_length;

	// rotation center
	double Center_t = dev_Size[0] / 2;
	double Center_s = dev_Size[1] / 2;
	double Center_z = dev_Size[2] / 2;

	// this is a little different from code on MATLAB
	// image pixel in ground coordinate
	double image_t = (Tindex + 0.5) * Resolution_t - Center_t;  
	double image_s = (Sindex + 0.5) * Resolution_s - Center_s; 
	double image_z = (Zindex + 0.5) * Resolution_z - Center_z;

	// rotate in ground coordinate
	double dect_t = image_t * cos(Beta) + image_s * sin(Beta);
	double dect_s = -image_t * sin(Beta) + image_s * cos(Beta);
	double dect_z = image_z;

	// define the projection position on the detector
	double LengthRatio = Distance / (Distance - dect_s);
	double Xigama = dect_z * LengthRatio;
	double P = dect_t * LengthRatio;

	unsigned int XigamaN1index = 0, XigamaN2index = 0, PN1index = 0, PN2index = 0;
	double P_domain1 = 0, P_domain2 = 0, Xigama_domain1 = 0, Xigama_domain2 = 0;
	double Xig1 = 0, Xig2 = 0, P1 = 0, P2 = 0;
	double Display_pBeta = 0;
	double backweight = 0;
	double LengthinROI = 0;

	if ((P >= minP) && (P < maxP) && (Xigama >= minXigama) && (Xigama < maxXigama))
	{
		XigamaN1index = floor(fabs(Xigama - dev_Xigamadomain[0]) / XigamaInt);
		XigamaN2index = XigamaN1index + 1;
		PN1index = floor(fabs(P - dev_Pdomain[0]) / PInt);
		PN2index = PN1index + 1;

		P_domain1 = dev_Pdomain[PN1index]; P_domain2 = dev_Pdomain[PN2index];
		Xigama_domain1 = dev_Xigamadomain[XigamaN1index]; Xigama_domain2 = dev_Xigamadomain[XigamaN2index];

		//bilinear interpolation
		Xig1 = fabs(Xigama - Xigama_domain1); Xig2 = fabs(Xigama_domain2 - Xigama);
		P1 = fabs(P - P_domain1); P2 = fabs(P_domain2 - P);

		Display_pBeta = (Xig2 * P2 * dev_R[betaIndex * LP * LXigama + XigamaN1index * LP + PN1index]
			+ Xig1 * P2 * dev_R[betaIndex * LP * LXigama + XigamaN2index * LP + PN1index] + Xig2 * P1 * dev_R[betaIndex * LP * LXigama + XigamaN1index * LP + PN2index]
			+ Xig1 * P1 * dev_R[betaIndex * LP * LXigama + XigamaN2index * LP  + PN2index]) / (PInt * XigamaInt);

	//	 the way to compute backweight is to get the cross length in the specific pixel and the whole ROI

	//	 according to euler equation   
		double source_t = Center_t - Distance * sin(Beta);      // define the source in matlab coordinate														
		double source_s = Center_s + Distance * cos(Beta);
		double source_z = Center_z;
		
		// in matlab coordinate
		//  assume the projection line go through the center of the current pixel 
		double DetectPoint_tend = image_t + Center_t;
		double DetectPoint_send = image_s + Center_s;
		double DetectPoint_zend = image_z + Center_z;

	//	first compute length in whole ROI
	//	 actual Size
		double tlow = 0, thigh = t_length * Resolution_t, slow = 0, shigh = s_length * Resolution_s,
			zlow = 0, zhigh = z_length * Resolution_z;

	//	compute the first and last point in the ROI
	//	 using DetectPoint_end set up projection equation
		double tlow_s = source_s + (tlow - source_t) * (DetectPoint_send - source_s) / (DetectPoint_tend - source_t);
		double tlow_z = source_z + (tlow - source_t) * (DetectPoint_zend - source_z) / (DetectPoint_tend - source_t);
		double thigh_s = source_s + (thigh - source_t) * (DetectPoint_send - source_s) / (DetectPoint_tend - source_t);
		double thigh_z = source_z + (thigh - source_t) * (DetectPoint_zend - source_z) / (DetectPoint_tend - source_t);

		double slow_t = source_t + (slow - source_s) * (DetectPoint_tend - source_t) / (DetectPoint_send - source_s);
		double slow_z = source_z + (slow - source_s) * (DetectPoint_zend - source_z) / (DetectPoint_send - source_s);
		double shigh_t = source_t + (shigh - source_s) * (DetectPoint_tend - source_t) / (DetectPoint_send - source_s);
		double shigh_z = source_z + (shigh - source_s) * (DetectPoint_zend - source_z) / (DetectPoint_send - source_s);

		double zlow_t = source_t + (zlow - source_z) * (DetectPoint_tend - source_t) / (DetectPoint_zend - source_z);
		double zlow_s = source_s + (zlow - source_z) * (DetectPoint_send - source_s) / (DetectPoint_zend - source_z);
		double zhigh_t = source_t + (zhigh - source_z) * (DetectPoint_tend - source_t) / (DetectPoint_zend - source_z);
		double zhigh_s = source_s + (zhigh - source_z) * (DetectPoint_send - source_s) / (DetectPoint_zend - source_z);

	//	double *Range = new double [6];   //  XYXY small-big(number)
		double T1 = 0, S1 = 0, Z1 = 0, T2 = 0, S2 = 0, Z2 = 0;

		if (tlow_s >= 0 && tlow_s <= shigh && tlow_z >= 0 && tlow_z <= zhigh)
		{
			T1 = tlow; S1 = tlow_s; Z1 = tlow_z;
			if (thigh_s >= 0 && thigh_s <= shigh && thigh_z >= 0 && thigh_z <= zhigh)
			{
				T2 = thigh; S2 = thigh_s; Z2 = thigh_z;
			}
			else if (slow_t >= 0 && slow_t <= thigh && slow_z >= 0 && slow_z <= zhigh)
			{
				T2 = slow_t; S2 = slow; Z2 = slow_z;
			}
			else if (shigh_t >= 0 && shigh_t <= thigh && shigh_z >= 0 && shigh_z <= zhigh)
			{
				T2 = shigh_t; S2 = shigh; Z2 = shigh_z;
			}
			else if (zlow_t >= 0 && zlow_t <= thigh && zlow_s >= 0 && zlow_s <= shigh)
			{
				T2 = zlow_t; S2 = zlow_s; Z2 = zlow;
			}
			else if (zhigh_t >= 0 && zhigh_t <= thigh && zhigh_s >= 0 && zhigh_s <= shigh)
			{
				T2 = zhigh_t; S2 = zhigh_s; Z2 = zhigh;
			}
		}
		else if (thigh_s >= 0 && thigh_s <= shigh && thigh_z >= 0 && thigh_z <= zhigh)
		{
			T1 = thigh; S1 = thigh_s; Z1 = thigh_z;
			if (slow_t >= 0 && slow_t <= thigh && slow_z >= 0 && slow_z <= zhigh)
			{
				T2 = slow_t; S2 = slow; Z2 = slow_z;
			}
			else if (shigh_t >= 0 && shigh_t <= thigh && shigh_z >= 0 && shigh_z <= zhigh)
			{
				T2 = shigh_t; S2 = shigh; Z2 = shigh_z;
			}
			else if (zlow_t >= 0 && zlow_t <= thigh && zlow_s >= 0 && zlow_s <= shigh)
			{
				T2 = zlow_t; S2 = zlow_s; Z2 = zlow;
			}
			else if (zhigh_t >= 0 && zhigh_t <= thigh && zhigh_s >= 0 && zhigh_s <= shigh)
			{
				T2 = zhigh_t; S2 = zhigh_s; Z2 = zhigh;
			}
		}
		else if (slow_t >= 0 && slow_t <= thigh && slow_z >= 0 && slow_z <= zhigh)
		{
			T1 = slow_t; S1 = slow; Z1 = slow_z;
			if (shigh_t >= 0 && shigh_t <= thigh && shigh_z >= 0 && shigh_z <= zhigh)
			{
				T2 = shigh_t; S2 = shigh; Z2 = shigh_z;
			}
			else if (zlow_t >= 0 && zlow_t <= thigh && zlow_s >= 0 && zlow_s <= shigh)
			{
				T2 = zlow_t; S2 = zlow_s; Z2 = zlow;
			}
			else if (zhigh_t >= 0 && zhigh_t <= thigh && zhigh_s >= 0 && zhigh_s <= shigh)
			{
				T2 = zhigh_t; S2 = zhigh_s; Z2 = zhigh;
			}
		}
		else if (shigh_t >= 0 && shigh_t <= thigh && shigh_z >= 0 && shigh_z <= zhigh)
		{
			T1 = shigh_t; S1 = shigh; Z1 = shigh_z;
			if (zlow_t >= 0 && zlow_t <= thigh && zlow_s >= 0 && zlow_s <= shigh)
			{
				T2 = zlow_t; S2 = zlow_s; Z2 = zlow;
			}
			else if (zhigh_t >= 0 && zhigh_t <= thigh && zhigh_s >= 0 && zhigh_s <= shigh)
			{
				T2 = zhigh_t; S2 = zhigh_s; Z2 = zhigh;
			}
		}
		else if (zlow_t >= 0 && zlow_t <= thigh && zlow_s >= 0 && zlow_s <= shigh)
		{
			T1 = zlow_t; S1 = zlow_s; Z1 = zlow;
			if (zhigh_t >= 0 && zhigh_t <= thigh && zhigh_s >= 0 && zhigh_s <= shigh)
			{
				T2 = zhigh_t; S2 = zhigh_s; Z2 = zhigh;
			}
		}
		else
		{
			//dev_Projection[threadid] = threadid;
			return;
		}

		LengthinROI = Distance(T1, S1, Z1, T2, S2, Z2);

	//	secondly compute length in a single pixel, the process is very similar to the previous.
	//	because this time the line goes through the center point of the pixel. So some kind of symmetry happens.
	//	since the global function can not call exterior function, so the previous code will be copied here.

	//	 actual Size in matlab coordinate
		tlow = Tindex * Resolution_t; thigh = (Tindex + 1) * Resolution_t; 
		slow = Sindex * Resolution_s; shigh = (Sindex + 1) * Resolution_s;
		zlow = Zindex * Resolution_z; zhigh = (Zindex + 1) * Resolution_z;

	//	compute the first and last point in the ROI
	//	 using DetectPoint_end set up projection equation

		tlow_s = source_s + (tlow - source_t) * (DetectPoint_send - source_s) / (DetectPoint_tend - source_t);
		tlow_z = source_z + (tlow - source_t) * (DetectPoint_zend - source_z) / (DetectPoint_tend - source_t);

		slow_t = source_t + (slow - source_s) * (DetectPoint_tend - source_t) / (DetectPoint_send - source_s);
		slow_z = source_z + (slow - source_s) * (DetectPoint_zend - source_z) / (DetectPoint_send - source_s);

		zlow_t = source_t + (zlow - source_z) * (DetectPoint_tend - source_t) / (DetectPoint_zend - source_z);
		zlow_s = source_s + (zlow - source_z) * (DetectPoint_send - source_s) / (DetectPoint_zend - source_z);

	//	double *Range = new double [6];   //  XYXY small-big(number)
		T1 = 0; S1 = 0; Z1 = 0; T2 = 0; S2 = 0; Z2 = 0;

		if (tlow_s >= slow && tlow_s <= shigh && tlow_z >= zlow && tlow_z <= zhigh)
		{
			T1 = tlow; S1 = tlow_s; Z1 = tlow_z;
			T2 = 2 * DetectPoint_tend - T1; 
			S2 = 2 * DetectPoint_send - S1; 
			Z2 = 2 * DetectPoint_zend - Z1;
	//		symmetry
		}
		else if (slow_t >= tlow && slow_t <= thigh && slow_z >= zlow && slow_z <= zhigh)
		{
			T1 = slow_t; S1 = slow; Z1 = slow_z;
			T2 = 2 * DetectPoint_tend - T1; 
			S2 = 2 * DetectPoint_send - S1; 
			Z2 = 2 * DetectPoint_zend - Z1;
	//		symmetry
		}
		else if (zlow_t >= tlow && zlow_t <= thigh && zlow_s >= slow && zlow_s <= shigh)
		{
			T1 = zlow_t; S1 = zlow_s; Z1 = zlow;
			T2 = 2 * DetectPoint_tend - T1; 
			S2 = 2 * DetectPoint_send - S1; 
			Z2 = 2 * DetectPoint_zend - Z1;
	//		symmetry		
		}
		else
		{
	//		dev_Projection[threadid] = threadid;
			return;
		}

		double LengthinPixel = Distance(T1, S1, Z1, T2, S2, Z2);
		backweight = LengthinPixel / LengthinROI ; 
	}

	dev_Display[thread_id] += Display_pBeta * backweight ;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t BackPro(float *Display, const float *R, const float *Xigamadomain, const float *Pdomain,
	const float *BetaScanRange, const double Distance, const int LBeta, const int LP, const int LXigama,
	const double *Size, const int t_length, const int s_length, const int z_length)
{
	mexPrintf("BackPro!\n");
	float *dev_R = 0, *dev_Rcov = 0;
	float *dev_G = 0;
	float *dev_BetaScanRange = 0, *dev_Pdomain = 0, *dev_Xigamadomain = 0;
	double *dev_Size = 0;

	float *dev_Display = 0;
	float PInt = fabs(Pdomain[1] - Pdomain[0]);
	float XigamaInt = fabs(Xigamadomain[1] - Xigamadomain[0]);
	float BetaScanInt = fabs(BetaScanRange[1] - BetaScanRange[0]);

	float maxP = MAX(Pdomain[0], Pdomain[LP - 1]);
	float minP = MIN(Pdomain[0], Pdomain[LP - 1]);
	float maxXigama = MAX(Xigamadomain[0], Xigamadomain[LXigama - 1]);
	float minXigama = MIN(Xigamadomain[0], Xigamadomain[LXigama - 1]);
	//mexPrintf("%lf %lf %lf %lf \n", maxGama, minGama, maxXigama, minXigama);

	const long LDisplay = t_length * s_length * z_length;
	const long LR = LP * LXigama * LBeta;
	const int LFilter = 2 * LP - 1;

	int thread_cubic_x = MIN(threadX, LP);
	int block_cubic_x = MIN(blockX, LXigama);

	const dim3 thread_cubic(thread_cubic_x, 1, 1);
	const dim3 block_cubic(block_cubic_x, 1, 1);

	dim3 thread_cubic_residual(1, 1, 1);  // initial
	dim3 block_cubic_residual(1, 1, 1);  // initial

	int LPResidual = LP % threadX;
	int LXigamaResidual = LXigama % blockX;
	int PTime = LP / threadX;
	int XigamaTime = LXigama / blockX;
	int Pstart = 0;
	int Xigamastart = 0;
	float Beta = 0;

	if (LPResidual != 0)
	{
		thread_cubic_residual.x = LPResidual;
	}
	if (LXigamaResidual != 0)
	{
		block_cubic_residual.x = LXigamaResidual;
	}

	const dim3 thread_cubic_Bp(t_length, 1, 1);
	const dim3 block_cubic_Bp(s_length, z_length, 1);

	hipError_t cudaStatus;

	mexPrintf("start cuda\n");

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?\n");
		mexPrintf("hipSetDevice failed!  Do you have a CUDA-capable GPU installed? %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	mexPrintf("call for space in GPU\n");

	// Allocate GPU buffers for four vectors (4 inputs).

	cudaStatus = hipMalloc((void**)&dev_R, LR * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_R hipMalloc failed!\n");
		mexPrintf("dev_R hipMalloc failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_Pdomain, LP * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_Pdomain hipMalloc failed!\n");
		mexPrintf("dev_Pdomain hipMalloc failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_BetaScanRange, LBeta * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_BetaScanRange hipMalloc failed!\n");
		mexPrintf("dev_BetaScanRange hipMalloc failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_Xigamadomain, LXigama * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_Xigamadomain hipMalloc failed!\n");
		mexPrintf("dev_Xigamadomain hipMalloc failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	//mexPrintf("copy data in CPU to GPU\n");

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_R, R, LR * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy R failed!\n");
		mexPrintf("hipMemcpy R failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_Pdomain, Pdomain, LP * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy Gamadomain failed!\n");
		mexPrintf("hipMemcpy Gamadomain failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_BetaScanRange, BetaScanRange, LBeta * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy BetaScanRange failed!\n");
		mexPrintf("hipMemcpy BetaScanRange failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_Xigamadomain, Xigamadomain, LXigama * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy Xigamadomain failed!\n");
		mexPrintf("hipMemcpy Xigamadomain failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	mexPrintf("start parallel computation\n");
	mexPrintf("backprojection\n");
	
	cudaStatus = hipMalloc((void**)&dev_Size, 3 * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_Size hipMalloc failed!\n");
		mexPrintf("dev_Size hipMalloc failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_Size, Size, 3 * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy Size failed!\n");
		mexPrintf("hipMemcpy Size failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}
	
    // output
	cudaStatus = hipMalloc((void**)&dev_Display, LDisplay * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_Display hipMalloc failed!\n");
		mexPrintf("dev_Display hipMalloc failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}
	hipMemset(dev_Display, 0, sizeof(float));

	//Backprojection
	for (int betaIndex = 0; betaIndex < LBeta; betaIndex++)
	{
		BackProjection << <block_cubic_Bp, thread_cubic_Bp >> > (dev_R, dev_Display, dev_Size, t_length, s_length, z_length,
			BetaScanRange[betaIndex], Distance, dev_Pdomain, dev_Xigamadomain, PInt, XigamaInt, BetaScanInt, minP, maxP, 
			minXigama, maxXigama, betaIndex, LP, LXigama);
	}

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "BackProjection launch failed: %s\n", hipGetErrorString(cudaStatus));
		mexPrintf("BackProjection launch failed %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		mexPrintf("hipDeviceSynchronize returned error code %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	 //Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(Display, dev_Display, LDisplay * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!\n");
		mexPrintf("hipMemcpy dev_Display failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

Error:
	hipFree(dev_R);
	hipFree(dev_BetaScanRange);
	hipFree(dev_Pdomain);
	hipFree(dev_Xigamadomain);
	hipFree(dev_Display);
	hipFree(dev_Size);

	mexPrintf("Exit Bakprojection\n");
	return cudaStatus;
}
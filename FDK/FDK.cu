#include "hip/hip_runtime.h"
#include "FDK.h"
// 2018/04/20 apply GPU acceleration
#define pow2(x) (1.0*(x)*(x))
__device__ const double PI = 3.141592653589793;

__global__ void GFunction(float *dev_G, const float PInt, const int Pstart, const int LP)
{

	const unsigned int Pindex = Pstart + threadIdx.x;

	// S_L filter 
	if (Pindex == 0)
		dev_G[LP - 1] = 1.0 / pow2(PI*PInt);
	else
	{
		dev_G[(LP - 1) - Pindex] = -1.0 / pow2(PInt * PI) / (4 * pow2(Pindex) - 1);
		dev_G[(LP - 1) + Pindex] = -1.0 / pow2(PInt * PI) / (4 * pow2(Pindex) - 1);
	}
}

__global__ void PreWeightFilter(float *dev_Rcov, float *dev_R, const float *dev_Pdomain, const float *dev_Xigamadomain,
	const double Distance, const float *dev_G, const float PInt, const int LP,
	const int LXigama, const int Pstart, const int Xigamastart, const int Betaindex, const int gstart,
	const int gend)
{
	const unsigned int Pindex = threadIdx.x + Pstart;
	const unsigned int Xigamaindex = blockIdx.x + Xigamastart;
	const unsigned long base_id = Betaindex * (LXigama * LP) + Xigamaindex * LP;
	const unsigned long thread_id = base_id + Pindex;

	float P = dev_Pdomain[Pindex];
	float Xigama = dev_Xigamadomain[Xigamaindex];

	double Proportion = Distance / sqrt(pow2(Distance) + pow2(P) + pow2(Xigama));

	dev_R[thread_id] = dev_R[thread_id] * Proportion;    // directly cover the input

	double Rcovsum = 0;
	// convolution

	for (int g = gstart; g < gend; g++)
	{
		//convolution
		Rcovsum += dev_R[base_id + g] * dev_G[(LP - 1) + Pindex - g];
	}

	dev_Rcov[thread_id] += PInt * Rcovsum;
}

__global__ void BackProjection(const float *dev_Rcov, float *dev_Display, const double *dev_Size,
	const int t_length, const int s_length, const int z_length, const float Beta,
	const double Distance,
	const float *dev_Pdomain, const float *dev_Xigamadomain, const float PInt, const float XigamaInt,
	const float BetaScanInt, const float minP, const float maxP, const float minXigama, const float maxXigama,
	const int betaIndex, const int LP, const int LXigama, const int T_start, const int S_start)
{
	// initialize
	const double Resolution_t = 1.0 * dev_Size[0] / t_length;
	const double Resolution_s = 1.0 * dev_Size[1] / s_length;
	const double Resolution_z = 1.0 * dev_Size[2] / z_length;
	// rotation center
	double Center_t = dev_Size[0] / 2;
	double Center_s = dev_Size[1] / 2;
	double Center_z = dev_Size[2] / 2;
	// index 
	const unsigned int Tindex = T_start + threadIdx.x;
	const unsigned int Sindex = S_start + blockIdx.x;

	double image_t, image_s, image_z, dect_t, dect_s, dect_z;
	unsigned int XigamaN1index, XigamaN2index, PN1index, PN2index;
	double P_domain1, P_domain2, Xigama_domain1, Xigama_domain2;
	double Xig1, Xig2, P1, P2;
	double Display_pBeta = 0;
	double LengthRatio;
	double Xigama, P;
	double Weight;
	unsigned long thread_id;

	for (int Zindex = 0; Zindex < z_length; Zindex++)
	{
		// this is a little different from code on MATLAB
		image_t = (Tindex + 0.5) * Resolution_t - Center_t;
		image_s = (Sindex + 0.5) * Resolution_s - Center_s;
		image_z = (Zindex + 0.5) * Resolution_z - Center_z;     // image pixel in ground coordinate

																// rotate in ground coordinate
		dect_t = image_t * cos(Beta) + image_s * sin(Beta);
		dect_s = -image_t * sin(Beta) + image_s * cos(Beta);
		dect_z = image_z;

		// define the projection position on the detector
		LengthRatio = Distance / (Distance - dect_s);
		Xigama = dect_z * LengthRatio;
		P = dect_t * LengthRatio;

		if ((P >= minP) && (P < maxP) && (Xigama >= minXigama) && (Xigama < maxXigama))
		{
			XigamaN1index = floor(fabs(Xigama - dev_Xigamadomain[0]) / XigamaInt);
			XigamaN2index = XigamaN1index + 1;
			PN1index = floor(fabs(P - dev_Pdomain[0]) / PInt);
			PN2index = PN1index + 1;

			P_domain1 = dev_Pdomain[PN1index]; P_domain2 = dev_Pdomain[PN2index];
			Xigama_domain1 = dev_Xigamadomain[XigamaN1index]; Xigama_domain2 = dev_Xigamadomain[XigamaN2index];

			//bilinear interpolation
			Xig1 = fabs(Xigama - Xigama_domain1); Xig2 = fabs(Xigama_domain2 - Xigama);
			P1 = fabs(P - P_domain1); P2 = fabs(P_domain2 - P);

			Weight = sqrt(1 + 0 * pow2(dect_z) / (pow2(Distance + dect_s) + pow2(dect_t)));
			//Display_pBeta = 1;
			Display_pBeta = (Xig2 * P2 * dev_Rcov[betaIndex * LP * LXigama + XigamaN1index * LP + PN1index]
				+ Xig1 * P2 * dev_Rcov[betaIndex * LP * LXigama + XigamaN2index * LP + PN1index]
				+ Xig2 * P1 * dev_Rcov[betaIndex * LP * LXigama + XigamaN1index * LP + PN2index]
				+ Xig1 * P1 * dev_Rcov[betaIndex * LP * LXigama + XigamaN2index * LP + PN2index])
				/ (PInt * XigamaInt) * pow2(LengthRatio) * BetaScanInt * Weight;
		}
		thread_id = Zindex * (t_length * s_length) + Sindex * t_length + Tindex;
		dev_Display[thread_id] += Display_pBeta;
	}

}

// Helper function for using CUDA to add vectors in parallel.
hipError_t FDKpro(float *Display, const float *R, const float *Xigamadomain, const float *Pdomain,
	const float *BetaScanRange, const double Distance, const int LBeta, const int LP, const int LXigama,
	const double *Size, const int t_length, const int s_length, const int z_length)
{
	mexPrintf("Hello FDK!\n");
	float *dev_R = 0, *dev_Rcov = 0;
	float *dev_G = 0;
	float *dev_BetaScanRange = 0, *dev_Pdomain = 0, *dev_Xigamadomain = 0;
	double *dev_Size = 0;

	float *dev_Display = 0;
	float PInt = fabs(Pdomain[1] - Pdomain[0]);
	float XigamaInt = fabs(Xigamadomain[1] - Xigamadomain[0]);
	float BetaScanInt = fabs(BetaScanRange[1] - BetaScanRange[0]);

	float maxP = MAX(Pdomain[0], Pdomain[LP - 1]);
	float minP = MIN(Pdomain[0], Pdomain[LP - 1]);
	float maxXigama = MAX(Xigamadomain[0], Xigamadomain[LXigama - 1]);
	float minXigama = MIN(Xigamadomain[0], Xigamadomain[LXigama - 1]);
	//mexPrintf("%lf %lf %lf %lf \n", maxGama, minGama, maxXigama, minXigama);

	const long LDisplay = t_length * s_length * z_length;
	const long LR = LP * LXigama * LBeta;
	const int LFilter = 2 * LP - 1;

	short thread_cubic_x = MIN(threadX, LP);
	short block_cubic_x = MIN(blockX, LXigama);
	short thread_cubic_Bp_x = MIN(threadX, t_length);
	short block_cubic_Bp_x = MIN(blockX, s_length);

	const dim3 thread_cubic(thread_cubic_x, 1, 1);
	const dim3 block_cubic(block_cubic_x, 1, 1);
	const dim3 thread_cubic_Bp(thread_cubic_Bp_x, 1, 1);
	const dim3 block_cubic_Bp(block_cubic_Bp_x, 1, 1);

	dim3 thread_cubic_residual(1, 1, 1);  // initial
	dim3 block_cubic_residual(1, 1, 1);  // initial
	dim3 thread_cubic_Bp_residual(1, 1, 1);  // initial
	dim3 block_cubic_Bp_residual(1, 1, 1);  // initial

	short LPResidual = LP % threadX;
	short LXigamaResidual = LXigama % blockX;
	short PTime = LP / threadX;
	short XigamaTime = LXigama / blockX;
	short Pstart = 0;
	short Xigamastart = 0;
	short TlengthResidual = t_length % threadX;
	short SlengthResidual = s_length % blockX;
	short T_Time = t_length / threadX;
	short S_Time = s_length / blockX;
	short T_start = 0;
	short S_start = 0;
	short gstart = 0, gend = 0;
	short gtime = 1;
	//mexPrintf("PTime: %d XigamaTime: %d\n", PTime, XigamaTime);

	if (LPResidual != 0)
	{
		thread_cubic_residual.x = LPResidual;
	}
	if (LXigamaResidual != 0)
	{
		block_cubic_residual.x = LXigamaResidual;
	}
	if (TlengthResidual != 0)
	{
		thread_cubic_Bp_residual.x = TlengthResidual;
	}
	if (SlengthResidual != 0)
	{
		block_cubic_Bp_residual.x = SlengthResidual;
	}

	mexPrintf("thread_cubic_x: %d block_cubic_x: %d LPResidual: %d LXigamaResidual: %d\n",
		thread_cubic_x, block_cubic_x, LPResidual, LXigamaResidual);

	hipError_t cudaStatus;

	///////////////////////////////////////////////////////////////////////////////////////////////	
	mexPrintf("start cuda\n");

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?\n");
		mexPrintf("hipSetDevice failed!  Do you have a CUDA-capable GPU installed? %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	mexPrintf("call for space in GPU\n");

	// Allocate GPU buffers for three vectors (two input, one output).

	cudaStatus = hipMalloc((void**)&dev_R, LR * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_R hipMalloc failed!\n");
		mexPrintf("dev_R hipMalloc failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error1;
	}

	cudaStatus = hipMalloc((void**)&dev_Pdomain, LP * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_Pdomain hipMalloc failed!\n");
		mexPrintf("dev_Pdomain hipMalloc failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_Xigamadomain, LXigama * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_Xigamadomain hipMalloc failed!\n");
		mexPrintf("dev_Xigamadomain hipMalloc failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_BetaScanRange, LBeta * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_BetaScanRange hipMalloc failed!\n");
		mexPrintf("dev_BetaScanRange hipMalloc failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	//mexPrintf("copy data in CPU to GPU\n");

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_R, R, LR * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy R failed!\n");
		mexPrintf("hipMemcpy R failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error1;
	}

	cudaStatus = hipMemcpy(dev_Pdomain, Pdomain, LP * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy Gamadomain failed!\n");
		mexPrintf("hipMemcpy Gamadomain failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_Xigamadomain, Xigamadomain, LXigama * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy Xigamadomain failed!\n");
		mexPrintf("hipMemcpy Xigamadomain failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_BetaScanRange, BetaScanRange, LBeta * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy BetaScanRange failed!\n");
		mexPrintf("hipMemcpy BetaScanRange failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	mexPrintf("start parallel computation\n");

	mexPrintf("Preweight and Filtering\n");
	// Preweight and filter 
	// to be adapted to GPU, we limited the number of threads in each grid (threadX * blockX * LBeta)
	// generate filter
	mexPrintf("Generate filter\n");
	cudaStatus = hipMalloc((void**)&dev_G, LFilter * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_G hipMalloc failed!\n");
		mexPrintf("dev_G hipMalloc failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error1;
	}
	hipMemset(dev_G, 0, sizeof(float));

	for (int numP = 0; numP < PTime; numP++)
	{
		Pstart = numP * threadX;
		GFunction << <1, thread_cubic_x >> > (dev_G, PInt, Pstart, LP);
		// Check for any errors launching the kernel
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "generate filter failed: %s\n", hipGetErrorString(cudaStatus));
			mexPrintf("generate filter launch failed %s\n", hipGetErrorString(cudaStatus));
			mexPrintf("Error happens at numP: %d\n", numP);
			goto Error;
		}
	}
	if (LPResidual != 0)
	{
		Pstart = LP - LPResidual;
		GFunction << <1, thread_cubic_residual.x >> > (dev_G, PInt, Pstart, LP);
		// Check for any errors launching the kernel
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "generate filter failed: %s\n", hipGetErrorString(cudaStatus));
			mexPrintf("generate filter launch failed %s\n", hipGetErrorString(cudaStatus));
			mexPrintf("Error happens at LXigamaResidual: %d\n", LXigamaResidual);
			goto Error;
		}
	}

	// store the result of convolution
	cudaStatus = hipMalloc((void**)&dev_Rcov, LR * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_Rcov hipMalloc failed!\n");
		mexPrintf("dev_Rcov hipMalloc failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}
	hipMemset(dev_Rcov, 0, sizeof(float));

	// convolution with limited scale
	gtime = ceil(1.0 * LP / Filterlengthlimit);
	mexPrintf("gtime: %d\n", gtime);

	for (int numB = 0; numB < LBeta; numB++)
	{
		//Beta = BetaScanRange[numB];
		for (int numP = 0; numP < PTime; numP++)
		{
			for (int numX = 0; numX < XigamaTime; numX++)
			{
				Pstart = numP * threadX;
				Xigamastart = numX * blockX;
				//mexPrintf("%d %d\n", Pstart, Xigamastart);
				for (int numg = 0; numg < gtime; numg++)
				{
					gstart = int(numg * Filterlengthlimit);
					if (numg != (gtime - 1))
					{
						gend = int((numg + 1)* Filterlengthlimit);
					}
					else
					{
						gend = LP;
					}
					/*mexPrintf("gstart: %d\n", gstart);
					mexPrintf("gend: %d\n", gend);*/
					PreWeightFilter << <block_cubic, thread_cubic >> > (dev_Rcov, dev_R, dev_Pdomain,
						dev_Xigamadomain, Distance, dev_G, PInt, LP, LXigama,
						Pstart, Xigamastart, numB, gstart, gend);
					// Check for any errors launching the kernel
					cudaStatus = hipGetLastError();
					if (cudaStatus != hipSuccess) {
						fprintf(stderr, "Filter launch failed: %s\n", hipGetErrorString(cudaStatus));
						mexPrintf("Filter launch failed %s\n", hipGetErrorString(cudaStatus));
						mexPrintf("Error happens at numB: %d Pstart: %d Xigamastart: %d numg: %d\n",
							numB, Pstart, Xigamastart, numg);
						goto Error;
					}
				}
			}
		}

		if (LPResidual != 0)
		{
			Pstart = LP - LPResidual;
			if (LXigamaResidual != 0)
			{
				Xigamastart = LXigama - LXigamaResidual;
				//("%d %d\n", Pstart, Xigamastart);
				for (int numg = 0; numg < gtime; numg++)
				{
					gstart = int(numg * Filterlengthlimit);
					if (numg != (gtime - 1))
					{
						gend = int((numg + 1)* Filterlengthlimit);
					}
					else
					{
						gend = LP;
					}
					/*mexPrintf("gstart: %d\n", gstart);
					mexPrintf("gend: %d\n", gend);*/
					PreWeightFilter << <block_cubic_residual, thread_cubic_residual >> > (dev_Rcov, dev_R, dev_Pdomain,
						dev_Xigamadomain, Distance, dev_G, PInt, LP, LXigama,
						Pstart, Xigamastart, numB, gstart, gend);
					// Check for any errors launching the kernel
					cudaStatus = hipGetLastError();
					if (cudaStatus != hipSuccess) {
						fprintf(stderr, "Filter launch failed: %s\n", hipGetErrorString(cudaStatus));
						mexPrintf("Filter launch failed %s\n", hipGetErrorString(cudaStatus));
						mexPrintf("Error happens at numB: %d Pstart: %d Xigamastart: %d numg: %d\n",
							numB, Pstart, Xigamastart, numg);
						goto Error;
					}
				}
			}

			for (int numX = 0; numX < XigamaTime; numX++)
			{
				Xigamastart = numX * blockX;
				//("%d %d\n", Pstart, Xigamastart);
				for (int numg = 0; numg < gtime; numg++)
				{
					gstart = int(numg * Filterlengthlimit);
					if (numg != (gtime - 1))
					{
						gend = int((numg + 1)* Filterlengthlimit);
					}
					else
					{
						gend = LP;
					}
					/*mexPrintf("gstart: %d\n", gstart);
					mexPrintf("gend: %d\n", gend);*/
					PreWeightFilter << <block_cubic, thread_cubic_residual >> > (dev_Rcov, dev_R, dev_Pdomain,
						dev_Xigamadomain, Distance, dev_G, PInt, LP, LXigama,
						Pstart, Xigamastart, numB, gstart, gend);
					// Check for any errors launching the kernel
					cudaStatus = hipGetLastError();
					if (cudaStatus != hipSuccess) {
						fprintf(stderr, "Filter launch failed: %s\n", hipGetErrorString(cudaStatus));
						mexPrintf("Filter launch failed %s\n", hipGetErrorString(cudaStatus));
						mexPrintf("Error happens at numB: %d Pstart: %d Xigamastart: %d numg: %d\n",
							numB, Pstart, Xigamastart, numg);
						goto Error;
					}
				}
			}
		}
		if (LXigamaResidual != 0)
		{
			Xigamastart = LXigama - LXigamaResidual;
			for (int numP = 0; numP < PTime; numP++)
			{
				Pstart = numP * threadX;
				//mexPrintf("%d %d\n", Pstart, Xigamastart);
				for (int numg = 0; numg < gtime; numg++)
				{
					gstart = int(numg * Filterlengthlimit);
					if (numg != (gtime - 1))
					{
						gend = int((numg + 1)* Filterlengthlimit);
					}
					else
					{
						gend = LP;
					}
					/*mexPrintf("gstart: %d\n", gstart);
					mexPrintf("gend: %d\n", gend);*/
					PreWeightFilter << <block_cubic_residual, thread_cubic >> > (dev_Rcov, dev_R, dev_Pdomain,
						dev_Xigamadomain, Distance, dev_G, PInt, LP, LXigama,
						Pstart, Xigamastart, numB, gstart, gend);
					// Check for any errors launching the kernel
					cudaStatus = hipGetLastError();
					if (cudaStatus != hipSuccess) {
						fprintf(stderr, "Filter launch failed: %s\n", hipGetErrorString(cudaStatus));
						mexPrintf("Filter launch failed %s\n", hipGetErrorString(cudaStatus));
						mexPrintf("Error happens at numB: %d Pstart: %d Xigamastart: %d numg: %d\n",
							numB, Pstart, Xigamastart, numg);
						goto Error;
					}
				}
			}
		}
	}

Error1:
	hipFree(dev_R);
	hipFree(dev_G);

	mexPrintf("backprojection\n");

	cudaStatus = hipMalloc((void**)&dev_Size, 3 * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_Size hipMalloc failed!\n");
		mexPrintf("dev_Size hipMalloc failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_Size, Size, 3 * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy Size failed!\n");
		mexPrintf("hipMemcpy Size failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_Display, LDisplay * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_Display hipMalloc failed!\n");
		mexPrintf("dev_Display hipMalloc failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}
	hipMemset(dev_Display, 0, sizeof(float));

	//Backprojection  
	for (int betaIndex = 0; betaIndex < LBeta; betaIndex++)
	{
		for (int numT = 0; numT < T_Time; numT++)
		{
			for (int numS = 0; numS < S_Time; numS++)
			{
				T_start = numT * threadX;
				S_start = numS * blockX;
				BackProjection << <block_cubic_Bp, thread_cubic_Bp >> > (dev_Rcov, dev_Display, dev_Size, t_length, s_length, z_length,
					BetaScanRange[betaIndex], Distance, dev_Pdomain, dev_Xigamadomain, PInt, XigamaInt, BetaScanInt, minP, maxP,
					minXigama, maxXigama, betaIndex, LP, LXigama, T_start, S_start);
				// Check for any errors launching the kernel
				cudaStatus = hipGetLastError();
				if (cudaStatus != hipSuccess) {
					fprintf(stderr, "BackProjection launch failed: %s\n", hipGetErrorString(cudaStatus));
					mexPrintf("BackProjection launch failed %s\n", hipGetErrorString(cudaStatus));
					mexPrintf("Error happens at betaIndex: %d\n", betaIndex);
					goto Error;
				}
			}
		}
		if (TlengthResidual != 0)
		{
			T_start = t_length - TlengthResidual;
			for (int numS = 0; numS < S_Time; numS++)
			{
				S_start = numS * blockX;
				BackProjection << <block_cubic_Bp, thread_cubic_Bp_residual >> > (dev_Rcov, dev_Display, dev_Size, t_length, s_length, z_length,
					BetaScanRange[betaIndex], Distance, dev_Pdomain, dev_Xigamadomain, PInt, XigamaInt, BetaScanInt, minP, maxP,
					minXigama, maxXigama, betaIndex, LP, LXigama, T_start, S_start);
				// Check for any errors launching the kernel
				cudaStatus = hipGetLastError();
				if (cudaStatus != hipSuccess) {
					fprintf(stderr, "BackProjection launch failed: %s\n", hipGetErrorString(cudaStatus));
					mexPrintf("BackProjection launch failed %s\n", hipGetErrorString(cudaStatus));
					mexPrintf("Error happens at betaIndex: %d\n", betaIndex);
					goto Error;
				}
			}
			if (SlengthResidual != 0)
			{
				S_start = s_length - SlengthResidual;
				BackProjection << <block_cubic_Bp_residual, thread_cubic_Bp_residual >> > (dev_Rcov, dev_Display, dev_Size, t_length, s_length, z_length,
					BetaScanRange[betaIndex], Distance, dev_Pdomain, dev_Xigamadomain, PInt, XigamaInt, BetaScanInt, minP, maxP,
					minXigama, maxXigama, betaIndex, LP, LXigama, T_start, S_start);
				// Check for any errors launching the kernel
				cudaStatus = hipGetLastError();
				if (cudaStatus != hipSuccess) {
					fprintf(stderr, "BackProjection launch failed: %s\n", hipGetErrorString(cudaStatus));
					mexPrintf("BackProjection launch failed %s\n", hipGetErrorString(cudaStatus));
					mexPrintf("Error happens at betaIndex: %d\n", betaIndex);
					goto Error;
				}
			}
		}
		if (SlengthResidual != 0)
		{
			S_start = s_length - SlengthResidual;
			for (int numT = 0; numT < T_Time; numT++)
			{
				T_start = numT * threadX;
				BackProjection << <block_cubic_Bp_residual, thread_cubic_Bp >> > (dev_Rcov, dev_Display, dev_Size, t_length, s_length, z_length,
					BetaScanRange[betaIndex], Distance, dev_Pdomain, dev_Xigamadomain, PInt, XigamaInt, BetaScanInt, minP, maxP,
					minXigama, maxXigama, betaIndex, LP, LXigama, T_start, S_start);
				// Check for any errors launching the kernel
				cudaStatus = hipGetLastError();
				if (cudaStatus != hipSuccess) {
					fprintf(stderr, "BackProjection launch failed: %s\n", hipGetErrorString(cudaStatus));
					mexPrintf("BackProjection launch failed %s\n", hipGetErrorString(cudaStatus));
					mexPrintf("Error happens at betaIndex: %d\n", betaIndex);
					goto Error;
				}
			}
		}		
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		mexPrintf("hipDeviceSynchronize returned error code %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	//Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(Display, dev_Display, LDisplay * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!\n");
		mexPrintf("hipMemcpy dev_Display failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

Error:
	hipFree(dev_BetaScanRange);
	hipFree(dev_Pdomain);
	hipFree(dev_Xigamadomain);
	hipFree(dev_Rcov);
	hipFree(dev_Display);
	hipFree(dev_Size);

	mexPrintf("Exit FDK\n");
	return cudaStatus;
}
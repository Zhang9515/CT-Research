#include "hip/hip_runtime.h"
#include "ProjectionParallel_2D.h"
// 2018/04/09
// 2019/03/24 edited by ZXZ
__device__ const double PI = 3.141592653589793;
__device__ const double EPS = 1e-10;

__global__ void ProjectionParallel(const float *dev_Pic, double *dev_Projection, const float *dev_t_Range, const float *dev_thetaRange,
	const double Center_y, const double Center_x, const double *dev_resolution, const int height, const int width, const int Lt, 
	const int tstart, const int thetastart)
{	
	const short tindex = threadIdx.x + tstart;
	const short thetaindex = blockIdx.x + thetastart;
	//for debug{
	//const int tindex = 364;
	//const int thetaindex = 2343;
	//}
	const int threadid = thetaindex * Lt + tindex;

	dev_Projection[threadid] = 0;

	float t = dev_t_Range[tindex];
	float theta = dev_thetaRange[thetaindex];
	double Smax = MAX(fabs(dev_t_Range[0]), fabs(dev_t_Range[Lt-1]));

	// according to euler equation
	double DetectPoint_xstart = Center_x + t * cos(-theta) + (-Smax) * sin(-theta);
	double DetectPoint_ystart = Center_y - t * sin(-theta) + (-Smax) * cos(-theta);
	// define end detect point in matlab coordinate, Note that : 0 is the start
	double DetectPoint_xend = Center_x + t * cos(-theta) + Smax * sin(-theta);
	double DetectPoint_yend = Center_y - t * sin(-theta) + Smax * cos(-theta);
	
	double X2Y = (DetectPoint_yend - DetectPoint_ystart) / (DetectPoint_xend - DetectPoint_xstart + EPS);
	double Y2X = 1 / (X2Y + EPS);

	// to determine the range of y
	short y_signal = 0;

	if (DetectPoint_yend >= DetectPoint_ystart)
		y_signal = 1;
	else if (DetectPoint_yend < DetectPoint_ystart)
		y_signal = -1;

	// to determine the range of x
	short x_signal = 0;

	if (DetectPoint_xend >= DetectPoint_xstart)			
		x_signal = 1;			
	else if (DetectPoint_xend < DetectPoint_xstart)
		x_signal = -1;			

	
	// // actual Size
	double Xlow = 0, Xhigh = width*dev_resolution[1], Ylow = 0, Yhigh = height*dev_resolution[0];

	//compute the first and last point in the ROI
	double Ylow_x = DetectPoint_xstart + (Ylow - DetectPoint_ystart) * Y2X /*/ (tan(theta + PI / 2) + EPS)*/;
	double Yhigh_x = DetectPoint_xstart + (Yhigh - DetectPoint_ystart) * Y2X /*/ (tan(theta + PI / 2) + EPS)*/;
	
	double Xlow_y = DetectPoint_ystart + (Xlow - DetectPoint_xstart) * X2Y /** tan(theta + PI / 2)*/;
	double Xhigh_y = DetectPoint_ystart + (Xhigh - DetectPoint_xstart) * X2Y /** tan(theta + PI / 2)*/;
	//double *Range = new double [4];   //  XYXY small-big(number)
	double X1 = 0, Y1 = 0, X2 = 0, Y2 = 0;

	if (Ylow_x >= 0 && Ylow_x <= Xhigh)
	{
		X1 = Ylow_x; Y1 = Ylow;
		if (Xlow_y >= 0 && Xlow_y <= Yhigh)
		{
			X2 = Xlow; Y2 = Xlow_y;
		}

		else if (Xhigh_y >= 0 && Xhigh_y <= Yhigh)
		{
			X2 = Xhigh; Y2 = Xhigh_y;
		}
		else if (Yhigh_x >= 0 && Yhigh_x <= Xhigh)
		{
			X2 = Yhigh_x; Y2 = Yhigh;
		}

	}
	else if (Yhigh_x >= 0 && Yhigh_x <= Xhigh)
	{
		X1 = Yhigh_x; Y1 = Yhigh;
		if (Xlow_y >= 0 && Xlow_y <= Yhigh)
		{
			X2 = Xlow; Y2 = Xlow_y;
		}
		else if (Xhigh_y >= 0 && Xhigh_y <= Yhigh)
		{
			X2 = Xhigh; Y2 = Xhigh_y;
		}

	}
	else if (Xlow_y >= 0 && Xlow_y <= Yhigh)
	{
		X1 = Xlow; Y1 = Xlow_y;
		if (Xhigh_y >= 0 && Xhigh_y <= Yhigh)
		{
			X2 = Xhigh; Y2 = Xhigh_y;
		}

	}
	else
	{
		//dev_Projection[threadid] = 8000;
		return;
	}
		
	// set the start point
	double XStart = 0, YStart = 0;
	if (Distancesq(X1, Y1, DetectPoint_xstart, DetectPoint_ystart) >= Distancesq(X2, Y2, DetectPoint_xstart, DetectPoint_ystart))
	{
		XStart = X2;
		YStart = Y2;
	}
	else
	{
		XStart = X1;
		YStart = Y1;
	}

	// adjust the order

	if (X2 < X1)
	{
		double c = X1;
		X1 = X2;
		X2 = c;
	}
	if (Y2 < Y1)
	{
		double c = Y1;
		Y1 = Y2;
		Y2 = c;
	}

	//// enter the ROI
	double weight = 0, Ray = 0;
	short GridX = 0, GridY = 0;        // candidate crosspoint index in matlab(0~256)
	double GridY_x = 0, GridX_y = 0;    // candidate crosspoint index in matlab(0~256)
	short DetectPoint_x = 0, DetectPoint_y = 0; // current pixel index in matlab pixel index in matlab(0~255)
	int	Pointid = 0;   
	double XCross = XStart / dev_resolution[1], YCross = YStart / dev_resolution[0];     // current crosspoint index in matlab(0~256)

	//while (((XCross * dev_resolution[1]) >= Range[0]) && ((XCross * dev_resolution[1]) <= Range[2]) 
	//	&& ((YCross * dev_resolution[0]) >= Range[1]) && ((YCross * dev_resolution[0]) <= Range[3]))
	for (short i = 0; i < (height + width - 1); i++)
	{
		//dev_Projection[threadid] = 10000;

		// judge whether XCross/YCross is integer
		if (XCross - (double)((short)XCross) < EPS)
		{
			GridX = XCross + x_signal;			
		}
		else
		{
			GridX = floor(XCross) + flag1to1or_1to0(x_signal);
		}
		GridX_y = (DetectPoint_ystart + (GridX * dev_resolution[1] - DetectPoint_xstart) * X2Y  /** tan(theta + PI / 2)*/) / dev_resolution[0];

		if (YCross - (double)((short)YCross) < EPS)
		{
			GridY = YCross + y_signal;			
		}
		else
		{
			GridY = floor(YCross) + flag1to1or_1to0(y_signal);
		}
		GridY_x = (DetectPoint_xstart + (GridY * dev_resolution[0] - DetectPoint_ystart) * Y2X /*/ (tan(theta + PI / 2) + EPS)*/) / dev_resolution[1];

		//judge which crosspoint is the nearest
		if (Distancesq(GridX, GridX_y, XCross, YCross) >= Distancesq(GridY_x, GridY, XCross, YCross))
		{
			weight = sqrt(Distancesq(GridY_x * dev_resolution[1], GridY * dev_resolution[0], XCross * dev_resolution[1], YCross * dev_resolution[0]));
			DetectPoint_x = floor(MID(GridY_x, XCross));                 // the midpoint locates the pixel
			DetectPoint_y = floor(MID(GridY, YCross));
			XCross = GridY_x;    // update
			YCross = GridY;
		}
		else
		{
			weight = sqrt(Distancesq(GridX * dev_resolution[1], GridX_y * dev_resolution[0], XCross * dev_resolution[1], YCross * dev_resolution[0]));
			DetectPoint_x = floor(MID(GridX, XCross));           
			DetectPoint_y = floor(MID(GridX_y, YCross));
			XCross = GridX;   // update
			YCross = GridX_y;
		}

		//judge whether the point is in the ROI
		if ((DetectPoint_x >= 0) && (DetectPoint_x < width) && (DetectPoint_y >= 0) && (DetectPoint_y < height ))
		{
			Pointid = DetectPoint_x * height + DetectPoint_y;
			Ray += weight * dev_Pic[Pointid];
			// for debug{
			//dev_Projection[threadid] = 8000;
			//break;
			// }
		}
		else
		{
			// for debug{
			//dev_Projection[threadid] = 9000;
			//}
			break;
		}
			
	}

	//__syncthreads();
	//for debug{
	//dev_Projection[threadid] = Ray;
	//}

	dev_Projection[threadid] = Ray;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t ProjectionParallel_2D(const float *Pic, double *Projection, const float *thetaRange, const float *t_Range,
	const int height, const int width, const double Center_y, const double Center_x, const int Ltheta, const int Lt,
	const double *resolution)
{
	mexPrintf("Hello GenMatParalell!\n");
	float *dev_Pic = 0, * dev_thetaRange = 0, *dev_t_Range = 0 ;
	double *dev_Projection = 0, *dev_resolution = 0;

	int threadcubic_x = MIN(threadX, Lt);
	int blockcubic_x = MIN(blockX, Ltheta);
	int LtResidual = Lt % threadX;
	int LthetaResidual = Ltheta % blockX;
	int tstart = 0;
	int thetastart = 0;
	int tTime = Lt / threadX;
	int thetaTime = Ltheta / blockX;

	const dim3 thread_cubic(threadcubic_x, 1, 1);
	const dim3 block_cubic(blockcubic_x, 1, 1);
	dim3 thread_cubic_residual(1, 1, 1);  // initial
	dim3 block_cubic_residual(1, 1, 1);  // initial

	mexPrintf("%d %d %d %d\n", threadcubic_x, blockcubic_x, LtResidual, LthetaResidual);

	if (LtResidual != 0)
	{
		thread_cubic_residual.x = LtResidual;
	}
	if (LthetaResidual != 0)
	{
		block_cubic_residual.x = LthetaResidual;
	}

	hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		mexPrintf("hipSetDevice failed!  Do you have a CUDA-capable GPU installed?\n");
        goto Error;
    }

	mexPrintf("Call for GPU space.\n");

    // Allocate GPU buffers for three vectors (two input, one output).
	cudaStatus = hipMalloc((void**)&dev_Pic, height * width * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_Pic hipMalloc failed!");
		mexPrintf("dev_Pic hipMalloc failed!\n");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_Projection, Ltheta * Lt * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dev_Projection hipMalloc failed!");
		mexPrintf("dev_Projection hipMalloc failed!\n");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_thetaRange, Ltheta * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dev_thetaRange hipMalloc failed!");
		mexPrintf("dev_thetaRange hipMalloc failed!\n");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_t_Range, Lt * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dev_t_Range hipMalloc failed!");
		mexPrintf("dev_t_Range hipMalloc failed!\n");
        goto Error;
    }

	cudaStatus = hipMalloc((void**)&dev_resolution, 2 * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_resolution hipMalloc failed!");
		mexPrintf("dev_resolution hipMalloc failed!\n");
		goto Error;
	}

	mexPrintf("Copy data from CPU to GPU.\n");

    // Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_Pic, Pic, height * width * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "thetaRange hipMemcpy failed!");
		mexPrintf("thetaRange hipMemcpy failed!\n");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_thetaRange, thetaRange, Ltheta * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "thetaRange hipMemcpy failed!");
		mexPrintf("thetaRange hipMemcpy failed!\n");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_t_Range, t_Range, Lt * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "t_Range hipMemcpy failed!");
		mexPrintf("t_Range hipMemcpy failed!\n");
        goto Error;
    }

	cudaStatus = hipMemcpy(dev_resolution, resolution, 2 * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_resolution hipMemcpy failed!");
		mexPrintf("dev_resolution hipMemcpy failed!\n");
		goto Error;
	}

	mexPrintf("Launch computation projection of each lines.\n");
    // Launch a kernel on the GPU with one thread for each element.
	
	for (int numt = 0; numt < tTime; numt++)
	{
		for (int numtheta = 0; numtheta < thetaTime; numtheta++)
		{
			tstart = numt * threadX;
			thetastart = numtheta * blockX;
			//mexPrintf("%d %d\n", Pstart, Xigamastart);
			ProjectionParallel << <block_cubic, thread_cubic >> > (dev_Pic, dev_Projection, dev_t_Range, dev_thetaRange,
				Center_y, Center_x, dev_resolution, height, width, Lt, tstart, thetastart);
			// Check for any errors launching the kernel
			cudaStatus = hipGetLastError();
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "ProjectionCone launch failed: %s\n", hipGetErrorString(cudaStatus));
				mexPrintf("ProjectionCone launch failed %s\n", hipGetErrorString(cudaStatus));
				mexPrintf("Error happens at tstarttstart: %d thetastart: %d \n",
					tstart, thetastart);
				goto Error;
			}
		}
	}

	if (LtResidual != 0)
	{
		tstart = Lt - LtResidual;
		if (LthetaResidual != 0)
		{
			thetastart = Ltheta - LthetaResidual;
			//("%d %d\n", Pstart, Xigamastart);
			ProjectionParallel << <block_cubic_residual, thread_cubic_residual >> > (dev_Pic, dev_Projection, dev_t_Range, dev_thetaRange,
				Center_y, Center_x, dev_resolution, height, width, Lt, tstart, thetastart);
			// Check for any errors launching the kernel
			cudaStatus = hipGetLastError();
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "ProjectionCone launch failed: %s\n", hipGetErrorString(cudaStatus));
				mexPrintf("ProjectionCone launch failed %s\n", hipGetErrorString(cudaStatus));
				mexPrintf("Error happens at tstarttstart: %d thetastart: %d \n",
					tstart, thetastart);
				goto Error;
			}
		}

		for (int numtheta = 0; numtheta < thetaTime; numtheta++)
		{
			thetastart = numtheta * blockX;
			//("%d %d\n", Pstart, Xigamastart);
			ProjectionParallel << <block_cubic, thread_cubic_residual >> > (dev_Pic, dev_Projection, dev_t_Range, dev_thetaRange,
				Center_y, Center_x, dev_resolution, height, width, Lt, tstart, thetastart);
			// Check for any errors launching the kernel
			cudaStatus = hipGetLastError();
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "ProjectionCone launch failed: %s\n", hipGetErrorString(cudaStatus));
				mexPrintf("ProjectionCone launch failed %s\n", hipGetErrorString(cudaStatus));
				mexPrintf("Error happens at tstarttstart: %d thetastart: %d \n",
					tstart, thetastart);
				goto Error;
			}
		}
	}
	if (LthetaResidual != 0)
	{
		thetastart = Ltheta - LthetaResidual;
		for (int numt = 0; numt < tTime; numt++)
		{
			tstart = numt * threadX;
			//mexPrintf("%d %d\n", Pstart, Xigamastart);
			ProjectionParallel << <block_cubic_residual, thread_cubic >> > (dev_Pic, dev_Projection, dev_t_Range, dev_thetaRange,
				Center_y, Center_x, dev_resolution, height, width, Lt, tstart, thetastart);
			// Check for any errors launching the kernel
			cudaStatus = hipGetLastError();
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "ProjectionCone launch failed: %s\n", hipGetErrorString(cudaStatus));
				mexPrintf("ProjectionCone launch failed %s\n", hipGetErrorString(cudaStatus));
				mexPrintf("Error happens at tstarttstart: %d thetastart: %d \n",
					tstart, thetastart);
				goto Error;
			}
		}
	}

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Projection launch failed: %s\n", hipGetErrorString(cudaStatus));
		mexPrintf("Projection launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}
   
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		mexPrintf("hipDeviceSynchronize returned error\n");
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(Projection, dev_Projection, Ltheta * Lt * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!\n");
        goto Error;
    }

Error:
    hipFree(dev_t_Range);
    hipFree(dev_thetaRange);
    hipFree(dev_Projection);
	hipFree(dev_Pic);
	hipFree(dev_resolution);

    return cudaStatus;
}

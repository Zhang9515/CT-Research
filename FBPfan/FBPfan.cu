#include "hip/hip_runtime.h"
#include "FBPfan.h"
// 2019/09/14 apply GPU acceleration
#define pow2(x) (1.0*(x)*(x))
__device__ const double PI = 3.141592653589793;

__global__ void GFunction(float *dev_G, const float PInt, const int pstart, const int LP)
{

	const unsigned int Pindex = pstart + threadIdx.x;

	// S_L filter 
	if (Pindex == 0)
		dev_G[LP - 1] = 1.0 / pow2(PI*PInt);
	else
	{
		dev_G[(LP - 1) - Pindex] = -1.0 / pow2(PInt * PI) / (4 * pow2(Pindex) - 1);
		dev_G[(LP - 1) + Pindex] = -1.0 / pow2(PInt * PI) / (4 * pow2(Pindex) - 1);
	}
}

__global__ void Filter(float *dev_Rcov, float *dev_R, const float *dev_G, const float PInt, 
	const int LP, const int pstart, const int Betastart, const int gstart, const int gend)
{
	const unsigned int Pindex = threadIdx.x + pstart;
	const unsigned int BetaIndex = blockIdx.x + Betastart;
	const unsigned long base_id = BetaIndex * LP;
	const unsigned long thread_id = base_id + Pindex;

	double Rcovsum = 0;
	// convolution

	for (int g = gstart; g < gend; g++)
	{
		//convolution
		Rcovsum += dev_R[base_id + g] * dev_G[(LP - 1) + Pindex - g];
	}

	dev_Rcov[thread_id] += PInt * Rcovsum;
}

__global__ void BackProjection(const float *dev_Rcov, float *dev_Display, const double *dev_Size, const int T_length, const int S_length,
	const float Beta, const float *dev_Pdomain, const float PInt, const float BetaScanInt, const float minP, 
	const float maxP, const int BetaIndex, const int LP, const int LBeta, const int T_start, const int S_start, const double Rscan)
{
	// here makes clear an important note: t in lower case means the location on the detector, while T in captial case means a dimension of the axis.
	// only take effect in this function
	// initialize
	const double Resolution_t = 1.0 * dev_Size[0] / T_length;
	const double Resolution_s = 1.0 * dev_Size[1] / S_length;
	// rotation center
	double Center_t = dev_Size[0] / 2;
	double Center_s = dev_Size[1] / 2;
	// index 
	const unsigned int Tindex = T_start + threadIdx.x;
	const unsigned int Sindex = S_start + blockIdx.x;

	double image_t, image_s;
	unsigned int P1index, P2index;
	double P_domain1, P_domain2;
	double P1, P2;
	double Display_pBeta = 0;
	double P, S, U, P_domain;
	unsigned long thread_id;

	// this is a little different from code on MATLAB
	image_t = (Tindex + 0.5) * Resolution_t - Center_t;    // center of the image pixel in ground coordinate
	image_s = (Sindex + 0.5) * Resolution_s - Center_s;

	P = image_t * cos(Beta) + image_s * sin(Beta);   // rotate in ground coordinate
	S = -image_t * sin(Beta) + image_s * cos(Beta);
	U = (Rscan - S) / Rscan;       // proportion of length

	P_domain = P / U;
	
	if ((P_domain >= minP) && (P_domain < maxP))
	{
		P1index = floor(fabs(P_domain - dev_Pdomain[0]) / PInt);
		P2index = P1index + 1;

		P_domain1 = dev_Pdomain[P1index]; P_domain2 = dev_Pdomain[P2index];
	
		//bilinear interpolation
		P1 = fabs(P_domain - P_domain1); P2 = fabs(P_domain2 - P_domain);

		//Display_pBeta = 1;
		Display_pBeta = ( P2 * dev_Rcov[BetaIndex * LP + P1index] + P1 * dev_Rcov[BetaIndex * LP + P2index] )
			/ (pow2(U) * PInt) * BetaScanInt;
	}
	thread_id = Sindex * T_length + Tindex;
	dev_Display[thread_id] += Display_pBeta;

}

// HeLTer function for using CUDA to add vectors in parallel.
hipError_t FBPfan(float *Display, const float *R, const float *Pdomain, const float *BetaScanRange, const int LBeta,
	const int LP, const double *Size, const int t_length, const int s_length, const double Rscan)
{
	mexPrintf("Hello FBPpara!\n");
	float *dev_R = 0, *dev_Rcov = 0;
	float *dev_G = 0;
	float *dev_BetaScanRange = 0, *dev_Pdomain = 0;
	double *dev_Size = 0;

	float *dev_Display = 0;
	float PInt = fabs(Pdomain[1] - Pdomain[0]);
	float BetaScanInt = fabs(BetaScanRange[1] - BetaScanRange[0]);

	float maxP = MAX(Pdomain[0], Pdomain[LP - 1]);
	float minP = MIN(Pdomain[0], Pdomain[LP - 1]);
	//mexPrintf("%lf %lf %lf %lf \n", maxGama, minGama, maxXigama, minXigama);

	const long LDisplay = t_length * s_length;
	const long LR = LP * LBeta;
	const int LFilter = 2 * LP - 1;

	short thread_cubic_x = MIN(threadX, LP);
	short block_cubic_x = MIN(blockX, LBeta);
	short thread_cubic_Bp_x = MIN(threadX, t_length);
	short block_cubic_Bp_x = MIN(blockX, s_length);

	const dim3 thread_cubic(thread_cubic_x, 1, 1);
	const dim3 block_cubic(block_cubic_x, 1, 1);
	const dim3 thread_cubic_Bp(thread_cubic_Bp_x, 1, 1);
	const dim3 block_cubic_Bp(block_cubic_Bp_x, 1, 1);

	dim3 thread_cubic_residual(1, 1, 1);  // initial
	dim3 block_cubic_residual(1, 1, 1);  // initial
	dim3 thread_cubic_Bp_residual(1, 1, 1);  // initial
	dim3 block_cubic_Bp_residual(1, 1, 1);  // initial

	short LPResidual = LP % threadX;
	short LBetaResidual = LBeta % blockX;
	short pTime = LP / threadX;
	short BetaTime = LBeta / blockX;
	short pstart = 0;
	short Betastart = 0;
	short TlengthResidual = t_length % threadX;
	short SlengthResidual = s_length % blockX;
	short T_Time = t_length / threadX;
	short S_Time = s_length / blockX;
	short T_start = 0;
	short S_start = 0;
	short gstart = 0, gend = 0;
	short gtime = 1;
	//mexPrintf("pTime: %d BetaTime: %d\n", pTime, BetaTime);

	if (LPResidual != 0)
	{
		thread_cubic_residual.x = LPResidual;
	}
	if (LBetaResidual != 0)
	{
		block_cubic_residual.x = LBetaResidual;
	}
	if (TlengthResidual != 0)
	{
		thread_cubic_Bp_residual.x = TlengthResidual;
	}
	if (SlengthResidual != 0)
	{
		block_cubic_Bp_residual.x = SlengthResidual;
	}

	mexPrintf("thread_cubic_x: %d block_cubic_x: %d LPResidual: %d LBetaResidual: %d\n",
		thread_cubic_x, block_cubic_x, LPResidual, LBetaResidual);
	mexPrintf("thread_cubic_Bp_x: %d block_cubic_Bp_x: %d TlengthResidual: %d SlengthResidual: %d\n",
		thread_cubic_Bp_x, block_cubic_Bp_x, TlengthResidual, SlengthResidual);

	hipError_t cudaStatus;

	///////////////////////////////////////////////////////////////////////////////////////////////	
	mexPrintf("start cuda\n");

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?\n");
		mexPrintf("hipSetDevice failed!  Do you have a CUDA-capable GPU installed? %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	mexPrintf("call for space in GPU\n");

	// Allocate GPU buffers for three vectors (two input, one output).

	cudaStatus = hipMalloc((void**)&dev_R, LR * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_R hipMalloc failed!\n");
		mexPrintf("dev_R hipMalloc failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error1;
	}

	cudaStatus = hipMalloc((void**)&dev_Pdomain, LP * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_Pdomain hipMalloc failed!\n");
		mexPrintf("dev_Pdomain hipMalloc failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_BetaScanRange, LBeta * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_BetaScanRange hipMalloc failed!\n");
		mexPrintf("dev_BetaScanRange hipMalloc failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	//mexPrintf("copy data in CPU to GPU\n");

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_R, R, LR * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy R failed!\n");
		mexPrintf("hipMemcpy R failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error1;
	}

	cudaStatus = hipMemcpy(dev_Pdomain, Pdomain, LP * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy Pdomain failed!\n");
		mexPrintf("hipMemcpy Pdomain failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_BetaScanRange, BetaScanRange, LBeta * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy BetaScanRange failed!\n");
		mexPrintf("hipMemcpy BetaScanRange failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	mexPrintf("start parallel computation\n");

	mexPrintf("Preweight and Filtering\n");
	// Preweight and filter 
	// to be adapted to GPU, we limited the number of threads in each grid (threadX * blockX * LBeta)
	// generate filter
	mexPrintf("Generate filter\n");
	cudaStatus = hipMalloc((void**)&dev_G, LFilter * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_G hipMalloc failed!\n");
		mexPrintf("dev_G hipMalloc failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error1;
	}
	hipMemset(dev_G, 0, sizeof(float));

	for (int numP = 0; numP < pTime; numP++)
	{
		pstart = numP * threadX;
		GFunction << <1, thread_cubic_x >> > (dev_G, PInt, pstart, LP);
		// Check for any errors launching the kernel
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "generate filter failed: %s\n", hipGetErrorString(cudaStatus));
			mexPrintf("generate filter launch failed %s\n", hipGetErrorString(cudaStatus));
			mexPrintf("Error happens at numP: %d\n", numP);
			goto Error;
		}
	}
	if (LPResidual != 0)
	{
		pstart = LP - LPResidual;
		GFunction << <1, thread_cubic_residual.x >> > (dev_G, PInt, pstart, LP);
		// Check for any errors launching the kernel
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "generate filter failed: %s\n", hipGetErrorString(cudaStatus));
			mexPrintf("generate filter launch failed %s\n", hipGetErrorString(cudaStatus));
			mexPrintf("Error happens at LPResidual: %d\n", LPResidual);
			goto Error;
		}
	}

	// store the result of convolution
	cudaStatus = hipMalloc((void**)&dev_Rcov, LR * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_Rcov hipMalloc failed!\n");
		mexPrintf("dev_Rcov hipMalloc failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}
	hipMemset(dev_Rcov, 0, sizeof(float));

	// convolution with limited scale
	gtime = ceil(1.0 * LP / Filterlengthlimit);
	mexPrintf("gtime: %d\n", gtime);

	for (int numP = 0; numP < pTime; numP++)
	{
		for (int numBeta = 0; numBeta < BetaTime; numBeta++)
		{
			pstart = numP * threadX;
			Betastart = numBeta * blockX;
			//mexPrintf("%d %d\n", pstart, Betastart);
			for (int numg = 0; numg < gtime; numg++)
			{
				gstart = int(numg * Filterlengthlimit);
				if (numg != (gtime - 1))
				{
					gend = int((numg + 1)* Filterlengthlimit);
				}
				else
				{
					gend = LP;
				}
				/*mexPrintf("gstart: %d\n", gstart);
				mexPrintf("gend: %d\n", gend);*/
				Filter << <block_cubic, thread_cubic >> > (dev_Rcov, dev_R, dev_G, PInt, LP, pstart, Betastart, gstart, gend);

				// Check for any errors launching the kernel
				cudaStatus = hipGetLastError();
				if (cudaStatus != hipSuccess) {
					fprintf(stderr, "Filter launch failed: %s\n", hipGetErrorString(cudaStatus));
					mexPrintf("Filter launch failed %s\n", hipGetErrorString(cudaStatus));
					mexPrintf("Error happens at pstart: %d Betastart: %d numg: %d\n",
						pstart, Betastart, numg);
					goto Error;
				}
			}
		}
	}

	if (LPResidual != 0)
	{
		pstart = LP - LPResidual;
		if (LBetaResidual != 0)
		{
			Betastart = LBeta - LBetaResidual;
			//("%d %d\n", pstart, Betastart);
			for (int numg = 0; numg < gtime; numg++)
			{
				gstart = int(numg * Filterlengthlimit);
				if (numg != (gtime - 1))
				{
					gend = int((numg + 1)* Filterlengthlimit);
				}
				else
				{
					gend = LP;
				}
				/*mexPrintf("gstart: %d\n", gstart);
				mexPrintf("gend: %d\n", gend);*/
				Filter << <block_cubic_residual, thread_cubic_residual >> > (dev_Rcov, dev_R, dev_G, PInt, LP, pstart, Betastart, gstart, gend);
				// Check for any errors launching the kernel
				cudaStatus = hipGetLastError();
				if (cudaStatus != hipSuccess) {
					fprintf(stderr, "Filter launch failed: %s\n", hipGetErrorString(cudaStatus));
					mexPrintf("Filter launch failed %s\n", hipGetErrorString(cudaStatus));
					mexPrintf("Error happens at pstart: %d Betastart: %d numg: %d\n",
						pstart, Betastart, numg);
					goto Error;
				}
			}
		}

		for (int numBeta = 0; numBeta < BetaTime; numBeta++)
		{
			Betastart = numBeta * blockX;
			//("%d %d\n", pstart, Betastart);
			for (int numg = 0; numg < gtime; numg++)
			{
				gstart = int(numg * Filterlengthlimit);
				if (numg != (gtime - 1))
				{
					gend = int((numg + 1)* Filterlengthlimit);
				}
				else
				{
					gend = LP;
				}
				/*mexPrintf("gstart: %d\n", gstart);
				mexPrintf("gend: %d\n", gend);*/
				Filter << <block_cubic, thread_cubic_residual >> > (dev_Rcov, dev_R, dev_G, PInt, LP, pstart, Betastart, gstart, gend);
				// Check for any errors launching the kernel
				cudaStatus = hipGetLastError();
				if (cudaStatus != hipSuccess) {
					fprintf(stderr, "Filter launch failed: %s\n", hipGetErrorString(cudaStatus));
					mexPrintf("Filter launch failed %s\n", hipGetErrorString(cudaStatus));
					mexPrintf("Error happens at pstart: %d Betastart: %d numg: %d\n",
						pstart, Betastart, numg);
					goto Error;
				}
			}
		}
	}
	if (LBetaResidual != 0)
	{
		Betastart = LBeta - LBetaResidual;
		for (int numP = 0; numP < pTime; numP++)
		{
			pstart = numP * threadX;
			//mexPrintf("%d %d\n", pstart, Betastart);
			for (int numg = 0; numg < gtime; numg++)
			{
				gstart = int(numg * Filterlengthlimit);
				if (numg != (gtime - 1))
				{
					gend = int((numg + 1)* Filterlengthlimit);
				}
				else
				{
					gend = LP;
				}
				/*mexPrintf("gstart: %d\n", gstart);
				mexPrintf("gend: %d\n", gend);*/
				Filter << <block_cubic_residual, thread_cubic >> > (dev_Rcov, dev_R, dev_G, PInt, LP, pstart, Betastart, gstart, gend);
				// Check for any errors launching the kernel
				cudaStatus = hipGetLastError();
				if (cudaStatus != hipSuccess) {
					fprintf(stderr, "Filter launch failed: %s\n", hipGetErrorString(cudaStatus));
					mexPrintf("Filter launch failed %s\n", hipGetErrorString(cudaStatus));
					mexPrintf("Error happens at pstart: %d Betastart: %d numg: %d\n",
						pstart, Betastart, numg);
					goto Error;
				}
			}
		}
	}

Error1:
	hipFree(dev_R);
	hipFree(dev_G);

	mexPrintf("backprojection\n");

	cudaStatus = hipMalloc((void**)&dev_Size, 2 * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_Size hipMalloc failed!\n");
		mexPrintf("dev_Size hipMalloc failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_Size, Size, 2 * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy Size failed!\n");
		mexPrintf("hipMemcpy Size failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_Display, LDisplay * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_Display hipMalloc failed!\n");
		mexPrintf("dev_Display hipMalloc failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}
	hipMemset(dev_Display, 0, sizeof(float));

	//Backprojection  
	for (int BetaIndex = 0; BetaIndex < LBeta; BetaIndex++)
	{
		for (int numT = 0; numT < T_Time; numT++)
		{
			for (int numS = 0; numS < S_Time; numS++)
			{
				T_start = numT * threadX;
				S_start = numS * blockX;
				BackProjection << <block_cubic_Bp, thread_cubic_Bp >> > (dev_Rcov, dev_Display, dev_Size, t_length, s_length,
					BetaScanRange[BetaIndex], dev_Pdomain, PInt, BetaScanInt, minP, maxP, BetaIndex, LP, LBeta, T_start, S_start, Rscan);
				
				// Check for any errors launching the kernel
				cudaStatus = hipGetLastError();
				if (cudaStatus != hipSuccess) {
					fprintf(stderr, "BackProjection launch failed: %s\n", hipGetErrorString(cudaStatus));
					mexPrintf("BackProjection launch failed %s\n", hipGetErrorString(cudaStatus));
					mexPrintf("Error happens at BetaIndex: %d\n", BetaIndex);
					goto Error;
				}
			}
		}
		if (TlengthResidual != 0)
		{
			T_start = t_length - TlengthResidual;
			for (int numS = 0; numS < S_Time; numS++)
			{
				S_start = numS * blockX;
				BackProjection << <block_cubic_Bp, thread_cubic_Bp_residual >> > (dev_Rcov, dev_Display, dev_Size, t_length, s_length,
					BetaScanRange[BetaIndex], dev_Pdomain, PInt, BetaScanInt, minP, maxP, BetaIndex, LP, LBeta, T_start, S_start, Rscan);
				// Check for any errors launching the kernel
				cudaStatus = hipGetLastError();
				if (cudaStatus != hipSuccess) {
					fprintf(stderr, "BackProjection launch failed: %s\n", hipGetErrorString(cudaStatus));
					mexPrintf("BackProjection launch failed %s\n", hipGetErrorString(cudaStatus));
					mexPrintf("Error happens at BetaIndex: %d\n", BetaIndex);
					goto Error;
				}
			}
			if (SlengthResidual != 0)
			{
				S_start = s_length - SlengthResidual;
				BackProjection << <block_cubic_Bp_residual, thread_cubic_Bp_residual >> > (dev_Rcov, dev_Display, dev_Size, t_length, s_length,
					BetaScanRange[BetaIndex], dev_Pdomain, PInt, BetaScanInt, minP, maxP, BetaIndex, LP, LBeta, T_start, S_start, Rscan);
				// Check for any errors launching the kernel
				cudaStatus = hipGetLastError();
				if (cudaStatus != hipSuccess) {
					fprintf(stderr, "BackProjection launch failed: %s\n", hipGetErrorString(cudaStatus));
					mexPrintf("BackProjection launch failed %s\n", hipGetErrorString(cudaStatus));
					mexPrintf("Error happens at BetaIndex: %d\n", BetaIndex);
					goto Error;
				}
			}
		}
		if (SlengthResidual != 0)
		{
			S_start = s_length - SlengthResidual;
			for (int numT = 0; numT < T_Time; numT++)
			{
				T_start = numT * threadX;
				BackProjection << <block_cubic_Bp_residual, thread_cubic_Bp >> > (dev_Rcov, dev_Display, dev_Size, t_length, s_length,
					BetaScanRange[BetaIndex], dev_Pdomain, PInt, BetaScanInt, minP, maxP, BetaIndex, LP, LBeta, T_start, S_start, Rscan);
				// Check for any errors launching the kernel
				cudaStatus = hipGetLastError();
				if (cudaStatus != hipSuccess) {
					fprintf(stderr, "BackProjection launch failed: %s\n", hipGetErrorString(cudaStatus));
					mexPrintf("BackProjection launch failed %s\n", hipGetErrorString(cudaStatus));
					mexPrintf("Error happens at BetaIndex: %d\n", BetaIndex);
					goto Error;
				}
			}
		}		
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		mexPrintf("hipDeviceSynchronize returned error code %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	//Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(Display, dev_Display, LDisplay * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!\n");
		mexPrintf("hipMemcpy dev_Display failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}
	//for debug{
	//cudaStatus = hipMemcpy(Rcov, dev_Rcov, LR * sizeof(float), hipMemcpyDeviceToHost);
	//if (cudaStatus != hipSuccess) {
	//	fprintf(stderr, "hipMemcpy failed!\n");
	//	mexPrintf("hipMemcpy dev_Display failed! %s\n", hipGetErrorString(cudaStatus));
	//	goto Error;
	//}
	//}end debug

Error:
	hipFree(dev_BetaScanRange);
	hipFree(dev_Pdomain);
	hipFree(dev_Rcov);
	hipFree(dev_Display);
	hipFree(dev_Size);

	mexPrintf("Exit FBP\n");
	return cudaStatus;
}
#include "hip/hip_runtime.h"
#include "ProjectionFan_2D.h"
#include <cstdlib>
// 2018/04/16
//__device__ const double PI = 3.141592653589793;
__device__ const double EPS = 1e-15;
__device__ const double ERR = 1e-5;

__global__ void ProjectionFan(const float *dev_Pic, double *dev_Projection, const double *dev_Pdomain, const double *dev_BetaScanRange,
	const double Center_t, const double Center_s, const double *dev_resolution, const int t_length, const int s_length, 
	const int Pstart, const int Betastart, const double Distance, const int LP, const int LBeta, const double *dev_RandomErr)
{  
	const unsigned short Pindex = threadIdx.x + Pstart;
	const unsigned short  Betaindex = blockIdx.x + Betastart;

	const unsigned long threadid = Betaindex * LP + Pindex;

	dev_Projection[threadid] = 0;

	double P = dev_Pdomain[Pindex];
	double Beta = dev_BetaScanRange[Betaindex];

	double resolution_1 = dev_resolution[0]; double resolution_2 = dev_resolution[1]; 

	// according to euler equation   
	double source_t = Center_t - Distance * sin(Beta);      // define the source in matlab coordinate
	//double source_t = Center_t - Distance * sin(Beta) + dev_RandomErr[threadid];  
	double source_s = Center_s + Distance * cos(Beta);

	double Theta = atan(P / Distance);        // radian angle in s'-t coordinate plane 

	double Smax = 2 * Distance;

	// define end detect point in matlab coordinate, Note that : 0 is the start
	double DetectPoint_tend = Center_t + Smax * sin(Theta) * cos(Beta) - (Distance - Smax * cos(Theta)) * sin(Beta);
	double DetectPoint_send = Center_s + Smax * sin(Theta) * sin(Beta) + (Distance - Smax * cos(Theta)) * cos(Beta);

	double T2S, S2T;
	if ((DetectPoint_tend - source_t) == 0)
		T2S = (DetectPoint_send - source_s) / (DetectPoint_tend - source_t + EPS);
	else
		T2S = (DetectPoint_send - source_s) / (DetectPoint_tend - source_t);		
	if ((DetectPoint_send - source_s) == 0)
		S2T = (DetectPoint_tend - source_t) / (DetectPoint_send - source_s + EPS);
	else
		S2T = (DetectPoint_tend - source_t) / (DetectPoint_send - source_s);
		
	// limit the range of slope
	T2S = Maxlim(T2S); T2S = Minlim(T2S);
	S2T = Maxlim(S2T); S2T = Minlim(S2T);

	// to determine the range of t
	short t_signal = 0;

	if (DetectPoint_tend >= source_t)
		t_signal = 1;
	else
		t_signal = -1;

	// to determine the range of s
	short s_signal = 0;

	if (DetectPoint_send >= source_s)
		s_signal = 1;
	else
		s_signal = -1;

	// actual Size
	double tlow = 0, thigh = t_length*resolution_1, slow = 0, shigh = s_length*resolution_2;

	//compute the first and last point in the ROI
	// using DetectPoint_end set up projection equation
	double tlow_s = source_s + (tlow - source_t) * T2S;
	double thigh_s = source_s + (thigh - source_t) * T2S;

	double slow_t = source_t + (slow - source_s) * S2T;
	double shigh_t = source_t + (shigh - source_s) * S2T;

	//double *Range = new double [6];   //  XYXY small-big(number)
	double T1 = 0, S1 = 0, T2 = 0, S2 = 0;

	if (tlow_s >= 0 && tlow_s <= shigh )
	{
		T1 = tlow; S1 = tlow_s; 
		if (thigh_s >= 0 && thigh_s <= shigh)
		{
			T2 = thigh; S2 = thigh_s;
		}
		else if (slow_t >= 0 && slow_t <= thigh)
		{
			T2 = slow_t; S2 = slow;
		}
		else if (shigh_t >= 0 && shigh_t <= thigh)
		{
			T2 = shigh_t; S2 = shigh;
		}
	}
	else if (thigh_s >= 0 && thigh_s <= shigh )
	{
		T1 = thigh; S1 = thigh_s;
		if (slow_t >= 0 && slow_t <= thigh)
		{
			T2 = slow_t; S2 = slow;
		}
		else if (shigh_t >= 0 && shigh_t <= thigh)
		{
			T2 = shigh_t; S2 = shigh;
		}
	}
	else if (slow_t >= 0 && slow_t <= thigh)
	{
		T1 = slow_t; S1 = slow;
		if (shigh_t >= 0 && shigh_t <= thigh)
		{
			T2 = shigh_t; S2 = shigh; 
		}
	}
	else
	{
		//dev_Projection[threadid] = threadid;
		return;
	}

	// set the start point
	double TStart = 0, SStart = 0;
	if (Distancesq(T1, S1, source_t, source_s) >= Distancesq(T2, S2, source_t, source_s))
	{
		TStart = T2;
		SStart = S2;
	}
	else
	{
		TStart = T1;
		SStart = S1;
	}

	// adjust the order

	if (T2 < T1)
	{
		double c = T1;
		T1 = T2;
		T2 = c;
	}
	if (S2 < S1)
	{
		double c = S1;
		S1 = S2;
		S2 = c;
	}

	//// enter the ROI
	double weight = 0, Ray = 0;
	short GridT = 0, GridS = 0;        // candidate crosspoint index in matlab(0~t_length)
	double GridT_s = 0, GridS_t = 0;    // candidate crosspoint index in matlab(0~256)
	short DetectPoint_t = 0, DetectPoint_s = 0;   // current pixel index in matlab pixel index in matlab(0~255)
	long Pointid = 0;
	double TCross = TStart / resolution_1, SCross = SStart / resolution_2;     // current crosspoint index in matlab(0~256)
	
	//while (((XCross * dev_resolution[1]) >= Range[0]) && ((XCross * dev_resolution[1]) <= Range[2]) 
	//	&& ((YCross * dev_resolution[0]) >= Range[1]) && ((YCross * dev_resolution[0]) <= Range[3]))

	for (short i = 0;i<(t_length + s_length - 1);i++)
	{
		// judge whether XCross/YCross is integer
		if (abs(TCross - round(TCross)) < EPS)
		{
			GridT = round(TCross) + t_signal;
		}
		else
		{
			GridT = floor(TCross) + flag1to1or_1to0(t_signal);
		}
		GridT_s = (source_s + (GridT * resolution_1 - source_t) * T2S) / resolution_2;

		if (abs(SCross - round(SCross)) < EPS)
		{
			GridS = round(SCross) + s_signal;
		}
		else
		{
			GridS = floor(SCross) + flag1to1or_1to0(s_signal);
		}
		GridS_t = (source_t + (GridS * resolution_2 - source_s) * S2T) / resolution_1;

		//judge which crosspoint is the nearest, means the smallest distance
		if (Distancesq(GridT, GridT_s, TCross, SCross) <= Distancesq(GridS_t, GridS, TCross, SCross))
		{
			weight = sqrt(Distancesq(GridT * resolution_1, GridT_s * resolution_2, TCross * resolution_1, SCross * resolution_2));
			DetectPoint_t = floor(MID(GridT, TCross));                 // the midpoint locates the pixel
			DetectPoint_s = floor(MID(GridT_s, SCross));
			
			TCross = GridT;    // update
			SCross = GridT_s;		
		}
		else
		{
			weight = sqrt(Distancesq(GridS_t * resolution_1, GridS * resolution_2, TCross * resolution_1, SCross * resolution_2));
			DetectPoint_t = floor(MID(GridS_t, TCross));                 // the midpoint locates the pixel
			DetectPoint_s = floor(MID(GridS, SCross));
			TCross = GridS_t;    // update
			SCross = GridS;
		}

		//judge whether the point is in the ROI
		if ((DetectPoint_t >= 0) && (DetectPoint_t < t_length) && (DetectPoint_s >= 0) && (DetectPoint_s < s_length))
		{
			Pointid = DetectPoint_s * t_length + DetectPoint_t;
			Ray += weight * dev_Pic[Pointid];
		}
		else
		{
			//dev_Projection[threadid] = 9000;
			break;
		}

	}

	__syncthreads();
	dev_Projection[threadid] = Ray;
}

// Helser function for using CUDA to add vectors in parallel.
hipError_t ProjectionFan_2D(const float *Pic, double *Projection, const double *BetaScanRange, const double *Pdomain,
	const int t_length, const int s_length, const double Center_t, const double Center_s, const int LBeta, const int LP,
	const double Distance, const double *resolution)
{
	mexPrintf("Hello GenMatParalell!\n");

	float *dev_Pic = 0;
	double *dev_Pdomain = 0, *dev_BetaScanRange = 0, *dev_Projection = 0, *dev_RandomErr = 0;
	double *dev_resolution = 0;

	int threadcubic_x = MIN(threadX, LP);
	int blockcubic_x = MIN(blockX, LBeta);
	int LPResidual = LP % threadX;
	int LBetaResidual = LBeta % blockX;
	int PTime = LP / threadX;
	int BetaTime = LBeta / blockX;
	int Pstart = 0;
	int Betastart = 0;
	double Beta = 0;

	const dim3 thread_cubic(threadcubic_x, 1, 1);
	const dim3 block_cubic(blockcubic_x, 1, 1);
	dim3 thread_cubic_residual(1, 1, 1);  // initial
	dim3 block_cubic_residual(1, 1, 1);  // initial

	mexPrintf("threadcubic_x: %d blockcubic_x: %d LPResidual: %d LBetaResidual: %d\n",
		threadcubic_x, blockcubic_x, LPResidual, LBetaResidual);

	if (LPResidual != 0)
	{
		thread_cubic_residual.x = LPResidual;
	}
	if (LBetaResidual != 0)
	{
		block_cubic_residual.x = LBetaResidual;
	}

	hipError_t cudaStatus;

	double *RandomErr =new double[LBeta * LP];
	for (int beta = 0; beta < LBeta; beta++)
	{
		for (int P = 0; P < LP; P++)
		{
			RandomErr[beta * LP + P] = 0.01 * (rand()/double(RAND_MAX)*2-1);
		}
	}

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		mexPrintf("hipSetDevice failed!  Do you have a CUDA-capable GPU installed?\n");
		goto Error;
	}

	mexPrintf("Call for GPU space.\n");

	// Allocate GPU buffers for three vectors (two input, one output).
	cudaStatus = hipMalloc((void**)&dev_Pic, t_length * s_length * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_Pic hipMalloc failed!");
		mexPrintf("dev_Pic hipMalloc failed!\n");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_Projection, LBeta * LP * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_Projection hipMalloc failed!");
		mexPrintf("dev_Projection hipMalloc failed!\n");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&dev_RandomErr, LBeta * LP * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_Projection hipMalloc failed!");
		mexPrintf("dev_Projection hipMalloc failed!\n");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_Pdomain, LP * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_t_Range hipMalloc failed!");
		mexPrintf("dev_t_Range hipMalloc failed!\n");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_BetaScanRange, LBeta * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_thetaRange hipMalloc failed!");
		mexPrintf("dev_thetaRange hipMalloc failed!\n");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_resolution, 2 * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_resolution hipMalloc failed!");
		mexPrintf("dev_resolution hipMalloc failed!\n");
		goto Error;
	}

	mexPrintf("Copy data from CPU to GPU.\n");

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_Pic, Pic, t_length * s_length * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "thetaRange hipMemcpy failed!");
		mexPrintf("thetaRange hipMemcpy failed!\n");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_Pdomain, Pdomain, LP * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "t_Range hipMemcpy failed!");
		mexPrintf("t_Range hipMemcpy failed!\n");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_BetaScanRange, BetaScanRange, LBeta * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "thetaRange hipMemcpy failed!");
		mexPrintf("thetaRange hipMemcpy failed!\n");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_resolution, resolution, 2 * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_resolution hipMemcpy failed!");
		mexPrintf("dev_resolution hipMemcpy failed!\n");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_RandomErr, RandomErr, LBeta * LP * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_RandomErr hipMemcpy failed!");
		mexPrintf("dev_RandomErr hipMemcpy failed!\n");
		goto Error;
	}

	mexPrintf("Launch computation projection of each lines.\n");
	
	// Launch a kernel on the GPU with one thread for each element.
	
	for (int numP = 0; numP < PTime; numP++)
	{
		for (int numB = 0; numB < BetaTime; numB++)
		{
			Pstart = numP * threadX;
			Betastart = numB * blockX;
			//mexPrintf("%d %d\n", Pstart, Betastart);
			ProjectionFan << <block_cubic, thread_cubic >> >(dev_Pic, dev_Projection, dev_Pdomain, dev_BetaScanRange,
				Center_t, Center_s, dev_resolution, t_length, s_length, Pstart, Betastart, Distance, LP, LBeta, dev_RandomErr);
			// Check for any errors launching the kernel
			cudaStatus = hipGetLastError();
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "ProjectionFan launch failed: %s\n", hipGetErrorString(cudaStatus));
				mexPrintf("ProjectionFan launch failed %s\n", hipGetErrorString(cudaStatus));
				mexPrintf("Error happens at Pstart: %d Betastart: %d \n", Pstart, Betastart);
				goto Error;
			}
		}		
	}
		
	if (LPResidual != 0)
	{
		Pstart = LP - LPResidual;			
		if (LBetaResidual != 0)
		{
			Betastart = LBeta - LBetaResidual;
			//("%d %d\n", Pstart, Betastart);
			ProjectionFan << <block_cubic_residual, thread_cubic_residual >> >(dev_Pic, dev_Projection, dev_Pdomain, dev_BetaScanRange,
				Center_t, Center_s, dev_resolution, t_length, s_length, Pstart, Betastart, Distance, LP, LBeta, dev_RandomErr);
			// Check for any errors launching the kernel
			cudaStatus = hipGetLastError();
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "ProjectionFan launch failed: %s\n", hipGetErrorString(cudaStatus));
				mexPrintf("ProjectionFan launch failed %s\n", hipGetErrorString(cudaStatus));
				mexPrintf("Error happens at Pstart: %d Betastart: %d \n", Pstart, Betastart);
				goto Error;
			}
		}

		for (int numB = 0; numB < BetaTime; numB++)
		{
			Betastart = numB * blockX;
			//("%d %d\n", Pstart, Betastart);
			ProjectionFan << <block_cubic, thread_cubic_residual >> >(dev_Pic, dev_Projection, dev_Pdomain, dev_BetaScanRange,
				Center_t, Center_s, dev_resolution, t_length, s_length, Pstart, Betastart, Distance, LP, LBeta, dev_RandomErr);
			// Check for any errors launching the kernel
			cudaStatus = hipGetLastError();
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "ProjectionFan launch failed: %s\n", hipGetErrorString(cudaStatus));
				mexPrintf("ProjectionFan launch failed %s\n", hipGetErrorString(cudaStatus));
				mexPrintf("Error happens at Pstart: %d Betastart: %d \n", Pstart, Betastart);
				goto Error;
			}
		}				
	}
	if (LBetaResidual != 0)
	{
		Betastart = LBeta - LBetaResidual;
		for (int numP = 0; numP < PTime; numP++)
		{
			Pstart = numP * threadX;
			//mexPrintf("%d %d\n", Pstart, Betastart);
			ProjectionFan << <block_cubic_residual, thread_cubic >> >(dev_Pic, dev_Projection, dev_Pdomain, dev_BetaScanRange,
				Center_t, Center_s, dev_resolution, t_length, s_length, Pstart, Betastart, Distance, LP, LBeta, dev_RandomErr);
			// Check for any errors launching the kernel
			cudaStatus = hipGetLastError();
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "ProjectionFan launch failed: %s\n", hipGetErrorString(cudaStatus));
				mexPrintf("ProjectionFan launch failed %s\n", hipGetErrorString(cudaStatus));
				mexPrintf("Error happens at Pstart: %d Betastart: %d \n", Pstart, Betastart);
				goto Error;
			}
		}		
	}


	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Projection launch failed: %s\n", hipGetErrorString(cudaStatus));
		mexPrintf("Projection launch failed\n");
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		mexPrintf("hipDeviceSynchronize returned error %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(Projection, dev_Projection, LP * LBeta * sizeof(double), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!\n");
		mexPrintf("hipMemcpy failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

Error:
	hipFree(dev_Pdomain);
	hipFree(dev_BetaScanRange);
	hipFree(dev_Projection);
	hipFree(dev_Pic);
	hipFree(dev_resolution);
	hipFree(dev_RandomErr);

	return cudaStatus;
}

#include "hip/hip_runtime.h"
#include "ProjectionCone_3D.h"
#include <cstdlib>
// 2018/04/16
//__device__ const double PI = 3.141592653589793;
__device__ const double EPS = 1e-6;

__global__ void ProjectionCone(const float *dev_Pic, float *dev_Projection, const float *dev_Pdomain, const float *dev_Xigamadomain,
	const double Center_t, const double Center_s, const double Center_z, const double *dev_resolution,
	const int t_length, const int s_length, const int z_length, const float Beta, const int numB, const int Pstart,
	const int Xigamastart, const double Distance, const int LP, const int LXigama, const float *dev_RandomErr)
{  
	const unsigned int Pindex = threadIdx.x + Pstart;
	const unsigned int  Xigamaindex = blockIdx.x + Xigamastart;

	const unsigned long threadid = numB * LXigama * LP + Xigamaindex * LP + Pindex;

	dev_Projection[threadid] = 0;

	float P = dev_Pdomain[Pindex];
	float Xigama = dev_Xigamadomain[Xigamaindex];

	// according to euler equation   
	double source_t = Center_t - Distance * sin(Beta);      // define the source in matlab coordinate
	//double source_t = Center_t - Distance * sin(Beta) + dev_RandomErr[threadid];  
	double source_s = Center_s + Distance * cos(Beta);
	double source_z = Center_z;
	//double source_z = Center_z + dev_RandomErr[threadid];

	double Gama = atan(Xigama / Distance);          // radian angle in s - z coordinate plane
	double Distance_shift = Distance / cos(Gama);    // length of DO'

	double Theta = atan(P / Distance_shift);        // radian angle in s'-t coordinate plane 

	double Smax = 2 * Distance;

	// define end detect point in matlab coordinate, Note that : 0 is the start
	double DetectPoint_tend = Center_t + Smax * sin(Theta) * cos(Beta) - (Distance - Smax * cos(Theta) * cos(Gama)) * sin(Beta);
	double DetectPoint_send = Center_s + Smax * sin(Theta) * sin(Beta) + (Distance - Smax * cos(Theta) * cos(Gama)) * cos(Beta);
	double DetectPoint_zend = Center_z + Smax * cos(Theta) * sin(Gama);

	// to determine the range of t
	short t_signal = 0;

	if (DetectPoint_tend >= source_t)
		t_signal = 1;
	else
		t_signal = -1;

	// to determine the range of s
	short s_signal = 0;

	if (DetectPoint_send >= source_s)
		s_signal = 1;
	else
		s_signal = -1;

	// to determine the range of z
	short z_signal = 0;

	if (DetectPoint_zend >= source_z)
		z_signal = 1;
	else
		z_signal = -1;

	// actual Size
	double tlow = 0, thigh = t_length*dev_resolution[0], slow = 0, shigh = s_length*dev_resolution[1],
		zlow = 0, zhigh = z_length*dev_resolution[2];

	//compute the first and last point in the ROI
	// using DetectPoint_end set up projection equation
	double tlow_s = source_s + (tlow - source_t) * (DetectPoint_send - source_s) / (DetectPoint_tend - source_t);
	double tlow_z = source_z + (tlow - source_t) * (DetectPoint_zend - source_z) / (DetectPoint_tend - source_t);
	double thigh_s = source_s + (thigh - source_t) * (DetectPoint_send - source_s) / (DetectPoint_tend - source_t);
	double thigh_z = source_z + (thigh - source_t) * (DetectPoint_zend - source_z) / (DetectPoint_tend - source_t);

	double slow_t = source_t + (slow - source_s) * (DetectPoint_tend - source_t) / (DetectPoint_send - source_s);
	double slow_z = source_z + (slow - source_s) * (DetectPoint_zend - source_z) / (DetectPoint_send - source_s);
	double shigh_t = source_t + (shigh - source_s) * (DetectPoint_tend - source_t) / (DetectPoint_send - source_s);
	double shigh_z = source_z + (shigh - source_s) * (DetectPoint_zend - source_z) / (DetectPoint_send - source_s);

	double zlow_t = source_t + (zlow - source_z) * (DetectPoint_tend - source_t) / (DetectPoint_zend - source_z);
	double zlow_s = source_s + (zlow - source_z) * (DetectPoint_send - source_s) / (DetectPoint_zend - source_z);
	double zhigh_t = source_t + (zhigh - source_z) * (DetectPoint_tend - source_t) / (DetectPoint_zend - source_z);
	double zhigh_s = source_s + (zhigh - source_z) * (DetectPoint_send - source_s) / (DetectPoint_zend - source_z);

	//double *Range = new double [6];   //  XYXY small-big(number)
	double T1 = 0, S1 = 0, Z1 = 0, T2 = 0, S2 = 0, Z2 = 0;

	if (tlow_s >= 0 && tlow_s <= shigh && tlow_z >= 0 && tlow_z <= zhigh)
	{
		T1 = tlow; S1 = tlow_s; Z1 = tlow_z;
		if (thigh_s >= 0 && thigh_s <= shigh && thigh_z >= 0 && thigh_z <= zhigh)
		{
			T2 = thigh; S2 = thigh_s; Z2 = thigh_z;
		}
		else if (slow_t >= 0 && slow_t <= thigh && slow_z >= 0 && slow_z <= zhigh)
		{
			T2 = slow_t; S2 = slow; Z2 = slow_z;
		}
		else if (shigh_t >= 0 && shigh_t <= thigh && shigh_z >= 0 && shigh_z <= zhigh)
		{
			T2 = shigh_t; S2 = shigh; Z2 = shigh_z;
		}
		else if (zlow_t >= 0 && zlow_t <= thigh && zlow_s>= 0 && zlow_s <= shigh)
		{
			T2 = zlow_t; S2 = zlow_s; Z2 = zlow;
		}
		else if (zhigh_t >= 0 && zhigh_t <= thigh && zhigh_s >= 0 && zhigh_s <= shigh)
		{
			T2 = zhigh_t; S2 = zhigh_s; Z2 = zhigh;
		}
	}
	else if (thigh_s >= 0 && thigh_s <= shigh && thigh_z >= 0 && thigh_z <= zhigh)
	{
		T1 = thigh; S1 = thigh_s; Z1 = thigh_z;
		if (slow_t >= 0 && slow_t <= thigh && slow_z >= 0 && slow_z <= zhigh)
		{
			T2 = slow_t; S2 = slow; Z2 = slow_z;
		}
		else if (shigh_t >= 0 && shigh_t <= thigh && shigh_z >= 0 && shigh_z <= zhigh)
		{
			T2 = shigh_t; S2 = shigh; Z2 = shigh_z;
		}
		else if (zlow_t >= 0 && zlow_t <= thigh && zlow_s >= 0 && zlow_s <= shigh)
		{
			T2 = zlow_t; S2 = zlow_s; Z2 = zlow;
		}
		else if (zhigh_t >= 0 && zhigh_t <= thigh && zhigh_s >= 0 && zhigh_s <= shigh)
		{
			T2 = zhigh_t; S2 = zhigh_s; Z2 = zhigh;
		}
	}
	else if (slow_t >= 0 && slow_t <= thigh && slow_z >= 0 && slow_z <= zhigh)
	{
		T1 = slow_t; S1 = slow; Z1 = slow_z;
		if (shigh_t >= 0 && shigh_t <= thigh && shigh_z >= 0 && shigh_z <= zhigh)
		{
			T2 = shigh_t; S2 = shigh; Z2 = shigh_z;
		}
		else if (zlow_t >= 0 && zlow_t <= thigh && zlow_s >= 0 && zlow_s <= shigh)
		{
			T2 = zlow_t; S2 = zlow_s; Z2 = zlow;
		}
		else if (zhigh_t >= 0 && zhigh_t <= thigh && zhigh_s >= 0 && zhigh_s <= shigh)
		{
			T2 = zhigh_t; S2 = zhigh_s; Z2 = zhigh;
		}
	}
	else if (shigh_t >= 0 && shigh_t <= thigh && shigh_z >= 0 && shigh_z <= zhigh)
	{
		T1 = shigh_t; S1 = shigh; Z1 = shigh_z;
		if (zlow_t >= 0 && zlow_t <= thigh && zlow_s >= 0 && zlow_s <= shigh)
		{
			T2 = zlow_t; S2 = zlow_s; Z2 = zlow;
		}
		else if (zhigh_t >= 0 && zhigh_t <= thigh && zhigh_s >= 0 && zhigh_s <= shigh)
		{
			T2 = zhigh_t; S2 = zhigh_s; Z2 = zhigh;
		}
	}
	else if (zlow_t >= 0 && zlow_t <= thigh && zlow_s >= 0 && zlow_s <= shigh)
	{
		T1 = zlow_t; S1 = zlow_s; Z1 = zlow;
		if (zhigh_t >= 0 && zhigh_t <= thigh && zhigh_s >= 0 && zhigh_s <= shigh)
		{
			T2 = zhigh_t; S2 = zhigh_s; Z2 = zhigh;
		}
	}
	else
	{
		//dev_Projection[threadid] = threadid;
		return;
	}

	// set the start point
	double TStart = 0, SStart = 0, ZStart = 0;
	if (Distancesq(T1, S1, Z1, source_t, source_s, source_z) >= Distancesq(T2, S2, Z2, source_t, source_s, source_z))
	{
		TStart = T2;
		SStart = S2;
		ZStart = Z2;
	}
	else
	{
		TStart = T1;
		SStart = S1;
		ZStart = Z1;
	}

	// adjust the order

	if (T2 < T1)
	{
		double c = T1;
		T1 = T2;
		T2 = c;
	}
	if (S2 < S1)
	{
		double c = S1;
		S1 = S2;
		S2 = c;
	}
	if (Z2 < Z1)
	{
		double c = Z1;
		Z1 = Z2;
		Z1 = c;
	}

	//// enter the ROI
	double weight = 0, Ray = 0;
	int GridT = 0, GridS = 0, GridZ = 0;        // candidate crosspoint index in matlab(0~t_length)
	double GridT_s = 0, GridT_z = 0,
		GridS_t = 0, GridS_z = 0,
		GridZ_t = 0, GridZ_s = 0;    // candidate crosspoint index in matlab(0~256)
	int DetectPoint_t = 0, DetectPoint_s = 0, DetectPoint_z = 0;   // current pixel index in matlab pixel index in matlab(0~255)
	long Pointid = 0;
	double TCross = TStart / dev_resolution[0], SCross = SStart / dev_resolution[1],
		ZCross = ZStart / dev_resolution[2];     // current crosspoint index in matlab(0~256)
	
	int i = 0;
	//while (((XCross * dev_resolution[1]) >= Range[0]) && ((XCross * dev_resolution[1]) <= Range[2]) 
	//	&& ((YCross * dev_resolution[0]) >= Range[1]) && ((YCross * dev_resolution[0]) <= Range[3]))
	while (i < (t_length + s_length + z_length -2))
	{
		i++;
		// judge whether XCross/YCross is integer
		if (TCross - (double)((int)TCross) < EPS)
		{
			GridT = TCross + t_signal;
		}
		else
		{
			GridT = floor(TCross) + flag1to1or_1to0(t_signal);
		}
		GridT_s = (source_s + (GridT * dev_resolution[0] - source_t) * (DetectPoint_send - source_s) / (DetectPoint_tend - source_t)) / dev_resolution[1];
		GridT_z = (source_z + (GridT * dev_resolution[0] - source_t) * (DetectPoint_zend - source_z) / (DetectPoint_tend - source_t)) / dev_resolution[2];

		if (SCross - (double)((int)SCross) < EPS)
		{
			GridS = SCross + s_signal;
		}
		else
		{
			GridS = floor(SCross) + flag1to1or_1to0(s_signal);
		}
		GridS_t = (source_t + (GridS * dev_resolution[1] - source_s) * (DetectPoint_tend - source_t) / (DetectPoint_send - source_s)) / dev_resolution[0];
		GridS_z = (source_z + (GridS * dev_resolution[1] - source_s) * (DetectPoint_zend - source_z) / (DetectPoint_send - source_s)) / dev_resolution[2];

		if (ZCross - (double)((int)ZCross) < EPS)
		{
			GridZ = ZCross + z_signal;
		}
		else
		{
			GridZ = floor(ZCross) + flag1to1or_1to0(z_signal);
		}
		GridZ_t = (source_t + (GridZ * dev_resolution[2] - source_z) * (DetectPoint_tend - source_t) / (DetectPoint_zend - source_z)) / dev_resolution[0];
		GridZ_s = (source_s + (GridZ * dev_resolution[2] - source_z) * (DetectPoint_send - source_s) / (DetectPoint_zend - source_z)) / dev_resolution[1];

		//judge which crosspoint is the nearest, means the smallest distance
		if (Distancesq(GridT, GridT_s, GridT_z, TCross, SCross, ZCross) <= Distancesq(GridS_t, GridS, GridS_z, TCross, SCross, ZCross))
		{
			if (Distancesq(GridZ_t, GridZ_s, GridZ, TCross, SCross, ZCross) <= Distancesq(GridT, GridT_s, GridT_z, TCross, SCross, ZCross))
			{
				weight = sqrt(Distancesq(GridZ_t * dev_resolution[0], GridZ_s * dev_resolution[1],
					GridZ * dev_resolution[2], TCross * dev_resolution[0], SCross * dev_resolution[1], 
					ZCross * dev_resolution[2]));
				DetectPoint_t = floor(MID(GridZ_t, TCross));                 // the midpoint locates the pixel
				DetectPoint_s = floor(MID(GridZ_s, SCross));
				DetectPoint_z = floor(MID(GridZ, ZCross));
				TCross = GridZ_t;    // update
				SCross = GridZ_s;
				ZCross = GridZ;
			}
			else
			{
				weight = sqrt(Distancesq(GridT * dev_resolution[0], GridT_s * dev_resolution[1],
					GridT_z * dev_resolution[2], TCross * dev_resolution[0], SCross * dev_resolution[1],
					ZCross * dev_resolution[2]));
				DetectPoint_t = floor(MID(GridT, TCross));                 // the midpoint locates the pixel
				DetectPoint_s = floor(MID(GridT_s, SCross));
				DetectPoint_z = floor(MID(GridT_z, ZCross));
				TCross = GridT;    // update
				SCross = GridT_s;
				ZCross = GridT_z;
			}			
		}
		else
		{
			if (Distancesq(GridZ_t, GridZ_s, GridZ, TCross, SCross, ZCross) <= Distancesq(GridS_t, GridS, GridS_z, TCross, SCross, ZCross))
			{
				weight = sqrt(Distancesq(GridZ_t * dev_resolution[0], GridZ_s * dev_resolution[1],
					GridZ * dev_resolution[2], TCross * dev_resolution[0], SCross * dev_resolution[1],
					ZCross * dev_resolution[2]));
				DetectPoint_t = floor(MID(GridZ_t, TCross));                 // the midpoint locates the pixel
				DetectPoint_s = floor(MID(GridZ_s, SCross));
				DetectPoint_z = floor(MID(GridZ, ZCross));
				TCross = GridZ_t;    // update
				SCross = GridZ_s;
				ZCross = GridZ;
			}
			else
			{
				weight = sqrt(Distancesq(GridS_t * dev_resolution[0], GridS * dev_resolution[1],
					GridS_z * dev_resolution[2], TCross * dev_resolution[0], SCross * dev_resolution[1],
					ZCross * dev_resolution[2]));
				DetectPoint_t = floor(MID(GridS_t, TCross));                 // the midpoint locates the pixel
				DetectPoint_s = floor(MID(GridS, SCross));
				DetectPoint_z = floor(MID(GridS_z, ZCross));
				TCross = GridS_t;    // update
				SCross = GridS;
				ZCross = GridS_z;
			}
		}

		//judge whether the point is in the ROI
		if ((DetectPoint_t >= 0) && (DetectPoint_t <= (t_length - 1)) && (DetectPoint_s >= 0) && (DetectPoint_s <= (s_length - 1))
			&& (DetectPoint_z >= 0) && (DetectPoint_z <= (z_length - 1)))
		{
			Pointid = DetectPoint_z * t_length * s_length + DetectPoint_s * t_length + DetectPoint_t;
			Ray += weight * dev_Pic[Pointid];
		}
		else
		{
			//dev_Projection[threadid] = 9000;
			break;
		}

	}

	//__syncthreads();
	dev_Projection[threadid] = Ray;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t ProjectionCone_3D(const float *Pic, float *Projection, const float *BetaScanRange, const float *Pdomain,
	const float *Xigamadomain, const int t_length, const int s_length, const int z_length, const double Center_t,
	const double Center_s, const double Center_z, const int LBeta, const int LP, const int LXigama, const double Distance,
	const double *resolution)
{
	mexPrintf("Hello GenMatParalell!\n");

	float *dev_Pic = 0, *dev_Pdomain = 0, *dev_Xigamadomain = 0, *dev_Projection = 0, *dev_RandomErr = 0;
	double *dev_resolution = 0;

	int threadcubic_x = MIN(threadX, LP);
	int blockcubic_x = MIN(blockX, LXigama);
	int LPResidual = LP % threadX;
	int LXigamaResidual = LXigama % blockX;
	int PTime = LP / threadX;
	int XigamaTime = LXigama / blockX;
	int Pstart = 0;
	int Xigamastart = 0;
	float Beta = 0;
	const dim3 thread_cubic(threadcubic_x, 1, 1);
	const dim3 block_cubic(blockcubic_x, 1, 1);
	dim3 thread_cubic_residual(1, 1, 1);  // initial
	dim3 block_cubic_residual(1, 1, 1);  // initial

	mexPrintf("%d %d %d %d\n", threadcubic_x, blockcubic_x, LPResidual, LXigamaResidual);

	if (LPResidual != 0)
	{
		thread_cubic_residual.x = LPResidual;
	}
	if (LXigamaResidual != 0)
	{
		block_cubic_residual.x = LXigamaResidual;
	}

	hipError_t cudaStatus;

	float *RandomErr =new float[LBeta * LP * LXigama];
	for (int beta = 0; beta < LBeta; beta++)
	{
		for (int P = 0; P < LP; P++)
		{
			for (int Xigama = 0; Xigama < LXigama; Xigama++)
			{
				RandomErr[beta * LXigama * LP + Xigama * LP + P] = 0.01 * (rand()/float(RAND_MAX)*2-1);
			}
		}
	}
	



	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		mexPrintf("hipSetDevice failed!  Do you have a CUDA-capable GPU installed?\n");
		goto Error;
	}

	mexPrintf("Call for GPU space.\n");

	// Allocate GPU buffers for three vectors (two input, one output).
	cudaStatus = hipMalloc((void**)&dev_Pic, t_length * s_length * z_length * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_Pic hipMalloc failed!");
		mexPrintf("dev_Pic hipMalloc failed!\n");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_Projection, LBeta * LP * LXigama * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_Projection hipMalloc failed!");
		mexPrintf("dev_Projection hipMalloc failed!\n");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&dev_RandomErr, LBeta * LP * LXigama * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_Projection hipMalloc failed!");
		mexPrintf("dev_Projection hipMalloc failed!\n");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_Pdomain, LP * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_t_Range hipMalloc failed!");
		mexPrintf("dev_t_Range hipMalloc failed!\n");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_Xigamadomain, LXigama * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_resolution hipMalloc failed!");
		mexPrintf("dev_resolution hipMalloc failed!\n");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_resolution, 3 * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_resolution hipMalloc failed!");
		mexPrintf("dev_resolution hipMalloc failed!\n");
		goto Error;
	}

	mexPrintf("Copy data from CPU to GPU.\n");

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_Pic, Pic, t_length * s_length * z_length * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "thetaRange hipMemcpy failed!");
		mexPrintf("thetaRange hipMemcpy failed!\n");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_Pdomain, Pdomain, LP * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "t_Range hipMemcpy failed!");
		mexPrintf("t_Range hipMemcpy failed!\n");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_Xigamadomain, Xigamadomain, LXigama * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "t_Range hipMemcpy failed!");
		mexPrintf("t_Range hipMemcpy failed!\n");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_resolution, resolution, 3 * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_resolution hipMemcpy failed!");
		mexPrintf("dev_resolution hipMemcpy failed!\n");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_RandomErr, RandomErr, LBeta * LP * LXigama * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_RandomErr hipMemcpy failed!");
		mexPrintf("dev_RandomErr hipMemcpy failed!\n");
		goto Error;
	}
	dev
	mexPrintf("Launch computation projection of each lines.\n");
	
	// Launch a kernel on the GPU with one thread for each element.
	for (int numB = 0; numB < LBeta; numB++)
	{
		Beta = BetaScanRange[numB];
		for (int numP = 0; numP < PTime; numP++)
		{
			for (int numX = 0; numX < XigamaTime; numX++)
			{
				Pstart = numP * threadX;
				Xigamastart = numX * blockX;
				//mexPrintf("%d %d\n", Pstart, Xigamastart);
				ProjectionCone << <block_cubic, thread_cubic >> >(dev_Pic, dev_Projection, dev_Pdomain, dev_Xigamadomain,
					Center_t, Center_s, Center_z, dev_resolution, t_length, s_length, z_length, Beta, numB, Pstart, Xigamastart,
					Distance, LP, LXigama, dev_RandomErr);
			}		
		}
		
		if (LPResidual != 0)
		{
			Pstart = LP - LPResidual;			
			if (LXigamaResidual != 0)
			{
				Xigamastart = LXigama - LXigamaResidual;
				//("%d %d\n", Pstart, Xigamastart);
				ProjectionCone << <block_cubic_residual, thread_cubic_residual >> >(dev_Pic, dev_Projection, dev_Pdomain, dev_Xigamadomain,
					Center_t, Center_s, Center_z, dev_resolution, t_length, s_length, z_length, Beta, numB, Pstart, Xigamastart,
					Distance, LP, LXigama, dev_RandomErr);
			}

			for (int numX = 0; numX < XigamaTime; numX++)
			{
				Xigamastart = numX * blockX;
				//("%d %d\n", Pstart, Xigamastart);
				ProjectionCone << <block_cubic, thread_cubic_residual >> >(dev_Pic, dev_Projection, dev_Pdomain, dev_Xigamadomain,
					Center_t, Center_s, Center_z, dev_resolution, t_length, s_length, z_length, Beta, numB, Pstart, Xigamastart,
					Distance, LP, LXigama, dev_RandomErr);
			}				
		}
		if (LXigamaResidual != 0)
		{
			Xigamastart = LXigama - LXigamaResidual;
			for (int numP = 0; numP < PTime; numP++)
			{
				Pstart = numP * threadX;
				//mexPrintf("%d %d\n", Pstart, Xigamastart);
				ProjectionCone << <block_cubic_residual, thread_cubic >> >(dev_Pic, dev_Projection, dev_Pdomain, dev_Xigamadomain,
					Center_t, Center_s, Center_z, dev_resolution, t_length, s_length, z_length, Beta, numB, Pstart, Xigamastart,
					Distance, LP, LXigama, dev_RandomErr);
			}		
		}
	}

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Projection launch failed: %s\n", hipGetErrorString(cudaStatus));
		mexPrintf("Projection launch failed\n");
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		mexPrintf("hipDeviceSynchronize returned error %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(Projection, dev_Projection, LBeta * LP * LXigama * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!\n");
		mexPrintf("hipMemcpy failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

Error:
	hipFree(dev_Pdomain);
	hipFree(dev_Xigamadomain);
	hipFree(dev_Projection);
	hipFree(dev_Pic);
	hipFree(dev_resolution);

	return cudaStatus;
}

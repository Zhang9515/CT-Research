#include "hip/hip_runtime.h"
#include "FDK.h"
// 2018/04/20 apply GPU acceleration
#define pow2(x) (1.0*(x)*(x))
__device__ const double PI = 3.141592653589793;

__global__ void GFunction(float *dev_G, const float PInt, const int LP)
{

	const unsigned int Pindex = threadIdx.x;

	// S_L filter 
	if (Pindex == 0)
		dev_G[LP - 1] = 1.0 / pow2(PI*PInt);
	else
	{
		dev_G[(LP - 1) - Pindex] = -1.0 / pow2(PInt * PI) / (4 * pow2(Pindex) - 1);
		dev_G[(LP - 1) + Pindex] = -1.0 / pow2(PInt * PI) / (4 * pow2(Pindex) - 1);
	}
}

__global__ void PreWeightFiltration(float *dev_Rcov, float *dev_R, const float *dev_G, const float *dev_Pdomain,
	const float *dev_Xigamadomain, const double Distance, const float PInt, const int LP, const int LXigama, const int Pstart,
	const int Xigamastart, const int Betaindex)
{

	//const unsigned int Gamaindex = blockIdx.x * blockDim.x + threadIdx.x;
	//const unsigned int Xigamaindex = blockIdx.y * blockDim.y + threadIdx.y;
	//const unsigned int Betaindex = blockIdx.z * blockDim.z + threadIdx.z;
	//const unsigned long thread_id = Betaindex * ( gridDim.x * gridDim.y * blockDim.x * blockDim.y ) 
	//	+ Xigamaindex * ( gridDim.x * blockDim.x ) + Gamaindex ;

	const unsigned int Pindex = threadIdx.x + Pstart;
	const unsigned int Xigamaindex = blockIdx.x + Xigamastart;
	const unsigned long thread_id = Betaindex * (LXigama * LP) + Xigamaindex * LP + Pindex;

	float P = dev_Pdomain[Pindex];
	float Xigama = dev_Xigamadomain[Xigamaindex];

	double Proportion = Distance / sqrt(pow2(Distance) + pow2(P) + pow2(Xigama));

	dev_R[thread_id] = dev_R[thread_id] * Proportion;    // directly cover the input

	__syncthreads();
	double Rcovsum = 0;
	// convolution
	for (int g = 0; g < LP; g++)
	{
		//convolution
		Rcovsum += dev_R[Betaindex * (LXigama * LP) + Xigamaindex * LP + g] * dev_G[(LP - 1) + Pindex - g];
	}
	//__syncthreads();
	dev_Rcov[thread_id] = PInt * Rcovsum;
}

__global__ void BackProjection(const float *dev_Rcov, float *dev_Display, const double *dev_Size,
	const int t_length, const int s_length, const int z_length, const float Beta, const double Distance, 
	const float *dev_Pdomain, const float *dev_Xigamadomain, const float PInt, const float XigamaInt, 
	const float BetaScanInt, const float minP, const float maxP, const float minXigama, const float maxXigama, 
	const int betaIndex, const int LP, const int LXigama)
{
	const unsigned int Tindex = threadIdx.x;
	const unsigned int Sindex = blockIdx.x;
	const unsigned int Zindex = blockIdx.y;
	const unsigned long thread_id = Zindex * (gridDim.x * blockDim.x)
		+ Sindex * blockDim.x + Tindex;
	// initialize

	const double Resolution_t = 1.0 * dev_Size[0] / t_length;
	const double Resolution_s = 1.0 * dev_Size[1] / s_length;
	const double Resolution_z = 1.0 * dev_Size[2] / z_length;

	// rotation center
	double Center_t = dev_Size[0] / 2;
	double Center_s = dev_Size[1] / 2;
	double Center_z = dev_Size[2] / 2;

	// this is a little different from code on MATLAB
	double image_t = (Tindex + 0.5) * Resolution_t - Center_t;  float image_s = (Sindex + 0.5) * Resolution_s - Center_s; float image_z = (Zindex + 0.5) * Resolution_z - Center_z;           // image pixel in ground coordinate

	// rotate in ground coordinate
	double dect_t = image_t * cos(Beta) + image_s * sin(Beta);
	double dect_s = -image_t * sin(Beta) + image_s * cos(Beta);
	double dect_z = image_z;

	// define the projection position on the detector
	double LengthRatio = Distance / (Distance - dect_s);
	double Xigama = dect_z * LengthRatio;
	double P = dect_t * LengthRatio;

	unsigned int XigamaN1index = 0, XigamaN2index = 0, PN1index = 0, PN2index = 0;
	double P_domain1 = 0, P_domain2 = 0, Xigama_domain1 = 0, Xigama_domain2 = 0;
	double Xig1 = 0, Xig2 = 0, P1 = 0, P2 = 0;
	double Display_pBeta = 0;

	if ((P >= minP) && (P < maxP) && (Xigama >= minXigama) && (Xigama < maxXigama))
	{
		XigamaN1index = floor(fabs(Xigama - dev_Xigamadomain[0]) / XigamaInt);
		XigamaN2index = XigamaN1index + 1;
		PN1index = floor(fabs(P - dev_Pdomain[0]) / PInt);
		PN2index = PN1index + 1;

		P_domain1 = dev_Pdomain[PN1index]; P_domain2 = dev_Pdomain[PN2index];
		Xigama_domain1 = dev_Xigamadomain[XigamaN1index]; Xigama_domain2 = dev_Xigamadomain[XigamaN2index];

		//bilinear interpolation
		Xig1 = fabs(Xigama - Xigama_domain1); Xig2 = fabs(Xigama_domain2 - Xigama);
		P1 = fabs(P - P_domain1); P2 = fabs(P_domain2 - P);

		double Weight = sqrt(1 + 0 * pow2(dect_z) / (pow2(Distance + dect_s) + pow2(dect_t)));
		//Display_pBeta = 1;
		Display_pBeta = (Xig2 * P2 * dev_Rcov[betaIndex * LP * LXigama + XigamaN1index * LP + PN1index]
			+ Xig1 * P2 * dev_Rcov[betaIndex * LP * LXigama + XigamaN2index * LP + PN1index] + Xig2 * P1 * dev_Rcov[betaIndex * LP * LXigama + XigamaN1index * LP + PN2index]
			+ Xig1 * P1 * dev_Rcov[betaIndex * LP * LXigama + XigamaN2index * LP  + PN2index]) / (PInt * XigamaInt) * pow2(LengthRatio) * BetaScanInt * Weight;
	}

	dev_Display[thread_id] += Display_pBeta;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t FDKpro(float *Display, const float *R, const float *Xigamadomain, const float *Pdomain,
	const float *BetaScanRange, const double Distance, const int LBeta, const int LP, const int LXigama,
	const double *Size, const int t_length, const int s_length, const int z_length)
{
	mexPrintf("Hello FDK!\n");
	float *dev_R = 0, *dev_Rcov = 0;
	float *dev_G = 0;
	float *dev_BetaScanRange = 0, *dev_Pdomain = 0, *dev_Xigamadomain = 0;
	double *dev_Size = 0;

	float *dev_Display = 0;
	float PInt = fabs(Pdomain[1] - Pdomain[0]);
	float XigamaInt = fabs(Xigamadomain[1] - Xigamadomain[0]);
	float BetaScanInt = fabs(BetaScanRange[1] - BetaScanRange[0]);

	float maxP = MAX(Pdomain[0], Pdomain[LP - 1]);
	float minP = MIN(Pdomain[0], Pdomain[LP - 1]);
	float maxXigama = MAX(Xigamadomain[0], Xigamadomain[LXigama - 1]);
	float minXigama = MIN(Xigamadomain[0], Xigamadomain[LXigama - 1]);
	//mexPrintf("%lf %lf %lf %lf \n", maxGama, minGama, maxXigama, minXigama);

	const long LDisplay = t_length * s_length * z_length;
	const long LR = LP * LXigama * LBeta;
	const int LFilter = 2 * LP - 1;

	int thread_cubic_x = MIN(threadX, LP);
	int block_cubic_x = MIN(blockX, LXigama);

	const dim3 thread_cubic(thread_cubic_x, 1, 1);
	const dim3 block_cubic(block_cubic_x, 1, 1);

	dim3 thread_cubic_residual(1, 1, 1);  // initial
	dim3 block_cubic_residual(1, 1, 1);  // initial

	int LPResidual = LP % threadX;
	int LXigamaResidual = LXigama % blockX;
	int PTime = LP / threadX;
	int XigamaTime = LXigama / blockX;
	int Pstart = 0;
	int Xigamastart = 0;
	float Beta = 0;

	if (LPResidual != 0)
	{
		thread_cubic_residual.x = LPResidual;
	}
	if (LXigamaResidual != 0)
	{
		block_cubic_residual.x = LXigamaResidual;
	}

	const dim3 thread_cubic_Bp(t_length, 1, 1);
	const dim3 block_cubic_Bp(s_length, z_length, 1);

	hipError_t cudaStatus;

	mexPrintf("start cuda\n");

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?\n");
		mexPrintf("hipSetDevice failed!  Do you have a CUDA-capable GPU installed? %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	mexPrintf("call for space in GPU\n");

	// Allocate GPU buffers for three vectors (two input, one output).

	cudaStatus = hipMalloc((void**)&dev_R, LR * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_R hipMalloc failed!\n");
		mexPrintf("dev_R hipMalloc failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error1;
	}

	cudaStatus = hipMalloc((void**)&dev_Pdomain, LP * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_Pdomain hipMalloc failed!\n");
		mexPrintf("dev_Pdomain hipMalloc failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_BetaScanRange, LBeta * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_BetaScanRange hipMalloc failed!\n");
		mexPrintf("dev_BetaScanRange hipMalloc failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_G, LFilter * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_G hipMalloc failed!\n");
		mexPrintf("dev_G hipMalloc failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error1;
	}

	cudaStatus = hipMalloc((void**)&dev_Rcov, LR * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_BetaScanRange hipMalloc failed!\n");
		mexPrintf("dev_BetaScanRange hipMalloc failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_Xigamadomain, LXigama * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_Xigamadomain hipMalloc failed!\n");
		mexPrintf("dev_Xigamadomain hipMalloc failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	//mexPrintf("copy data in CPU to GPU\n");

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_R, R, LR * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy R failed!\n");
		mexPrintf("hipMemcpy R failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error1;
	}

	cudaStatus = hipMemcpy(dev_Pdomain, Pdomain, LP * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy Gamadomain failed!\n");
		mexPrintf("hipMemcpy Gamadomain failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_BetaScanRange, BetaScanRange, LBeta * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy BetaScanRange failed!\n");
		mexPrintf("hipMemcpy v failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_Xigamadomain, Xigamadomain, LXigama * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy Xigamadomain failed!\n");
		mexPrintf("hipMemcpy Xigamadomain failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	mexPrintf("start parallel computation\n");
	mexPrintf("Filter\n");
	// Generate Filter
	GFunction << <1, LP >> >(dev_G, PInt, LP);

	mexPrintf("Preweight and filtration\n");
	// Preweight and filtration
	// to be adapted to GPU, we limited the number of threads in each grid (threadX * blockX * LBeta)
	for (int numB = 0; numB < LBeta; numB++)
	{
		Beta = BetaScanRange[numB];
		for (int numP = 0; numP < PTime; numP++)
		{
			for (int numX = 0; numX < XigamaTime; numX++)
			{
				Pstart = numP * threadX;
				Xigamastart = numX * blockX;
				//mexPrintf("%d %d\n", Pstart, Xigamastart);
				PreWeightFiltration << <block_cubic, thread_cubic >> > (dev_Rcov, dev_R, dev_G, dev_Pdomain,
					dev_Xigamadomain, Distance, PInt, LP, LXigama, Pstart, Xigamastart, numB);
			}
		}

		if (LPResidual != 0)
		{
			Pstart = LP - LPResidual;
			if (LXigamaResidual != 0)
			{
				Xigamastart = LXigama - LXigamaResidual;
				//("%d %d\n", Pstart, Xigamastart);
				PreWeightFiltration << <block_cubic_residual, thread_cubic_residual >> > (dev_Rcov, dev_R, dev_G, dev_Pdomain,
					dev_Xigamadomain, Distance, PInt, LP, LXigama, Pstart, Xigamastart, numB);
			}

			for (int numX = 0; numX < XigamaTime; numX++)
			{
				Xigamastart = numX * blockX;
				//("%d %d\n", Pstart, Xigamastart);
				PreWeightFiltration << <block_cubic, thread_cubic_residual >> > (dev_Rcov, dev_R, dev_G, dev_Pdomain,
					dev_Xigamadomain, Distance, PInt, LP, LXigama, Pstart, Xigamastart, numB);
			}
		}
		if (LXigamaResidual != 0)
		{
			Xigamastart = LXigama - LXigamaResidual;
			for (int numP = 0; numP < PTime; numP++)
			{
				Pstart = numP * threadX;
				//mexPrintf("%d %d\n", Pstart, Xigamastart);
				PreWeightFiltration << <block_cubic_residual, thread_cubic >> > (dev_Rcov, dev_R, dev_G, dev_Pdomain,
					dev_Xigamadomain, Distance, PInt, LP, LXigama, Pstart, Xigamastart, numB);
			}
		}
	}
	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "PreWeight and Filtration failed: %s\n", hipGetErrorString(cudaStatus));
		mexPrintf("PreWeight and Filtration failed\n");
		goto Error1;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error1;
	}

Error1:
	hipFree(dev_R);
	hipFree(dev_G);

	mexPrintf("backprojection\n");
	
	cudaStatus = hipMalloc((void**)&dev_Size, 3 * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_Size hipMalloc failed!\n");
		mexPrintf("dev_Size hipMalloc failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_Size, Size, 3 * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy Size failed!\n");
		mexPrintf("hipMemcpy Size failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_Display, LDisplay * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_Display hipMalloc failed!\n");
		mexPrintf("dev_Display hipMalloc failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}
	hipMemset(dev_Display, 0, sizeof(float));

	//Backprojection
	for (int betaIndex = 0; betaIndex < LBeta; betaIndex++)
	{
		BackProjection << <block_cubic_Bp, thread_cubic_Bp >> > (dev_Rcov, dev_Display, dev_Size, t_length, s_length, z_length,
			BetaScanRange[betaIndex], Distance, dev_Pdomain, dev_Xigamadomain, PInt, XigamaInt, BetaScanInt, minP, maxP, 
			minXigama, maxXigama, betaIndex, LP, LXigama);
	}

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "BackProjection launch failed: %s\n", hipGetErrorString(cudaStatus));
		mexPrintf("BackProjection launch failed %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		mexPrintf("hipDeviceSynchronize returned error code %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	 //Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(Display, dev_Display, LDisplay * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!\n");
		mexPrintf("hipMemcpy dev_Display failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

Error:
	hipFree(dev_BetaScanRange);
	hipFree(dev_Pdomain);
	hipFree(dev_Xigamadomain);
	hipFree(dev_Rcov);
	hipFree(dev_Display);
	hipFree(dev_Size);

	mexPrintf("Exit FDK\n");
	return cudaStatus;
}
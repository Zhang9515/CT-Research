#include "hip/hip_runtime.h"
#include "Backprojection3D.h"
// 2018/11/16 apply GPU acceleration

//__device__ const double PI = 3.141592653589793;
__device__ const double EPS = 1e-15;

// thiss
__global__ void BackProjection3D(const float *dev_R, float *dev_Display, const double *dev_Size,
	const int t_length, const int s_length, const int z_length, const float Beta, const double Distance, 
	const float *dev_Pdomain, const float *dev_Xigamadomain, const float PInt, const float XigamaInt, 
	const float BetaScanInt, const float minP, const float maxP, const float minXigama, const float maxXigama, 
	const int betaIndex, const int LP, const int LXigama, const int T_start, const int S_start)
{
	const unsigned int Tindex = T_start + threadIdx.x;
	const unsigned int Sindex = S_start + blockIdx.x;
	//const unsigned int Zindex = blockIdx.y;
	unsigned long thread_id;
	// initialize

	const double Resolution_t = 1.0 * dev_Size[0] / t_length;
	const double Resolution_s = 1.0 * dev_Size[1] / s_length;
	const double Resolution_z = 1.0 * dev_Size[2] / z_length;

	// rotation center
	double Center_t = dev_Size[0] / 2;
	double Center_s = dev_Size[1] / 2;
	double Center_z = dev_Size[2] / 2;

	// this is a little different from code on MATLAB
	// image pixel in ground coordinate
	double image_t = (Tindex + 0.5) * Resolution_t - Center_t;  
	double image_s = (Sindex + 0.5) * Resolution_s - Center_s; 
	double image_z;

	// rotate in ground coordinate
	double dect_t = image_t * cos(Beta) + image_s * sin(Beta);
	double dect_s = -image_t * sin(Beta) + image_s * cos(Beta);
	double dect_z;

	// define the projection position on the detector
	double LengthRatio = Distance / (Distance - dect_s);
	double Xigama;
	double P = dect_t * LengthRatio;

	unsigned short XigamaN1index = 0, XigamaN2index = 0, PN1index = 0, PN2index = 0;
	double P_domain1 = 0, P_domain2 = 0, Xigama_domain1 = 0, Xigama_domain2 = 0;
	double Xig1 = 0, Xig2 = 0, P1 = 0, P2 = 0;
	double Display_pBeta = 0;
	double backweight = 0;
	//double LengthinROI = 0;

	//	 according to euler equation   
	// define the source in matlab coordinate
	double source_t = Center_t - Distance * sin(Beta), source_s = Center_s + Distance * cos(Beta), source_z;
	// in matlab coordinate
	//  assume the projection line go through the center of the current pixel 
	double DetectPoint_tend = image_t + Center_t, DetectPoint_send = image_s + Center_s, DetectPoint_zend;
	//	 actual Size
	double tlow = 0, thigh = t_length * Resolution_t, slow = 0, shigh = s_length * Resolution_s,
		zlow = 0, zhigh = z_length * Resolution_z;
	double tlow_s, tlow_z, slow_t, slow_z, zlow_t, zlow_s/*, thigh_s, thigh_z, shigh_t, shigh_z, zhigh_t, zhigh_s*/;

	double T1 = 0, S1 = 0, Z1 = 0/*, T2 = 0, S2 = 0, Z2 = 0*/;
	double LengthinPixel;

	for (short Zindex = 0; Zindex < z_length; Zindex++)
	{
		image_z = (Zindex + 0.5) * Resolution_z - Center_z;
		dect_z = image_z;
		Xigama = dect_z * LengthRatio;
		if ((P >= minP) && (P < maxP) && (Xigama >= minXigama) && (Xigama < maxXigama))
		{
			XigamaN1index = floor(fabs(Xigama - dev_Xigamadomain[0]) / XigamaInt);
			XigamaN2index = XigamaN1index + 1;
			PN1index = floor(fabs(P - dev_Pdomain[0]) / PInt);
			PN2index = PN1index + 1;

			P_domain1 = dev_Pdomain[PN1index]; P_domain2 = dev_Pdomain[PN2index];
			Xigama_domain1 = dev_Xigamadomain[XigamaN1index]; Xigama_domain2 = dev_Xigamadomain[XigamaN2index];

			//bilinear interpolation
			Xig1 = fabs(Xigama - Xigama_domain1); Xig2 = fabs(Xigama_domain2 - Xigama);
			P1 = fabs(P - P_domain1); P2 = fabs(P_domain2 - P);

			Display_pBeta = (Xig2 * P2 * dev_R[betaIndex * LP * LXigama + XigamaN1index * LP + PN1index]
				+ Xig1 * P2 * dev_R[betaIndex * LP * LXigama + XigamaN2index * LP + PN1index] + Xig2 * P1 * dev_R[betaIndex * LP * LXigama + XigamaN1index * LP + PN2index]
				+ Xig1 * P1 * dev_R[betaIndex * LP * LXigama + XigamaN2index * LP + PN2index]) / (PInt * XigamaInt);

			//	 the way to compute backweight is to get the cross length in the specific pixel and the whole ROI

			//	 according to euler equation   
			// define the source in matlab coordinate
			source_z = Center_z;

			// in matlab coordinate
			//  assume the projection line go through the center of the current pixel 
			DetectPoint_zend = image_z + Center_z;

			//	first compute length in whole ROI

			//	compute the first and last point in the ROI
			//	 using DetectPoint_end set up projection equation
			//tlow_s = source_s + (tlow - source_t) * (DetectPoint_send - source_s) / (DetectPoint_tend - source_t + EPS);
			//tlow_z = source_z + (tlow - source_t) * (DetectPoint_zend - source_z) / (DetectPoint_tend - source_t + EPS);
			//thigh_s = source_s + (thigh - source_t) * (DetectPoint_send - source_s) / (DetectPoint_tend - source_t);
			//thigh_z = source_z + (thigh - source_t) * (DetectPoint_zend - source_z) / (DetectPoint_tend - source_t);

			//slow_t = source_t + (slow - source_s) * (DetectPoint_tend - source_t) / (DetectPoint_send - source_s + EPS);
			//slow_z = source_z + (slow - source_s) * (DetectPoint_zend - source_z) / (DetectPoint_send - source_s + EPS);
			//shigh_t = source_t + (shigh - source_s) * (DetectPoint_tend - source_t) / (DetectPoint_send - source_s);
			//shigh_z = source_z + (shigh - source_s) * (DetectPoint_zend - source_z) / (DetectPoint_send - source_s);

			//zlow_t = source_t + (zlow - source_z) * (DetectPoint_tend - source_t) / (DetectPoint_zend - source_z + EPS);
			//zlow_s = source_s + (zlow - source_z) * (DetectPoint_send - source_s) / (DetectPoint_zend - source_z + EPS);
			//zhigh_t = source_t + (zhigh - source_z) * (DetectPoint_tend - source_t) / (DetectPoint_zend - source_z);
			//zhigh_s = source_s + (zhigh - source_z) * (DetectPoint_send - source_s) / (DetectPoint_zend - source_z);

			//	double *Range = new double [6];   //  XYXY small-big(number)

			//if (tlow_s >= 0 && tlow_s <= shigh && tlow_z >= 0 && tlow_z <= zhigh)
			//{
			//	T1 = tlow; S1 = tlow_s; Z1 = tlow_z;
			//	if (thigh_s >= 0 && thigh_s <= shigh && thigh_s != S1
			//		&& thigh_z >= 0 && thigh_z <= zhigh && thigh_z != Z1)
			//	{
			//		T2 = thigh; S2 = thigh_s; Z2 = thigh_z;
			//	}
			//	else if (slow_t >= 0 && slow_t <= thigh && slow_t != T1
			//		&& slow_z >= 0 && slow_z <= zhigh && slow_z != Z1)
			//	{
			//		T2 = slow_t; S2 = slow; Z2 = slow_z;
			//	}
			//	else if (shigh_t >= 0 && shigh_t <= thigh && shigh_t != T1
			//		&& shigh_z >= 0 && shigh_z <= zhigh && shigh_z != Z1)
			//	{
			//		T2 = shigh_t; S2 = shigh; Z2 = shigh_z;
			//	}
			//	else if (zlow_t >= 0 && zlow_t <= thigh && zlow_t != T1
			//		&& zlow_s >= 0 && zlow_s <= shigh && zlow_s != S1)
			//	{
			//		T2 = zlow_t; S2 = zlow_s; Z2 = zlow;
			//	}
			//	else if (zhigh_t >= 0 && zhigh_t <= thigh && zhigh_t != T1
			//		&& zhigh_s >= 0 && zhigh_s <= shigh && zhigh_s != S1)
			//	{
			//		T2 = zhigh_t; S2 = zhigh_s; Z2 = zhigh;
			//	}
			//	else
			//		return;
			//}
			//else if (thigh_s >= 0 && thigh_s <= shigh && thigh_z >= 0 && thigh_z <= zhigh)
			//{
			//	T1 = thigh; S1 = thigh_s; Z1 = thigh_z;
			//	if (slow_t >= 0 && slow_t <= thigh && slow_t != T1
			//		&& slow_z >= 0 && slow_z <= zhigh && slow_z != Z1)
			//	{
			//		T2 = slow_t; S2 = slow; Z2 = slow_z;
			//	}
			//	else if (shigh_t >= 0 && shigh_t <= thigh && shigh_t != T1
			//		&& shigh_z >= 0 && shigh_z <= zhigh && shigh_z != Z1)
			//	{
			//		T2 = shigh_t; S2 = shigh; Z2 = shigh_z;
			//	}
			//	else if (zlow_t >= 0 && zlow_t <= thigh && zlow_t != T1
			//		&& zlow_s >= 0 && zlow_s <= shigh && zlow_s != S1)
			//	{
			//		T2 = zlow_t; S2 = zlow_s; Z2 = zlow;
			//	}
			//	else if (zhigh_t >= 0 && zhigh_t <= thigh && zhigh_t != T1
			//		&& zhigh_s >= 0 && zhigh_s <= shigh && zhigh_s != S1)
			//	{
			//		T2 = zhigh_t; S2 = zhigh_s; Z2 = zhigh;
			//	}
			//	else
			//		return;
			//}
			//else if (slow_t >= 0 && slow_t <= thigh && slow_z >= 0 && slow_z <= zhigh)
			//{
			//	T1 = slow_t; S1 = slow; Z1 = slow_z;
			//	if (shigh_t >= 0 && shigh_t <= thigh && shigh_t != T1
			//		&& shigh_z >= 0 && shigh_z <= zhigh && shigh_z != Z1)
			//	{
			//		T2 = shigh_t; S2 = shigh; Z2 = shigh_z;
			//	}
			//	else if (zlow_t >= 0 && zlow_t <= thigh && zlow_t != T1
			//		&& zlow_s >= 0 && zlow_s <= shigh && zlow_s != S1)
			//	{
			//		T2 = zlow_t; S2 = zlow_s; Z2 = zlow;
			//	}
			//	else if (zhigh_t >= 0 && zhigh_t <= thigh && zhigh_t != T1
			//		&& zhigh_s >= 0 && zhigh_s <= shigh && zhigh_s != S1)
			//	{
			//		T2 = zhigh_t; S2 = zhigh_s; Z2 = zhigh;
			//	}
			//	else
			//		return;
			//}
			//else if (shigh_t >= 0 && shigh_t <= thigh && shigh_z >= 0 && shigh_z <= zhigh)
			//{
			//	T1 = shigh_t; S1 = shigh; Z1 = shigh_z;
			//	if (zlow_t >= 0 && zlow_t <= thigh && zlow_t != T1
			//		&& zlow_s >= 0 && zlow_s <= shigh && zlow_s != S1)
			//	{
			//		T2 = zlow_t; S2 = zlow_s; Z2 = zlow;
			//	}
			//	else if (zhigh_t >= 0 && zhigh_t <= thigh && zhigh_t != T1
			//		&& zhigh_s >= 0 && zhigh_s <= shigh && zhigh_s != S1)
			//	{
			//		T2 = zhigh_t; S2 = zhigh_s; Z2 = zhigh;
			//	}
			//	else
			//		return;
			//}
			//else if (zlow_t >= 0 && zlow_t <= thigh && zlow_s >= 0 && zlow_s <= shigh)
			//{
			//	T1 = zlow_t; S1 = zlow_s; Z1 = zlow;
			//	if (zhigh_t >= 0 && zhigh_t <= thigh && zhigh_t != T1
			//		&& zhigh_s >= 0 && zhigh_s <= shigh && zhigh_s != S1)
			//	{
			//		T2 = zhigh_t; S2 = zhigh_s; Z2 = zhigh;
			//	}
			//	else
			//		return;
			//}
			//else
			//{
			//	//dev_Projection[threadid] = threadid;
			//	return;
			//}

			//LengthinROI = Distance(T1, S1, Z1, T2, S2, Z2);
			//if (T1 == T2 && S1 == S2 && Z1 == Z2)   // to solve the special case
			//{
			//	dev_Display[thread_id] += T1 * 100000 + S1 * 1000 + Z1 * 10;
			//	return;
			//}

			//	secondly compute length in a single pixel, the process is very similar to the previous.
			//	because this time the line goes through the center point of the pixel. So some kind of symmetry happens.
			//	since the global function can not call exterior function, so the previous code will be copied here.

			//	 actual Size in matlab coordinate
			tlow = Tindex * Resolution_t; thigh = (Tindex + 1) * Resolution_t;
			slow = Sindex * Resolution_s; shigh = (Sindex + 1) * Resolution_s;
			zlow = Zindex * Resolution_z; zhigh = (Zindex + 1) * Resolution_z;

			//	compute the first and last point in the ROI
			//	 using DetectPoint_end set up projection equation

			tlow_s = source_s + (tlow - source_t) * (DetectPoint_send - source_s) / (DetectPoint_tend - source_t + EPS);
			tlow_z = source_z + (tlow - source_t) * (DetectPoint_zend - source_z) / (DetectPoint_tend - source_t + EPS);

			slow_t = source_t + (slow - source_s) * (DetectPoint_tend - source_t) / (DetectPoint_send - source_s + EPS);
			slow_z = source_z + (slow - source_s) * (DetectPoint_zend - source_z) / (DetectPoint_send - source_s + EPS);

			zlow_t = source_t + (zlow - source_z) * (DetectPoint_tend - source_t) / (DetectPoint_zend - source_z + EPS);
			zlow_s = source_s + (zlow - source_z) * (DetectPoint_send - source_s) / (DetectPoint_zend - source_z + EPS);

			//	double *Range = new double [6];   //  XYXY small-big(number)
			T1 = 0; S1 = 0; Z1 = 0; /*T2 = 0; S2 = 0; Z2 = 0;*/

			if (tlow_s >= slow && tlow_s <= shigh && tlow_z >= zlow && tlow_z <= zhigh)
			{
				T1 = tlow; S1 = tlow_s; Z1 = tlow_z;
				//		for the symmetry, there is no need to compute T2
			}
			else if (slow_t >= tlow && slow_t <= thigh && slow_z >= zlow && slow_z <= zhigh)
			{
				T1 = slow_t; S1 = slow; Z1 = slow_z;
				//		for the symmetry, there is no need to compute T2
			}
			else if (zlow_t >= tlow && zlow_t <= thigh && zlow_s >= slow && zlow_s <= shigh)
			{
				T1 = zlow_t; S1 = zlow_s; Z1 = zlow;
				//		for the symmetry, there is no need to compute T2		
			}
			else
			{
				//		dev_Projection[threadid] = threadid;
				return;
			}

			LengthinPixel = 2 * Distance(T1, S1, Z1, DetectPoint_tend, DetectPoint_send, DetectPoint_zend);
			//if (LengthinROI == 0)
			//	return;
			backweight = LengthinPixel /*/ LengthinROI*/;   // no need for normalization
			thread_id = Zindex * (t_length * s_length) + Sindex * t_length + Tindex;
			dev_Display[thread_id] += Display_pBeta * backweight;
		}
	}	
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t BackPro(float *Display, const float *R, const float *Xigamadomain, const float *Pdomain,
	const float *BetaScanRange, const double Distance, const int LBeta, const int LP, const int LXigama,
	const double *Size, const int t_length, const int s_length, const int z_length)
{
	mexPrintf("BackPro!\n");
	float *dev_R = 0;
	float *dev_BetaScanRange = 0, *dev_Pdomain = 0, *dev_Xigamadomain = 0;
	double *dev_Size = 0;

	float *dev_Display = 0;
	float PInt = fabs(Pdomain[1] - Pdomain[0]);
	float XigamaInt = fabs(Xigamadomain[1] - Xigamadomain[0]);
	float BetaScanInt = fabs(BetaScanRange[1] - BetaScanRange[0]);

	float maxP = MAX(Pdomain[0], Pdomain[LP - 1]);
	float minP = MIN(Pdomain[0], Pdomain[LP - 1]);
	float maxXigama = MAX(Xigamadomain[0], Xigamadomain[LXigama - 1]);
	float minXigama = MIN(Xigamadomain[0], Xigamadomain[LXigama - 1]);
	//mexPrintf("%lf %lf %lf %lf \n", maxGama, minGama, maxXigama, minXigama);

	const long LDisplay = t_length * s_length * z_length;
	const long LR = LP * LXigama * LBeta;

	short thread_cubic_Bp_x = MIN(threadX, t_length);
	short block_cubic_Bp_x = MIN(blockX, s_length);

	const dim3 thread_cubic_Bp(thread_cubic_Bp_x, 1, 1);
	const dim3 block_cubic_Bp(block_cubic_Bp_x, 1, 1);

	dim3 thread_cubic_Bp_residual(1, 1, 1);  // initial
	dim3 block_cubic_Bp_residual(1, 1, 1);  // initial

	short TlengthResidual = t_length % threadX;
	short SlengthResidual = s_length % blockX;
	short T_Time = t_length / threadX;
	short S_Time = s_length / blockX;
	short T_start = 0;
	short S_start = 0;
	//float Beta = 0;

	if (TlengthResidual != 0)
	{
		thread_cubic_Bp_residual.x = TlengthResidual;
	}
	if (SlengthResidual != 0)
	{
		block_cubic_Bp_residual.x = SlengthResidual;
	}

	hipError_t cudaStatus;

	mexPrintf("start cuda\n");

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?\n");
		mexPrintf("hipSetDevice failed!  Do you have a CUDA-capable GPU installed? %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	mexPrintf("call for space in GPU\n");

	// Allocate GPU buffers for four vectors (4 inputs).

	cudaStatus = hipMalloc((void**)&dev_R, LR * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_R hipMalloc failed!\n");
		mexPrintf("dev_R hipMalloc failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_Pdomain, LP * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_Pdomain hipMalloc failed!\n");
		mexPrintf("dev_Pdomain hipMalloc failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_BetaScanRange, LBeta * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_BetaScanRange hipMalloc failed!\n");
		mexPrintf("dev_BetaScanRange hipMalloc failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_Xigamadomain, LXigama * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_Xigamadomain hipMalloc failed!\n");
		mexPrintf("dev_Xigamadomain hipMalloc failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	//mexPrintf("copy data in CPU to GPU\n");

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_R, R, LR * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy R failed!\n");
		mexPrintf("hipMemcpy R failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_Pdomain, Pdomain, LP * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy Gamadomain failed!\n");
		mexPrintf("hipMemcpy Gamadomain failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_BetaScanRange, BetaScanRange, LBeta * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy BetaScanRange failed!\n");
		mexPrintf("hipMemcpy BetaScanRange failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_Xigamadomain, Xigamadomain, LXigama * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy Xigamadomain failed!\n");
		mexPrintf("hipMemcpy Xigamadomain failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	mexPrintf("start parallel computation\n");
	mexPrintf("backprojection\n");
	
	cudaStatus = hipMalloc((void**)&dev_Size, 3 * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_Size hipMalloc failed!\n");
		mexPrintf("dev_Size hipMalloc failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_Size, Size, 3 * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy Size failed!\n");
		mexPrintf("hipMemcpy Size failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}
	
    // output
	cudaStatus = hipMalloc((void**)&dev_Display, LDisplay * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_Display hipMalloc failed!\n");
		mexPrintf("dev_Display hipMalloc failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}
	hipMemset(dev_Display, 0, sizeof(float));

	//Backprojection
	for (int betaIndex = 0; betaIndex < LBeta; betaIndex++)
	{
		for (int numT = 0; numT < T_Time; numT++)
		{
			for (int numS = 0; numS < S_Time; numS++)
			{
				T_start = numT * threadX;
				S_start = numS * blockX;
				BackProjection3D << <block_cubic_Bp, thread_cubic_Bp >> > (dev_R, dev_Display, dev_Size, t_length, s_length, z_length,
					BetaScanRange[betaIndex], Distance, dev_Pdomain, dev_Xigamadomain, PInt, XigamaInt, BetaScanInt, minP, maxP,
					minXigama, maxXigama, betaIndex, LP, LXigama, T_start, S_start);
				// Check for any errors launching the kernel
				cudaStatus = hipGetLastError();
				if (cudaStatus != hipSuccess) {
					fprintf(stderr, "BackProjection launch failed: %s\n", hipGetErrorString(cudaStatus));
					mexPrintf("BackProjection launch failed %s\n", hipGetErrorString(cudaStatus));
					mexPrintf("Error happens at betaIndex: %d\n", betaIndex);
					goto Error;
				}
			}
		}
		if (TlengthResidual != 0)
		{
			T_start = t_length - TlengthResidual;
			for (int numS = 0; numS < S_Time; numS++)
			{
				S_start = numS * blockX;
				BackProjection3D << <block_cubic_Bp, thread_cubic_Bp_residual >> > (dev_R, dev_Display, dev_Size, t_length, s_length, z_length,
					BetaScanRange[betaIndex], Distance, dev_Pdomain, dev_Xigamadomain, PInt, XigamaInt, BetaScanInt, minP, maxP,
					minXigama, maxXigama, betaIndex, LP, LXigama, T_start, S_start);
				// Check for any errors launching the kernel
				cudaStatus = hipGetLastError();
				if (cudaStatus != hipSuccess) {
					fprintf(stderr, "BackProjection launch failed: %s\n", hipGetErrorString(cudaStatus));
					mexPrintf("BackProjection launch failed %s\n", hipGetErrorString(cudaStatus));
					mexPrintf("Error happens at betaIndex: %d\n", betaIndex);
					goto Error;
				}
			}
			if (SlengthResidual != 0)
			{
				S_start = s_length - SlengthResidual;
				BackProjection3D << <block_cubic_Bp_residual, thread_cubic_Bp_residual >> > (dev_R, dev_Display, dev_Size, t_length, s_length, z_length,
					BetaScanRange[betaIndex], Distance, dev_Pdomain, dev_Xigamadomain, PInt, XigamaInt, BetaScanInt, minP, maxP,
					minXigama, maxXigama, betaIndex, LP, LXigama, T_start, S_start);
				// Check for any errors launching the kernel
				cudaStatus = hipGetLastError();
				if (cudaStatus != hipSuccess) {
					fprintf(stderr, "BackProjection launch failed: %s\n", hipGetErrorString(cudaStatus));
					mexPrintf("BackProjection launch failed %s\n", hipGetErrorString(cudaStatus));
					mexPrintf("Error happens at betaIndex: %d\n", betaIndex);
					goto Error;
				}
			}
		}
		if (SlengthResidual != 0)
		{
			S_start = s_length - SlengthResidual;
			for (int numT = 0; numT < T_Time; numT++)
			{
				T_start = numT * threadX;
				BackProjection3D << <block_cubic_Bp_residual, thread_cubic_Bp >> > (dev_R, dev_Display, dev_Size, t_length, s_length, z_length,
					BetaScanRange[betaIndex], Distance, dev_Pdomain, dev_Xigamadomain, PInt, XigamaInt, BetaScanInt, minP, maxP,
					minXigama, maxXigama, betaIndex, LP, LXigama, T_start, S_start);
				// Check for any errors launching the kernel
				cudaStatus = hipGetLastError();
				if (cudaStatus != hipSuccess) {
					fprintf(stderr, "BackProjection launch failed: %s\n", hipGetErrorString(cudaStatus));
					mexPrintf("BackProjection launch failed %s\n", hipGetErrorString(cudaStatus));
					mexPrintf("Error happens at betaIndex: %d\n", betaIndex);
					goto Error;
				}
			}
		}
		
	}

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "BackProjection launch failed: %s\n", hipGetErrorString(cudaStatus));
		mexPrintf("BackProjection launch failed %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		mexPrintf("hipDeviceSynchronize returned error code %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	 //Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(Display, dev_Display, LDisplay * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!\n");
		mexPrintf("hipMemcpy dev_Display failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

Error:
	hipFree(dev_R);
	hipFree(dev_BetaScanRange);
	hipFree(dev_Pdomain);
	hipFree(dev_Xigamadomain);
	hipFree(dev_Display);
	hipFree(dev_Size);

	mexPrintf("Exit Bakprojection3D\n");
	return cudaStatus;
}
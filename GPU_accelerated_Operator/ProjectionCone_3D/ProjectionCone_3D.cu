#include "hip/hip_runtime.h"
#include "ProjectionCone_3D.h"
#include <cstdlib>
// 2018/04/16
//__device__ const double PI = 3.141592653589793;
__device__ const double EPS = 1e-15;
__device__ const double ERR = 1e-5;

__global__ void ProjectionCone(const float *dev_Pic, float *dev_Projection, const float *dev_Pdomain, const float *dev_Xigamadomain,
	const double Center_t, const double Center_s, const double Center_z, const double *dev_resolution,
	const int t_length, const int s_length, const int z_length, const float Beta, const int numB, const int Pstart,
	const int Xigamastart, const double Distance, const int LP, const int LXigama, const float *dev_RandomErr)
{  
	const unsigned short Pindex = threadIdx.x + Pstart;
	const unsigned short  Xigamaindex = blockIdx.x + Xigamastart;

	const unsigned long threadid = numB * LXigama * LP + Xigamaindex * LP + Pindex;

	dev_Projection[threadid] = 0;

	float P = dev_Pdomain[Pindex];
	float Xigama = dev_Xigamadomain[Xigamaindex];

	double resolution_1 = dev_resolution[0]; double resolution_2 = dev_resolution[1]; double resolution_3 = dev_resolution[2];

	// according to euler equation   
	double source_t = Center_t - Distance * sin(Beta);      // define the source in matlab coordinate
	//double source_t = Center_t - Distance * sin(Beta) + dev_RandomErr[threadid];  
	double source_s = Center_s + Distance * cos(Beta);
	double source_z = Center_z;
	//double source_z = Center_z + dev_RandomErr[threadid];

	double Gama = atan(Xigama / Distance);          // radian angle in s - z coordinate plane
	double Distance_shift = Distance / cos(Gama);    // length of DO'

	double Theta = atan(P / Distance_shift);        // radian angle in s'-t coordinate plane 

	double Smax = 2 * Distance;

	// define end detect point in matlab coordinate, Note that : 0 is the start
	double DetectPoint_tend = Center_t + Smax * sin(Theta) * cos(Beta) - (Distance - Smax * cos(Theta) * cos(Gama)) * sin(Beta);
	double DetectPoint_send = Center_s + Smax * sin(Theta) * sin(Beta) + (Distance - Smax * cos(Theta) * cos(Gama)) * cos(Beta);
	double DetectPoint_zend = Center_z + Smax * cos(Theta) * sin(Gama);

	double T2S, S2T, T2Z, Z2T, S2Z, Z2S;
	if ((DetectPoint_tend - source_t) == 0)
	{
		T2S = (DetectPoint_send - source_s) / (DetectPoint_tend - source_t + EPS);
		T2Z = (DetectPoint_zend - source_z) / (DetectPoint_tend - source_t + EPS);
	}
	else
	{
		T2S = (DetectPoint_send - source_s) / (DetectPoint_tend - source_t);
		T2Z = (DetectPoint_zend - source_z) / (DetectPoint_tend - source_t);
	}
		
	if ((DetectPoint_send - source_s) == 0)
	{
		S2T = (DetectPoint_tend - source_t) / (DetectPoint_send - source_s + EPS);
		S2Z = (DetectPoint_zend - source_z) / (DetectPoint_send - source_s + EPS);
	}	
	else
	{
		S2T = (DetectPoint_tend - source_t) / (DetectPoint_send - source_s);
		S2Z = (DetectPoint_zend - source_z) / (DetectPoint_send - source_s);
	}
		
	if ((DetectPoint_zend - source_z) == 0)
	{
		Z2T = (DetectPoint_tend - source_t) / (DetectPoint_zend - source_z + EPS);
		Z2S = (DetectPoint_send - source_s) / (DetectPoint_zend - source_z + EPS);
	}		
	else
	{
		Z2T = (DetectPoint_tend - source_t) / (DetectPoint_zend - source_z);
		Z2S = (DetectPoint_send - source_s) / (DetectPoint_zend - source_z);
	}

	// limit the range of slope
	T2S = Maxlim(T2S); T2S = Minlim(T2S);
	T2Z = Maxlim(T2Z); T2Z = Minlim(T2Z);
	S2T = Maxlim(S2T); S2T = Minlim(S2T);
	S2Z = Maxlim(S2Z); S2Z = Minlim(S2Z);
	Z2T = Maxlim(Z2T); Z2T = Minlim(Z2T);
	Z2S = Maxlim(Z2S); Z2S = Minlim(Z2S);

	// to determine the range of t
	short t_signal = 0;

	if (DetectPoint_tend >= source_t)
		t_signal = 1;
	else
		t_signal = -1;

	// to determine the range of s
	short s_signal = 0;

	if (DetectPoint_send >= source_s)
		s_signal = 1;
	else
		s_signal = -1;

	// to determine the range of z
	short z_signal = 0;

	if (DetectPoint_zend >= source_z)
		z_signal = 1;
	else
		z_signal = -1;

	// actual Size
	double tlow = 0, thigh = t_length*resolution_1, slow = 0, shigh = s_length*resolution_2,
		zlow = 0, zhigh = z_length*resolution_3;

	//compute the first and last point in the ROI
	// using DetectPoint_end set up projection equation
	double tlow_s = source_s + (tlow - source_t) * T2S;
	double tlow_z = source_z + (tlow - source_t) * T2Z;
	double thigh_s = source_s + (thigh - source_t) * T2S;
	double thigh_z = source_z + (thigh - source_t) * T2Z;

	double slow_t = source_t + (slow - source_s) * S2T;
	double slow_z = source_z + (slow - source_s) * S2Z;
	double shigh_t = source_t + (shigh - source_s) * S2T;
	double shigh_z = source_z + (shigh - source_s) * S2Z;

	double zlow_t = source_t + (zlow - source_z) * Z2T;
	double zlow_s = source_s + (zlow - source_z) * Z2S;
	double zhigh_t = source_t + (zhigh - source_z) * Z2T;
	double zhigh_s = source_s + (zhigh - source_z) * Z2S;

	//double *Range = new double [6];   //  XYXY small-big(number)
	double T1 = 0, S1 = 0, Z1 = 0, T2 = 0, S2 = 0, Z2 = 0;

	if (tlow_s >= 0 && tlow_s <= shigh && tlow_z >= 0 && tlow_z <= zhigh)
	{
		T1 = tlow; S1 = tlow_s; Z1 = tlow_z;
		if (thigh_s >= 0 && thigh_s <= shigh && thigh_z >= 0 && thigh_z <= zhigh)
		{
			T2 = thigh; S2 = thigh_s; Z2 = thigh_z;
		}
		else if (slow_t >= 0 && slow_t <= thigh && slow_z >= 0 && slow_z <= zhigh)
		{
			T2 = slow_t; S2 = slow; Z2 = slow_z;
		}
		else if (shigh_t >= 0 && shigh_t <= thigh && shigh_z >= 0 && shigh_z <= zhigh)
		{
			T2 = shigh_t; S2 = shigh; Z2 = shigh_z;
		}
		else if (zlow_t >= 0 && zlow_t <= thigh && zlow_s>= 0 && zlow_s <= shigh)
		{
			T2 = zlow_t; S2 = zlow_s; Z2 = zlow;
		}
		else if (zhigh_t >= 0 && zhigh_t <= thigh && zhigh_s >= 0 && zhigh_s <= shigh)
		{
			T2 = zhigh_t; S2 = zhigh_s; Z2 = zhigh;
		}
	}
	else if (thigh_s >= 0 && thigh_s <= shigh && thigh_z >= 0 && thigh_z <= zhigh)
	{
		T1 = thigh; S1 = thigh_s; Z1 = thigh_z;
		if (slow_t >= 0 && slow_t <= thigh && slow_z >= 0 && slow_z <= zhigh)
		{
			T2 = slow_t; S2 = slow; Z2 = slow_z;
		}
		else if (shigh_t >= 0 && shigh_t <= thigh && shigh_z >= 0 && shigh_z <= zhigh)
		{
			T2 = shigh_t; S2 = shigh; Z2 = shigh_z;
		}
		else if (zlow_t >= 0 && zlow_t <= thigh && zlow_s >= 0 && zlow_s <= shigh)
		{
			T2 = zlow_t; S2 = zlow_s; Z2 = zlow;
		}
		else if (zhigh_t >= 0 && zhigh_t <= thigh && zhigh_s >= 0 && zhigh_s <= shigh)
		{
			T2 = zhigh_t; S2 = zhigh_s; Z2 = zhigh;
		}
	}
	else if (slow_t >= 0 && slow_t <= thigh && slow_z >= 0 && slow_z <= zhigh)
	{
		T1 = slow_t; S1 = slow; Z1 = slow_z;
		if (shigh_t >= 0 && shigh_t <= thigh && shigh_z >= 0 && shigh_z <= zhigh)
		{
			T2 = shigh_t; S2 = shigh; Z2 = shigh_z;
		}
		else if (zlow_t >= 0 && zlow_t <= thigh && zlow_s >= 0 && zlow_s <= shigh)
		{
			T2 = zlow_t; S2 = zlow_s; Z2 = zlow;
		}
		else if (zhigh_t >= 0 && zhigh_t <= thigh && zhigh_s >= 0 && zhigh_s <= shigh)
		{
			T2 = zhigh_t; S2 = zhigh_s; Z2 = zhigh;
		}
	}
	else if (shigh_t >= 0 && shigh_t <= thigh && shigh_z >= 0 && shigh_z <= zhigh)
	{
		T1 = shigh_t; S1 = shigh; Z1 = shigh_z;
		if (zlow_t >= 0 && zlow_t <= thigh && zlow_s >= 0 && zlow_s <= shigh)
		{
			T2 = zlow_t; S2 = zlow_s; Z2 = zlow;
		}
		else if (zhigh_t >= 0 && zhigh_t <= thigh && zhigh_s >= 0 && zhigh_s <= shigh)
		{
			T2 = zhigh_t; S2 = zhigh_s; Z2 = zhigh;
		}
	}
	else if (zlow_t >= 0 && zlow_t <= thigh && zlow_s >= 0 && zlow_s <= shigh)
	{
		T1 = zlow_t; S1 = zlow_s; Z1 = zlow;
		if (zhigh_t >= 0 && zhigh_t <= thigh && zhigh_s >= 0 && zhigh_s <= shigh)
		{
			T2 = zhigh_t; S2 = zhigh_s; Z2 = zhigh;
		}
	}
	else
	{
		//dev_Projection[threadid] = threadid;
		return;
	}

	// set the start point
	double TStart = 0, SStart = 0, ZStart = 0;
	if (Distancesq(T1, S1, Z1, source_t, source_s, source_z) >= Distancesq(T2, S2, Z2, source_t, source_s, source_z))
	{
		TStart = T2;
		SStart = S2;
		ZStart = Z2;
	}
	else
	{
		TStart = T1;
		SStart = S1;
		ZStart = Z1;
	}

	// adjust the order

	if (T2 < T1)
	{
		double c = T1;
		T1 = T2;
		T2 = c;
	}
	if (S2 < S1)
	{
		double c = S1;
		S1 = S2;
		S2 = c;
	}
	if (Z2 < Z1)
	{
		double c = Z1;
		Z1 = Z2;
		Z1 = c;
	}

	//// enter the ROI
	double weight = 0, Ray = 0;
	short GridT = 0, GridS = 0, GridZ = 0;        // candidate crosspoint index in matlab(0~t_length)
	double GridT_s = 0, GridT_z = 0,
		GridS_t = 0, GridS_z = 0,
		GridZ_t = 0, GridZ_s = 0;    // candidate crosspoint index in matlab(0~256)
	short DetectPoint_t = 0, DetectPoint_s = 0, DetectPoint_z = 0;   // current pixel index in matlab pixel index in matlab(0~255)
	long Pointid = 0;
	double TCross = TStart / resolution_1, SCross = SStart / resolution_2,
		ZCross = ZStart / resolution_3;     // current crosspoint index in matlab(0~256)
	
	//while (((XCross * dev_resolution[1]) >= Range[0]) && ((XCross * dev_resolution[1]) <= Range[2]) 
	//	&& ((YCross * dev_resolution[0]) >= Range[1]) && ((YCross * dev_resolution[0]) <= Range[3]))

	for (short i = 0;i<(t_length + s_length + z_length - 2);i++)
	{
		// judge whether XCross/YCross is integer
		if (TCross - (double)((short)TCross) < EPS)
		{
			GridT = TCross + t_signal;
		}
		else
		{
			GridT = floor(TCross) + flag1to1or_1to0(t_signal);
		}
		GridT_s = (source_s + (GridT * resolution_1 - source_t) * T2S) / resolution_2;
		GridT_z = (source_z + (GridT * resolution_1 - source_t) * T2Z) / resolution_3;

		if (SCross - (double)((short)SCross) < EPS)
		{
			GridS = SCross + s_signal;
		}
		else
		{
			GridS = floor(SCross) + flag1to1or_1to0(s_signal);
		}
		GridS_t = (source_t + (GridS * resolution_2 - source_s) * S2T) / resolution_1;
		GridS_z = (source_z + (GridS * resolution_2 - source_s) * S2Z) / resolution_3;

		if (ZCross - (double)((short)ZCross) < EPS)
		{
			GridZ = ZCross + z_signal;
		}
		else
		{
			GridZ = floor(ZCross) + flag1to1or_1to0(z_signal);
		}
		GridZ_t = (source_t + (GridZ * resolution_3 - source_z) * Z2T) / resolution_1;
		GridZ_s = (source_s + (GridZ * resolution_3 - source_z) * Z2S) / resolution_2;

		//judge which crosspoint is the nearest, means the smallest distance
		if (Distancesq(GridT, GridT_s, GridT_z, TCross, SCross, ZCross) <= Distancesq(GridS_t, GridS, GridS_z, TCross, SCross, ZCross))
		{
			if (Distancesq(GridZ_t, GridZ_s, GridZ, TCross, SCross, ZCross) <= Distancesq(GridT, GridT_s, GridT_z, TCross, SCross, ZCross))
			{
				weight = sqrt(Distancesq(GridZ_t * resolution_1, GridZ_s * resolution_2,
					GridZ * resolution_3, TCross * resolution_1, SCross * resolution_2,
					ZCross * resolution_3));
				DetectPoint_t = floor(MID(GridZ_t, TCross));                 // the midpoint locates the pixel
				DetectPoint_s = floor(MID(GridZ_s, SCross));
				DetectPoint_z = floor(MID(GridZ, ZCross));
				TCross = GridZ_t;    // update
				SCross = GridZ_s;
				ZCross = GridZ;
			}
			else
			{
				weight = sqrt(Distancesq(GridT * resolution_1, GridT_s * resolution_2,
					GridT_z * resolution_3, TCross * resolution_1, SCross * resolution_2,
					ZCross * resolution_3));
				DetectPoint_t = floor(MID(GridT, TCross));                 // the midpoint locates the pixel
				DetectPoint_s = floor(MID(GridT_s, SCross));
				DetectPoint_z = floor(MID(GridT_z, ZCross));
				TCross = GridT;    // update
				SCross = GridT_s;
				ZCross = GridT_z;
			}			
		}
		else
		{
			if (Distancesq(GridZ_t, GridZ_s, GridZ, TCross, SCross, ZCross) <= Distancesq(GridS_t, GridS, GridS_z, TCross, SCross, ZCross))
			{
				weight = sqrt(Distancesq(GridZ_t * resolution_1, GridZ_s * resolution_2,
					GridZ * resolution_3, TCross * resolution_1, SCross * resolution_2,
					ZCross * resolution_3));
				DetectPoint_t = floor(MID(GridZ_t, TCross));                 // the midpoint locates the pixel
				DetectPoint_s = floor(MID(GridZ_s, SCross));
				DetectPoint_z = floor(MID(GridZ, ZCross));
				TCross = GridZ_t;    // update
				SCross = GridZ_s;
				ZCross = GridZ;
			}
			else
			{
				weight = sqrt(Distancesq(GridS_t * resolution_1, GridS * resolution_2,
					GridS_z * resolution_3, TCross * resolution_1, SCross * resolution_2,
					ZCross * resolution_3));
				DetectPoint_t = floor(MID(GridS_t, TCross));                 // the midpoint locates the pixel
				DetectPoint_s = floor(MID(GridS, SCross));
				DetectPoint_z = floor(MID(GridS_z, ZCross));
				TCross = GridS_t;    // update
				SCross = GridS;
				ZCross = GridS_z;
			}
		}

		//judge whether the point is in the ROI
		if ((DetectPoint_t >= 0) && (DetectPoint_t < t_length) && (DetectPoint_s >= 0) && (DetectPoint_s < s_length)
			&& (DetectPoint_z >= 0) && (DetectPoint_z < z_length))
		{
			Pointid = DetectPoint_z * t_length * s_length + DetectPoint_s * t_length + DetectPoint_t;
			Ray += weight * dev_Pic[Pointid];
		}
		else
		{
			//dev_Projection[threadid] = 9000;
			break;
		}

	}

	__syncthreads();
	dev_Projection[threadid] = Ray;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t ProjectionCone_3D(const float *Pic, float *Projection, const float *BetaScanRange, const float *Pdomain,
	const float *Xigamadomain, const int t_length, const int s_length, const int z_length, const double Center_t,
	const double Center_s, const double Center_z, const int LBeta, const int LP, const int LXigama, const double Distance,
	const double *resolution)
{
	mexPrintf("Hello GenMatParalell!\n");

	float *dev_Pic = 0, *dev_Pdomain = 0, *dev_Xigamadomain = 0, *dev_Projection = 0, *dev_RandomErr = 0;
	double *dev_resolution = 0;

	int threadcubic_x = MIN(threadX, LP);
	int blockcubic_x = MIN(blockX, LXigama);
	int LPResidual = LP % threadX;
	int LXigamaResidual = LXigama % blockX;
	int PTime = LP / threadX;
	int XigamaTime = LXigama / blockX;
	int Pstart = 0;
	int Xigamastart = 0;
	float Beta = 0;
	const dim3 thread_cubic(threadcubic_x, 1, 1);
	const dim3 block_cubic(blockcubic_x, 1, 1);
	dim3 thread_cubic_residual(1, 1, 1);  // initial
	dim3 block_cubic_residual(1, 1, 1);  // initial

	mexPrintf("threadcubic_x: %d blockcubic_x: %d LPResidual: %d LXigamaResidual: %d\n",
		threadcubic_x, blockcubic_x, LPResidual, LXigamaResidual);

	if (LPResidual != 0)
	{
		thread_cubic_residual.x = LPResidual;
	}
	if (LXigamaResidual != 0)
	{
		block_cubic_residual.x = LXigamaResidual;
	}

	hipError_t cudaStatus;

	float *RandomErr =new float[LBeta * LP * LXigama];
	for (int beta = 0; beta < LBeta; beta++)
	{
		for (int P = 0; P < LP; P++)
		{
			for (int Xigama = 0; Xigama < LXigama; Xigama++)
			{
				RandomErr[beta * LXigama * LP + Xigama * LP + P] = 0.01 * (rand()/float(RAND_MAX)*2-1);
			}
		}
	}

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		mexPrintf("hipSetDevice failed!  Do you have a CUDA-capable GPU installed?\n");
		goto Error;
	}

	mexPrintf("Call for GPU space.\n");

	// Allocate GPU buffers for three vectors (two input, one output).
	cudaStatus = hipMalloc((void**)&dev_Pic, t_length * s_length * z_length * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_Pic hipMalloc failed!");
		mexPrintf("dev_Pic hipMalloc failed!\n");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_Projection, LBeta * LP * LXigama * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_Projection hipMalloc failed!");
		mexPrintf("dev_Projection hipMalloc failed!\n");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&dev_RandomErr, LBeta * LP * LXigama * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_Projection hipMalloc failed!");
		mexPrintf("dev_Projection hipMalloc failed!\n");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_Pdomain, LP * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_t_Range hipMalloc failed!");
		mexPrintf("dev_t_Range hipMalloc failed!\n");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_Xigamadomain, LXigama * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_resolution hipMalloc failed!");
		mexPrintf("dev_resolution hipMalloc failed!\n");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_resolution, 3 * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_resolution hipMalloc failed!");
		mexPrintf("dev_resolution hipMalloc failed!\n");
		goto Error;
	}

	mexPrintf("Copy data from CPU to GPU.\n");

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_Pic, Pic, t_length * s_length * z_length * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "thetaRange hipMemcpy failed!");
		mexPrintf("thetaRange hipMemcpy failed!\n");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_Pdomain, Pdomain, LP * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "t_Range hipMemcpy failed!");
		mexPrintf("t_Range hipMemcpy failed!\n");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_Xigamadomain, Xigamadomain, LXigama * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "t_Range hipMemcpy failed!");
		mexPrintf("t_Range hipMemcpy failed!\n");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_resolution, resolution, 3 * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_resolution hipMemcpy failed!");
		mexPrintf("dev_resolution hipMemcpy failed!\n");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_RandomErr, RandomErr, LBeta * LP * LXigama * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_RandomErr hipMemcpy failed!");
		mexPrintf("dev_RandomErr hipMemcpy failed!\n");
		goto Error;
	}

	mexPrintf("Launch computation projection of each lines.\n");
	
	// Launch a kernel on the GPU with one thread for each element.
	for (int numB = 0; numB < LBeta; numB++)
	{
		Beta = BetaScanRange[numB];
		for (int numP = 0; numP < PTime; numP++)
		{
			for (int numX = 0; numX < XigamaTime; numX++)
			{
				Pstart = numP * threadX;
				Xigamastart = numX * blockX;
				//mexPrintf("%d %d\n", Pstart, Xigamastart);
				ProjectionCone << <block_cubic, thread_cubic >> >(dev_Pic, dev_Projection, dev_Pdomain, dev_Xigamadomain,
					Center_t, Center_s, Center_z, dev_resolution, t_length, s_length, z_length, Beta, numB, Pstart, Xigamastart,
					Distance, LP, LXigama, dev_RandomErr);
				// Check for any errors launching the kernel
				cudaStatus = hipGetLastError();
				if (cudaStatus != hipSuccess) {
					fprintf(stderr, "ProjectionCone launch failed: %s\n", hipGetErrorString(cudaStatus));
					mexPrintf("ProjectionCone launch failed %s\n", hipGetErrorString(cudaStatus));
					mexPrintf("Error happens at numB: %d Pstart: %d Xigamastart: %d \n",
						numB, Pstart, Xigamastart);
					goto Error;
				}
			}		
		}
		
		if (LPResidual != 0)
		{
			Pstart = LP - LPResidual;			
			if (LXigamaResidual != 0)
			{
				Xigamastart = LXigama - LXigamaResidual;
				//("%d %d\n", Pstart, Xigamastart);
				ProjectionCone << <block_cubic_residual, thread_cubic_residual >> >(dev_Pic, dev_Projection, dev_Pdomain, dev_Xigamadomain,
					Center_t, Center_s, Center_z, dev_resolution, t_length, s_length, z_length, Beta, numB, Pstart, Xigamastart,
					Distance, LP, LXigama, dev_RandomErr);
				// Check for any errors launching the kernel
				cudaStatus = hipGetLastError();
				if (cudaStatus != hipSuccess) {
					fprintf(stderr, "ProjectionCone launch failed: %s\n", hipGetErrorString(cudaStatus));
					mexPrintf("ProjectionCone launch failed %s\n", hipGetErrorString(cudaStatus));
					mexPrintf("Error happens at numB: %d Pstart: %d Xigamastart: %d \n",
						numB, Pstart, Xigamastart);
					goto Error;
				}
			}

			for (int numX = 0; numX < XigamaTime; numX++)
			{
				Xigamastart = numX * blockX;
				//("%d %d\n", Pstart, Xigamastart);
				ProjectionCone << <block_cubic, thread_cubic_residual >> >(dev_Pic, dev_Projection, dev_Pdomain, dev_Xigamadomain,
					Center_t, Center_s, Center_z, dev_resolution, t_length, s_length, z_length, Beta, numB, Pstart, Xigamastart,
					Distance, LP, LXigama, dev_RandomErr);
				// Check for any errors launching the kernel
				cudaStatus = hipGetLastError();
				if (cudaStatus != hipSuccess) {
					fprintf(stderr, "ProjectionCone launch failed: %s\n", hipGetErrorString(cudaStatus));
					mexPrintf("ProjectionCone launch failed %s\n", hipGetErrorString(cudaStatus));
					mexPrintf("Error happens at numB: %d Pstart: %d Xigamastart: %d \n",
						numB, Pstart, Xigamastart);
					goto Error;
				}
			}				
		}
		if (LXigamaResidual != 0)
		{
			Xigamastart = LXigama - LXigamaResidual;
			for (int numP = 0; numP < PTime; numP++)
			{
				Pstart = numP * threadX;
				//mexPrintf("%d %d\n", Pstart, Xigamastart);
				ProjectionCone << <block_cubic_residual, thread_cubic >> >(dev_Pic, dev_Projection, dev_Pdomain, dev_Xigamadomain,
					Center_t, Center_s, Center_z, dev_resolution, t_length, s_length, z_length, Beta, numB, Pstart, Xigamastart,
					Distance, LP, LXigama, dev_RandomErr);
				// Check for any errors launching the kernel
				cudaStatus = hipGetLastError();
				if (cudaStatus != hipSuccess) {
					fprintf(stderr, "ProjectionCone launch failed: %s\n", hipGetErrorString(cudaStatus));
					mexPrintf("ProjectionCone launch failed %s\n", hipGetErrorString(cudaStatus));
					mexPrintf("Error happens at numB: %d Pstart: %d Xigamastart: %d \n",
						numB, Pstart, Xigamastart);
					goto Error;
				}
			}		
		}
	}

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Projection launch failed: %s\n", hipGetErrorString(cudaStatus));
		mexPrintf("Projection launch failed\n");
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		mexPrintf("hipDeviceSynchronize returned error %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(Projection, dev_Projection, LBeta * LP * LXigama * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!\n");
		mexPrintf("hipMemcpy failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

Error:
	hipFree(dev_Pdomain);
	hipFree(dev_Xigamadomain);
	hipFree(dev_Projection);
	hipFree(dev_Pic);
	hipFree(dev_resolution);

	return cudaStatus;
}

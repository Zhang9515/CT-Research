#include "hip/hip_runtime.h"
#include "BackprojectionFan2D.h"
// 2019/09/14 apply GPU acceleration
#define pow2(x) (1.0*(x)*(x))
__device__ const double PI = 3.141592653589793;
__device__ const double EPS = 1e-15;

__global__ void BackProjection(const float *dev_R, float *dev_Display, const double *dev_Size, const int T_length, const int S_length,
	const float Beta, const float *dev_Pdomain, const float PInt, const float BetaScanInt, const float minP, 
	const float maxP, const int BetaIndex, const int LP, const int LBeta, const int T_start, const int S_start, const double Rscan)
{
	// here makes clear an important note: t in lower case means the location on the detector, while T in captial case means a dimension of the axis.
	// only take effect in this function
	// initialize
	const double Resolution_t = 1.0 * dev_Size[0] / T_length;
	const double Resolution_s = 1.0 * dev_Size[1] / S_length;
	// rotation center
	double Center_t = dev_Size[0] / 2;
	double Center_s = dev_Size[1] / 2;
	// index 
	const unsigned int Tindex = T_start + threadIdx.x;
	const unsigned int Sindex = S_start + blockIdx.x;

	double image_t, image_s;
	unsigned int P1index, P2index;
	double P_domain1, P_domain2;
	double P1, P2;
	double Display_pBeta = 0;
	double P, S, U, P_domain;
	unsigned long thread_id;
	double backweight = 0;

	// this is a little different from code on MATLAB
	image_t = (Tindex + 0.5) * Resolution_t - Center_t;    // center of the image pixel in ground coordinate
	image_s = (Sindex + 0.5) * Resolution_s - Center_s;

	P = image_t * cos(Beta) + image_s * sin(Beta);   // rotate in ground coordinate
	S = -image_t * sin(Beta) + image_s * cos(Beta);
	U = (Rscan - S) / Rscan;       // proportion of length

	P_domain = P / U;
	
	if ((P_domain >= minP) && (P_domain < maxP))
	{
		P1index = floor(fabs(P_domain - dev_Pdomain[0]) / PInt);
		P2index = P1index + 1;

		P_domain1 = dev_Pdomain[P1index]; P_domain2 = dev_Pdomain[P2index];
	
		//bilinear interpolation
		P1 = fabs(P_domain - P_domain1); P2 = fabs(P_domain2 - P_domain);

		//Display_pBeta = 1;
		Display_pBeta = ( P2 * dev_R[BetaIndex * LP + P1index] + P1 * dev_R[BetaIndex * LP + P2index] ) / PInt ;
	}

	double source_t = Center_t - Rscan * sin(Beta);      // define the source in matlab coordinate
	double source_s = Center_s + Rscan * cos(Beta);

	double Theta = atan(P_domain / Rscan);        // radian angle in s'-t coordinate plane 

	double Smax = 2 * Rscan;

	// define end detect point in matlab coordinate, Note that : 0 is the start
	double DetectPoint_tend = Center_t + Smax * sin(Theta) * cos(Beta) - (Rscan - Smax * cos(Theta)) * sin(Beta);
	double DetectPoint_send = Center_s + Smax * sin(Theta) * sin(Beta) + (Rscan - Smax * cos(Theta)) * cos(Beta);

	double T2S, S2T;
	if ((DetectPoint_tend - source_t) == 0)
		T2S = (DetectPoint_send - source_s) / (DetectPoint_tend - source_t + EPS);
	else
		T2S = (DetectPoint_send - source_s) / (DetectPoint_tend - source_t);
	if ((DetectPoint_send - source_s) == 0)
		S2T = (DetectPoint_tend - source_t) / (DetectPoint_send - source_s + EPS);
	else
		S2T = (DetectPoint_tend - source_t) / (DetectPoint_send - source_s);

	// limit the range of slope
	T2S = Maxlim(T2S); T2S = Minlim(T2S);
	S2T = Maxlim(S2T); S2T = Minlim(S2T);

	// actual Size
	double tlow = Tindex * Resolution_t, thigh = (Tindex+1) * Resolution_t, slow = Sindex * Resolution_s, shigh = (Sindex+1) * Resolution_s;

	//compute the first and last point in the ROI
	// using DetectPoint_end set up projection equation
	double tlow_s = source_s + (tlow - source_t) * T2S;
	double thigh_s = source_s + (thigh - source_t) * T2S;

	double slow_t = source_t + (slow - source_s) * S2T;
	double shigh_t = source_t + (shigh - source_s) * S2T;

	double T1 = 0, S1 = 0;
	double LengthinPixel =0;

	if (tlow_s >= 0 && tlow_s <= shigh)
	{
		T1 = tlow; S1 = tlow_s;
	}
	else if (slow_t >= 0 && slow_t <= thigh)
	{
		T1 = slow_t; S1 = slow;
	}
	else
	{
		//dev_Projection[threadid] = threadid;
		return;
	}

	LengthinPixel = 2 * Distance(T1, S1, DetectPoint_tend, DetectPoint_send);
	//if (LengthinROI == 0)
	//	return;
	backweight = LengthinPixel /*/ LengthinROI*/;   // no need for normalization
	thread_id = Sindex * T_length + Tindex;
	dev_Display[thread_id] += Display_pBeta * backweight;

}

// HeLTer function for using CUDA to add vectors in parallel.
hipError_t FBPfan(float *Display, const float *R, const float *Pdomain, const float *BetaScanRange, const int LBeta,
	const int LP, const double *Size, const int t_length, const int s_length, const double Rscan)
{
	mexPrintf("Hello FBPpara!\n");
	float *dev_R = 0;
	float *dev_Pdomain = 0;
	double *dev_Size = 0;

	float *dev_Display = 0;
	float PInt = fabs(Pdomain[1] - Pdomain[0]);
	float BetaScanInt = fabs(BetaScanRange[1] - BetaScanRange[0]);

	float maxP = MAX(Pdomain[0], Pdomain[LP - 1]);
	float minP = MIN(Pdomain[0], Pdomain[LP - 1]);
	//mexPrintf("%lf %lf %lf %lf \n", maxGama, minGama, maxXigama, minXigama);

	const long LDisplay = t_length * s_length;
	const long LR = LP * LBeta;

	short thread_cubic_Bp_x = MIN(threadX, t_length);
	short block_cubic_Bp_x = MIN(blockX, s_length);

	const dim3 thread_cubic_Bp(thread_cubic_Bp_x, 1, 1);
	const dim3 block_cubic_Bp(block_cubic_Bp_x, 1, 1);

	dim3 thread_cubic_Bp_residual(1, 1, 1);  // initial
	dim3 block_cubic_Bp_residual(1, 1, 1);  // initial

	short TlengthResidual = t_length % threadX;
	short SlengthResidual = s_length % blockX;
	short T_Time = t_length / threadX;
	short S_Time = s_length / blockX;
	short T_start = 0;
	short S_start = 0;

	//mexPrintf("pTime: %d BetaTime: %d\n", pTime, BetaTime);

	if (TlengthResidual != 0)
	{
		thread_cubic_Bp_residual.x = TlengthResidual;
	}
	if (SlengthResidual != 0)
	{
		block_cubic_Bp_residual.x = SlengthResidual;
	}

	mexPrintf("thread_cubic_Bp_x: %d block_cubic_Bp_x: %d TlengthResidual: %d SlengthResidual: %d\n",
		thread_cubic_Bp_x, block_cubic_Bp_x, TlengthResidual, SlengthResidual);

	hipError_t cudaStatus;

	///////////////////////////////////////////////////////////////////////////////////////////////	
	mexPrintf("start cuda\n");

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?\n");
		mexPrintf("hipSetDevice failed!  Do you have a CUDA-capable GPU installed? %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	mexPrintf("call for space in GPU\n");

	// Allocate GPU buffers for three vectors (two input, one output).

	cudaStatus = hipMalloc((void**)&dev_R, LR * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_R hipMalloc failed!\n");
		mexPrintf("dev_R hipMalloc failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_Pdomain, LP * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_Pdomain hipMalloc failed!\n");
		mexPrintf("dev_Pdomain hipMalloc failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	//mexPrintf("copy data in CPU to GPU\n");

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_R, R, LR * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy R failed!\n");
		mexPrintf("hipMemcpy R failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_Pdomain, Pdomain, LP * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy Pdomain failed!\n");
		mexPrintf("hipMemcpy Pdomain failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	mexPrintf("start parallel computation\n");

	mexPrintf("backprojection\n");

	cudaStatus = hipMalloc((void**)&dev_Size, 2 * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_Size hipMalloc failed!\n");
		mexPrintf("dev_Size hipMalloc failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_Size, Size, 2 * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy Size failed!\n");
		mexPrintf("hipMemcpy Size failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_Display, LDisplay * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_Display hipMalloc failed!\n");
		mexPrintf("dev_Display hipMalloc failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}
	hipMemset(dev_Display, 0, sizeof(float));

	//Backprojection  
	for (int BetaIndex = 0; BetaIndex < LBeta; BetaIndex++)
	{
		for (int numT = 0; numT < T_Time; numT++)
		{
			for (int numS = 0; numS < S_Time; numS++)
			{
				T_start = numT * threadX;
				S_start = numS * blockX;
				BackProjection << <block_cubic_Bp, thread_cubic_Bp >> > (dev_R, dev_Display, dev_Size, t_length, s_length,
					BetaScanRange[BetaIndex], dev_Pdomain, PInt, BetaScanInt, minP, maxP, BetaIndex, LP, LBeta, T_start, S_start, Rscan);
				
				// Check for any errors launching the kernel
				cudaStatus = hipGetLastError();
				if (cudaStatus != hipSuccess) {
					fprintf(stderr, "BackProjection launch failed: %s\n", hipGetErrorString(cudaStatus));
					mexPrintf("BackProjection launch failed %s\n", hipGetErrorString(cudaStatus));
					mexPrintf("Error happens at BetaIndex: %d\n", BetaIndex);
					goto Error;
				}
			}
		}
		if (TlengthResidual != 0)
		{
			T_start = t_length - TlengthResidual;
			for (int numS = 0; numS < S_Time; numS++)
			{
				S_start = numS * blockX;
				BackProjection << <block_cubic_Bp, thread_cubic_Bp_residual >> > (dev_R, dev_Display, dev_Size, t_length, s_length,
					BetaScanRange[BetaIndex], dev_Pdomain, PInt, BetaScanInt, minP, maxP, BetaIndex, LP, LBeta, T_start, S_start, Rscan);
				// Check for any errors launching the kernel
				cudaStatus = hipGetLastError();
				if (cudaStatus != hipSuccess) {
					fprintf(stderr, "BackProjection launch failed: %s\n", hipGetErrorString(cudaStatus));
					mexPrintf("BackProjection launch failed %s\n", hipGetErrorString(cudaStatus));
					mexPrintf("Error happens at BetaIndex: %d\n", BetaIndex);
					goto Error;
				}
			}
			if (SlengthResidual != 0)
			{
				S_start = s_length - SlengthResidual;
				BackProjection << <block_cubic_Bp_residual, thread_cubic_Bp_residual >> > (dev_R, dev_Display, dev_Size, t_length, s_length,
					BetaScanRange[BetaIndex], dev_Pdomain, PInt, BetaScanInt, minP, maxP, BetaIndex, LP, LBeta, T_start, S_start, Rscan);
				// Check for any errors launching the kernel
				cudaStatus = hipGetLastError();
				if (cudaStatus != hipSuccess) {
					fprintf(stderr, "BackProjection launch failed: %s\n", hipGetErrorString(cudaStatus));
					mexPrintf("BackProjection launch failed %s\n", hipGetErrorString(cudaStatus));
					mexPrintf("Error happens at BetaIndex: %d\n", BetaIndex);
					goto Error;
				}
			}
		}
		if (SlengthResidual != 0)
		{
			S_start = s_length - SlengthResidual;
			for (int numT = 0; numT < T_Time; numT++)
			{
				T_start = numT * threadX;
				BackProjection << <block_cubic_Bp_residual, thread_cubic_Bp >> > (dev_R, dev_Display, dev_Size, t_length, s_length,
					BetaScanRange[BetaIndex], dev_Pdomain, PInt, BetaScanInt, minP, maxP, BetaIndex, LP, LBeta, T_start, S_start, Rscan);
				// Check for any errors launching the kernel
				cudaStatus = hipGetLastError();
				if (cudaStatus != hipSuccess) {
					fprintf(stderr, "BackProjection launch failed: %s\n", hipGetErrorString(cudaStatus));
					mexPrintf("BackProjection launch failed %s\n", hipGetErrorString(cudaStatus));
					mexPrintf("Error happens at BetaIndex: %d\n", BetaIndex);
					goto Error;
				}
			}
		}		
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		mexPrintf("hipDeviceSynchronize returned error code %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	//Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(Display, dev_Display, LDisplay * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!\n");
		mexPrintf("hipMemcpy dev_Display failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}
	//for debug{
	//cudaStatus = hipMemcpy(Rcov, dev_Rcov, LR * sizeof(float), hipMemcpyDeviceToHost);
	//if (cudaStatus != hipSuccess) {
	//	fprintf(stderr, "hipMemcpy failed!\n");
	//	mexPrintf("hipMemcpy dev_Display failed! %s\n", hipGetErrorString(cudaStatus));
	//	goto Error;
	//}
	//}end debug

Error:
	hipFree(dev_Pdomain);
	hipFree(dev_R);
	hipFree(dev_Display);
	hipFree(dev_Size);

	mexPrintf("Exit FBP\n");
	return cudaStatus;
}
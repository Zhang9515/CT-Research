#include "hip/hip_runtime.h"
#include "IterativeFDKerrCompute.h"
#include <cstdlib>
// 2018/08/07
hipError_t IterativeFDKerrCompute(const float *Pic, float *ErrorSlicecuda, const float *BetaScanRange, const float *Udomain,
	const double *Pdomain, const double *G, const double *Size, const int t_length, const int s_length, const int z_length,
	const double Center_t, const double Center_s, const double Center_z, const int LBeta, const int LP, const int LU,
	const int LG, const double Distance, const double *resolution2, const int t, const float BetaScanInt, const float dU,
	const double PInt);

__device__ const double PI = 3.141592653589793;

__global__ void IterativeFDKerr(const float *dev_Pic, float *dev_ErrorSlicecuda, const float U, const double *dev_Pdomain,
	 const double *dev_G, const double *dev_Size, const double *dev_resolution2, const double Center_t,
	const double Center_s, const double Center_z, const int t_length, const int s_length, 
	const int z_length, const float Beta, const int numB, const int sstart, const int zstart, const double Distance, 
	const int LBeta, const int LP, const int LU, const int LG, const int t, const float BetaScanInt, const float dU, const double PInt)
	
{
	const unsigned int s = threadIdx.x + sstart;
	const unsigned int z = blockIdx.x + zstart;
	 
	const unsigned long threadid = numB * s_length * z_length + z * s_length + s;
	//const unsigned long threadid = z * s_length * LBeta + s * LBeta + numB;

	float image_t = 0, image_s = 0, image_z = 0, dect_t = 0, dect_s = 0, dect_z = 0, LengthRatio = 0, T_deriv = 0, S_deriv = 0,
		Z_deriv = 0, X_deriv = 0, Y_deriv = 0;
	double P = 0 ;

	int filter_index = 0, X_deriv_index = 0, Y_deriv_index = 0, Z_deriv_index = 0, Pic_index1 = 0, Pic_index2 = 0;

	// image pixel in ground coordinate
	image_t = (t + 0.5) * dev_resolution2[0] - Center_t;
	image_s = (s + 0.5) * dev_resolution2[1] - Center_s;
	image_z = (z + 0.5) * dev_resolution2[2] - Center_z;

	dect_t = image_t * cos(Beta) + image_s * sin(Beta);          // in rotate coordinate
	dect_s = -image_t * sin(Beta) + image_s * cos(Beta);
	dect_z = image_z;

	LengthRatio = Distance / (Distance - dect_s);
	for (int p = 0; p<LP; p++)
	{
		P = dev_Pdomain[p];
		filter_index = floor((dect_t * LengthRatio - P) / PInt) + LP;
		if (filter_index >= 0 && filter_index < LG)
		{
		/*	for (int u = 0; u<LU; u++)
			{*/
				T_deriv = P * (Distance - U) / Distance;                         // in rotate coordinate
				S_deriv = U;
				Z_deriv = dect_z + (dect_s - U) * dect_z / (Distance - dect_s);

				X_deriv = T_deriv * cos(Beta) - S_deriv * sin(Beta);          // in ground coordinate
				Y_deriv = T_deriv * sin(Beta) + S_deriv * cos(Beta);

				X_deriv_index = floor((X_deriv + Center_t) / dev_resolution2[0]);
				Y_deriv_index = floor((Y_deriv + Center_s) / dev_resolution2[1]);
				Z_deriv_index = floor((Z_deriv + Center_z) / dev_resolution2[2]);
				
				
				if (X_deriv_index >= 0 && X_deriv_index < t_length  &&  Y_deriv_index >= 0
					&& Y_deriv_index < s_length && Z_deriv_index >= 0 && Z_deriv_index < z_length)
				{
					Pic_index1 = Z_deriv_index * (t_length * s_length) + Y_deriv_index * t_length + X_deriv_index;
					Pic_index2 = z * (t_length * s_length) + Y_deriv_index * t_length + X_deriv_index;

					//dev_ErrorSlicecuda[threadid] = BetaScanInt ;
					dev_ErrorSlicecuda[threadid] += 0.5 * pow2(LengthRatio) * (dev_Pic[Pic_index1]
						- dev_Pic[Pic_index2]) * sqrt(1 + pow2(P / Distance))
						* dev_G[filter_index] * Distance / sqrt(pow2(Distance) + pow2(P)) * dU * PInt * BetaScanInt;
				}//if-xyzindex
			//}// for-u
		}//if-filter
	}//for-p
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t IterativeFDKerrCompute(const float *Pic, float *ErrorSlicecuda, const float *BetaScanRange, const float *Udomain,
	const double *Pdomain, const double *G, const double *Size, const int t_length, const int s_length, const int z_length,
	const double Center_t, const double Center_s, const double Center_z, const int LBeta, const int LP, const int LU,
	const int LG, const double Distance, const double *resolution2, const int t, const float BetaScanInt, const float dU,
	const double PInt)
{
	mexPrintf("Start CPU-GPU interface!\n");
	
	float *dev_Pic = 0, *dev_Udomain = 0;
	float *dev_ErrorSlicecuda = 0;
	double *dev_Pdomain = 0;
	double *dev_resolution2 = 0, *dev_Size = 0, *dev_G = 0 ;

	int threadcubic_x = MIN(threadX, s_length);
	int blockcubic_x = MIN(blockX, z_length);
	int slresidual = s_length % threadX;
	int zlresidual = z_length % blockX;
	int sTime = s_length / threadX;
	int zTime = z_length / blockX;
	int sstart = 0, zstart = 0;
	float beta = 0, U = 0; double P = 0;
	const dim3 thread_cubic(threadcubic_x,1,1);     // num of threads in each block depends on s_length
	const dim3 block_cubic(blockcubic_x, 1, 1);      // num of blocks in each grid depends on s_length
	dim3 thread_cubic_residual(1, 1, 1);    // initial, size of dim3 depends on slresidual/zlresidual
	dim3 block_cubic_residual(1, 1, 1);  
	
	float *Zeros = new float[LBeta * s_length * z_length];      // for initialize the output
	memset(Zeros, 0, sizeof(float));

	if (slresidual != 0)
	{
		thread_cubic_residual.x = slresidual;
	}
	if (zlresidual != 0)
	{
		block_cubic_residual.x = zlresidual;
	}

    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		mexPrintf("hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

	mexPrintf("Start to call for GPU space.\n");

    // Allocate GPU buffers for three vectors (6 input, 1 output)    .
    cudaStatus = hipMalloc((void**)&dev_Pic, t_length * s_length * z_length * sizeof(float));
    if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_Pic hipMalloc failed!");
		mexPrintf("dev_Pic hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_Udomain, LU * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dev_Udomain hipMalloc failed!");
		mexPrintf("dev_Udomain hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_Pdomain, LP * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dev_Pdomain hipMalloc failed!");
		mexPrintf("dev_Pdomain hipMalloc failed!");
        goto Error;
    }

	cudaStatus = hipMalloc((void**)&dev_G, LG * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_G hipMalloc failed!");
		mexPrintf("dev_G hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_resolution2, 3 * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_resolution2 hipMalloc failed!");
		mexPrintf("dev_resolution2 hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_Size, 3 * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_Size hipMalloc failed!");
		mexPrintf( "dev_Size hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_ErrorSlicecuda, LBeta * s_length * z_length * sizeof(float));    // output
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_ErrorSlice cuda hipMalloc failed!");
		mexPrintf("dev_ErrorSlice cuda hipMalloc failed!");
		goto Error;
	}

	mexPrintf("Start to copy data from CPU to GPU space.\n");

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_Pic, Pic, t_length * s_length * z_length * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dev_Pic hipMemcpy failed!");
		mexPrintf("dev_Pic hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_Udomain, Udomain, LU * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dev_Udomain hipMemcpy failed!");
		mexPrintf("dev_Udomain hipMemcpy failed!");
        goto Error;
    }

	cudaStatus = hipMemcpy(dev_Pdomain, Pdomain, LP * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_Pdomain hipMemcpy failed!");
		mexPrintf("dev_Pdomain hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_G, G, LG * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_G hipMemcpy failed!");
		mexPrintf("dev_G hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_resolution2, resolution2, 3 * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_resolution2 hipMemcpy failed!");
		mexPrintf("dev_resolution2 hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_Size, Size, 3 * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_Size hipMemcpy failed!");
		mexPrintf("dev_Size hipMemcpy failed!");
		goto Error;
	}

	// initialize the output
	cudaStatus = hipMemcpy(dev_ErrorSlicecuda, Zeros, LBeta * s_length * z_length * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_ErrorSlicecuda initialization failed!");
		mexPrintf("dev_ErrorSlicecuda initialization failed!");
		goto Error;
	}

	mexPrintf("start to compute discrepancy occur in each point.\n");

    // Launch a kernel on the GPU with one thread for each element.
	for (int numB = 0; numB < LBeta; numB++)            // rotation angle
	{
		beta = BetaScanRange[numB];
		//mexPrintf("numbeta: %d\n", numB);
		for (int numU = 0; numU < LU; numU++)      // row on the detector
		{
			U = Udomain[numU];
			//for (int numP = 0; numP < LP; numP++)      // row on the detector
			//{
			//	P = Pdomain[numP];
		
				for (int nums = 0; nums < sTime; nums++)
				{
					for (int numz = 0; numz < zTime; numz++)
					{
						sstart = nums * threadX;
						zstart = numz * blockX;
						//mexPrintf("%d %d\n", sstart, zstart);

						IterativeFDKerr << <block_cubic, thread_cubic >> >(dev_Pic, dev_ErrorSlicecuda, U, dev_Pdomain, dev_G,
							dev_Size, dev_resolution2, Center_t, Center_s, Center_z, t_length, s_length, z_length, beta, numB, sstart,
							zstart, Distance, LBeta, LP, LU, LG, t, BetaScanInt, dU, PInt);
					}
				}

				if (slresidual != 0)
				{
					sstart = s_length - slresidual;
					if (zlresidual != 0)
					{
						zstart = z_length - zlresidual;
						//("%d %d\n", sstart, zstart);
						IterativeFDKerr << <block_cubic, thread_cubic >> >(dev_Pic, dev_ErrorSlicecuda, U, dev_Pdomain, dev_G,
							dev_Size, dev_resolution2, Center_t, Center_s, Center_z, t_length, s_length, z_length, beta, numB, sstart,
							zstart, Distance, LBeta, LP, LU, LG, t, BetaScanInt, dU, PInt);
					}

					for (int numz = 0; numz < zTime; numz++)
					{
						zstart = numz * blockX;
						//("%d %d\n", sstart, zstart);
						IterativeFDKerr << <block_cubic, thread_cubic >> >(dev_Pic, dev_ErrorSlicecuda, U, dev_Pdomain, dev_G,
							dev_Size, dev_resolution2, Center_t, Center_s, Center_z, t_length, s_length, z_length, beta, numB, sstart,
							zstart, Distance, LBeta, LP, LU, LG, t, BetaScanInt, dU, PInt);
					}
				}
				if (zlresidual != 0)
				{
					zstart = z_length - zlresidual;
					for (int nums = 0; nums < sTime; nums++)
					{
						sstart = nums * threadX;
						//mexPrintf("%d %d\n", sstart, zstart);
						IterativeFDKerr << <block_cubic, thread_cubic >> >(dev_Pic, dev_ErrorSlicecuda, U, dev_Pdomain, dev_G,
							dev_Size, dev_resolution2, Center_t, Center_s, Center_z, t_length, s_length, z_length, beta, numB, sstart,
							zstart, Distance, LBeta, LP, LU, LG, t, BetaScanInt, dU, PInt);
					}
				}
				
			//}//for-nump
		}//for-numU		
	}//for-numB

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		mexPrintf("addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		mexPrintf("hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(ErrorSlicecuda, dev_ErrorSlicecuda, LBeta * s_length * z_length * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
		mexPrintf("ErrorSlicecuda hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_ErrorSlicecuda);
    hipFree(dev_Pic);
    hipFree(dev_Udomain);
	hipFree(dev_Pdomain);
	hipFree(dev_G);
	hipFree(dev_Size);
	hipFree(dev_resolution2);

    return cudaStatus;
}

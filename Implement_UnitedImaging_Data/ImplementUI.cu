#include "hip/hip_runtime.h"
#include "ImplementUI.h"

#define pow2(x) (1.0*(x)*(x))
__device__ const double PI = 3.141592653589793;

__global__ void GFunction(double *dev_G, const double GamaInt, const double *dev_Xigamadomain, const double Distance_s2d)
{

	const unsigned int Gamaindex = threadIdx.x;
	const unsigned int Xigamaindex = blockIdx.x;
	double xigama = dev_Xigamadomain[Xigamaindex];

	double Proportion = Distance_s2d / sqrt(pow2(Distance_s2d) + pow2(xigama));
	double GamaIntP = GamaInt * Proportion;

	// S_L filter 
	if (Gamaindex == 0)
		dev_G[THREAD_SIZE_X - 1 + Xigamaindex * Filter_SIZE] = 2.0 / pow2(PI*GamaIntP);
	else
	{
		dev_G[(THREAD_SIZE_X - 1) - Gamaindex + Xigamaindex * Filter_SIZE] = -2.0 * pow2(Gamaindex) / pow2(PI) / (4 * pow2(Gamaindex) - 1) / pow2(sin(Gamaindex * GamaIntP));
		dev_G[(THREAD_SIZE_X - 1) + Gamaindex + Xigamaindex * Filter_SIZE] = -2.0 * pow2(Gamaindex) / pow2(PI) / (4 * pow2(Gamaindex) - 1) / pow2(sin(Gamaindex * GamaIntP));
	}
}

__global__ void PreWeightFiltration(double *dev_Rcov, double *dev_R, const double *dev_G, const double *dev_Gamadomain,
	const double *dev_BetaScanRange, const double *dev_Xigamadomain, const double betaStartAngle, const double MaxGama,
	const double Distance, const double Distance_s2d, const double GamaInt)
{
  
	//const unsigned int Gamaindex = blockIdx.x * blockDim.x + threadIdx.x;
	//const unsigned int Xigamaindex = blockIdx.y * blockDim.y + threadIdx.y;
	//const unsigned int Betaindex = blockIdx.z * blockDim.z + threadIdx.z;
	//const unsigned long thread_id = Betaindex * ( gridDim.x * gridDim.y * blockDim.x * blockDim.y ) 
	//	+ Xigamaindex * ( gridDim.x * blockDim.x ) + Gamaindex ;
	
	const unsigned int Gamaindex = threadIdx.x;
	const unsigned int Xigamaindex = blockIdx.x;
	const unsigned int Betaindex = blockIdx.y;
	const unsigned long thread_id = Betaindex * (gridDim.x * blockDim.x)
		+ Xigamaindex * blockDim.x + Gamaindex;

	double Weight = 0 ;

	double beta = dev_BetaScanRange[Betaindex];
	double gama = dev_Gamadomain[Gamaindex];
	double xigama = dev_Xigamadomain[Xigamaindex];

	double Proportion = Distance_s2d / sqrt(pow2(Distance_s2d) + pow2(xigama));
	double betaP = beta * Proportion;
	double gamaP = gama * Proportion;
	double GamaIntP = GamaInt * Proportion;
	double DistanceP = Distance / Proportion;
	double betaStartAngleP = betaStartAngle * Proportion;
	double MaxGamaP = MaxGama * Proportion;

	// Parker function
	if (betaP >= betaStartAngleP && betaP < betaStartAngleP + MaxGamaP - 2 * gamaP)
		Weight = pow2(sin(PI / 4 * betaP / (MaxGamaP / 2 - gamaP)));
	else if(betaP >= betaStartAngleP + MaxGamaP - 2 * gamaP && betaP <= betaStartAngleP + PI - 2 * gamaP)
		Weight = 1;
	else if(betaP > betaStartAngleP + PI - 2 * gamaP && betaP <= betaStartAngleP + PI + MaxGamaP)
		Weight = pow2(sin(PI / 4 * (PI + MaxGamaP - betaP) / (MaxGamaP / 2 + gamaP)));
	else
		Weight = 0;

	dev_R[thread_id] = dev_R[thread_id] * DistanceP * cos(gamaP) * Weight;    // directly cover the input

	__syncthreads();
	double Rcovsum = 0;
	// convolution
	for (int g=0 ; g<THREAD_SIZE_X; g++)
	{
		//convolution
		//dev_Rcov[thread_id] += dev_R[Betaindex * (gridDim.x * gridDim.y * blockDim.x * blockDim.y)
		//	+ Xigamaindex * (gridDim.x * blockDim.x) + g] * dev_G[(gridDim.x*blockDim.x - 1) + Gamaindex - g];
		Rcovsum += dev_R[Betaindex * (gridDim.x * blockDim.x)
			+ Xigamaindex * blockDim.x + g] * dev_G[(blockDim.x - 1) + Gamaindex - g + Xigamaindex * Filter_SIZE];
	}
	__syncthreads();

	dev_Rcov[thread_id] = GamaIntP * Rcovsum;

}

__global__ void BackProjection(const double *dev_Rcov, double *dev_Display, const int *dev_Size,
	const int z_length, const double Beta, const double Distance, const double Distance_s2d, 
	const double *dev_Gamadomain, const double *dev_Xigamadomain, const double GamaInt, const double XigamaInt, 
	const double BetaScanInt, const double minGama, const double maxGama, const double minXigama, const double maxXigama, 
	const int betaIndex, const int LGama, const int LXigama)
{
	const unsigned int Tindex = threadIdx.x;
	const unsigned int Sindex = blockIdx.x;
	const unsigned int Zindex = blockIdx.y;
	const unsigned long thread_id = Zindex * (gridDim.x * blockDim.x)
		+ Sindex * blockDim.x + Tindex;
	// initialize

	const double Resolution_z = 1.0 * dev_Size[2] / z_length;

	// define the source in ground coordinate
	double source_t = Distance * cos(Beta + PI / 2);            
	double source_s = Distance * sin(Beta + PI / 2);
	double source_z = 0;

	// rotation center
	double Center_t = dev_Size[0] / 2;                          
	double Center_s = dev_Size[1] / 2;
	double Center_z = dev_Size[2] / 2;

	double image_t = (Tindex - 0.5) - Center_t;  double image_s = (Sindex - 0.5) - Center_s; double image_z = (Zindex - 0.5) * Resolution_z - Center_z;           // image pixel in ground coordinate
	double L2 = pow2(image_t - source_t) + pow2(image_s - source_s) + pow2(image_z - source_z);
	
	// rotate in ground coordinate
	double dect_t = image_t * cos(Beta) + image_s * sin(Beta);
	double dect_s = -image_t * sin(Beta) + image_s * cos(Beta);
	double dect_z = image_z;
	
	// define the projection position on the detector
	double Xigama = Distance_s2d * tan(asin(dect_z / sqrt(L2)));
	double Gama = atan(dect_t / (Distance - dect_s));

	double Proportion = sqrt(L2) / sqrt(L2 - pow2(dect_z));

	unsigned int XigamaN1index = 0, XigamaN2index = 0, GamaN1index = 0, GamaN2index = 0;
	double Gama_domain1 = 0, Gama_domain2 = 0, Xigama_domain1 = 0, Xigama_domain2 = 0;
	double Xig1 = 0, Xig2 = 0, Gama1 = 0, Gama2 = 0;
	double Display_pBeta = 0;

	if ((Gama >= minGama) && (Gama < maxGama) && (Xigama >= minXigama) && (Xigama < maxXigama))
	{
		XigamaN1index = floor(fabs(Xigama - dev_Xigamadomain[0]) / XigamaInt);
		XigamaN2index = XigamaN1index + 1;
		GamaN1index = floor(fabs(Gama - dev_Gamadomain[0]) / GamaInt);
		GamaN2index = GamaN1index + 1;

		Gama_domain1 = dev_Gamadomain[GamaN1index]; Gama_domain2 = dev_Gamadomain[GamaN2index];
		Xigama_domain1 = dev_Xigamadomain[XigamaN1index]; Xigama_domain2 = dev_Xigamadomain[XigamaN2index];

		//bilinear interpolation
		Xig1 = fabs(Xigama - Xigama_domain1); Xig2 = fabs(Xigama_domain2 - Xigama);
		Gama1 = fabs(Gama - Gama_domain1); Gama2 = fabs(Gama_domain2 - Gama);

		//Display_pBeta = 1;
		Display_pBeta = (Xig2 * Gama2 * dev_Rcov[betaIndex * LGama * LXigama + XigamaN1index * LGama + GamaN1index]
			+ Xig1 * Gama2 * dev_Rcov[betaIndex * LGama * LXigama + XigamaN2index * LGama + GamaN1index] + Xig2 * Gama1 * dev_Rcov[betaIndex * LGama * LXigama + XigamaN1index * LGama + GamaN2index]
			+ Xig1 * Gama1 * dev_Rcov[betaIndex * LGama * LXigama + XigamaN2index * LGama + GamaN2index]) / (GamaInt * XigamaInt) / L2 *  (BetaScanInt / Proportion);

	}

	dev_Display[thread_id] += Display_pBeta;

}



// Helper function for using CUDA to add vectors in parallel.
hipError_t FDKUI(double *Display, const double *R, const double *Xigamadomain, const double *Gamadomain,
	const double *BetaScanRange, const double betaStartAngle, const double MaxGama, const double Distance,
	const double Distance_s2d, const int LGama, const int LBeta, const int LXigama, const int *Size, const int z_length)
{
	mexPrintf("Hello FDKUI!\n");
	double *dev_R = 0, *dev_Rcov = 0;
	double *dev_G = 0;
	double *dev_Gamadomain = 0, *dev_Xigamadomain =0, *dev_BetaScanRange = 0;
	int *dev_Size = 0;

	double *dev_Display = 0;
	double GamaInt = fabs(Gamadomain[1] - Gamadomain[0]);
	double XigamaInt = fabs(Xigamadomain[1] - Xigamadomain[0]);
	double BetaScanInt = fabs(BetaScanRange[1] - BetaScanRange[0]);
	
	double maxGama = Gamadomain[0] > Gamadomain[LGama - 1] ? Gamadomain[0] : Gamadomain[LGama - 1];
	double minGama = Gamadomain[0] < Gamadomain[LGama - 1] ? Gamadomain[0] : Gamadomain[LGama - 1];
	double maxXigama = Xigamadomain[0] > Xigamadomain[LXigama - 1] ? Xigamadomain[0] : Xigamadomain[LXigama - 1];
	double minXigama = Xigamadomain[0] < Xigamadomain[LXigama - 1] ? Xigamadomain[0] : Xigamadomain[LXigama - 1];
	//mexPrintf("%lf %lf %lf %lf \n", maxGama, minGama, maxXigama, minXigama);

	const long LDisplay = Size[0] * Size[1] * z_length; 

	const dim3 thread_cubic(THREAD_SIZE_X, 1, 1);
	const dim3 block_cubic(BLOCK_SIZE_X, BLOCK_SIZE_Y, 1);
	
	const dim3 thread_cubic_Bp(Size[0], 1, 1);
	const dim3 block_cubic_Bp(Size[1], z_length, 1);

	hipError_t cudaStatus;

	mexPrintf("start cuda\n");

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?\n");
		mexPrintf("hipSetDevice failed!  Do you have a CUDA-capable GPU installed?\n");
        goto Error;
    }

	mexPrintf("call for space in GPU\n");

    // Allocate GPU buffers for three vectors (two input, one output).
    cudaStatus = hipMalloc((void**)&dev_R, ARRAY_SIZE_IN_BYTES);
    if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_R hipMalloc failed!\n");
		mexPrintf("dev_R hipMalloc failed!\n");
        goto Error1;
    }

	cudaStatus = hipMalloc((void**)&dev_Gamadomain, LGama * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_Gamadomain hipMalloc failed!\n");
		mexPrintf("dev_Gamadomain hipMalloc failed!\n");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_Xigamadomain, LXigama * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_Xigamadomain hipMalloc failed!\n");
		mexPrintf("dev_Xigamadomain hipMalloc failed!\n");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_BetaScanRange, LBeta * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_BetaScanRange hipMalloc failed!\n");
		mexPrintf("dev_BetaScanRange hipMalloc failed!\n");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_G, LXigama * Filter_SIZE * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_G hipMalloc failed!\n");
		mexPrintf("dev_G hipMalloc failed!\n");
		goto Error1;
	}

	cudaStatus = hipMalloc((void**)&dev_Rcov, ARRAY_SIZE_OUT_BYTES);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_BetaScanRange hipMalloc failed!\n");
		mexPrintf("dev_BetaScanRange hipMalloc failed!\n");
		goto Error;
	}

	//mexPrintf("copy data in CPU to GPU\n");

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_R, R, ARRAY_SIZE_IN_BYTES, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy R failed!\n");
		mexPrintf("hipMemcpy R failed!\n");
        goto Error1;
    }

	cudaStatus = hipMemcpy(dev_Gamadomain, Gamadomain, LGama * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy Gamadomain failed!\n");
		mexPrintf("hipMemcpy Gamadomain failed!\n");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_Xigamadomain, Xigamadomain, LXigama * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy Xigamadomain failed!\n");
		mexPrintf("hipMemcpy Xigamadomain failed!\n");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_BetaScanRange, BetaScanRange, LBeta * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy BetaScanRange failed!\n");
		mexPrintf("hipMemcpy v failed!\n");
		goto Error;
	}

	mexPrintf("start parallel computation\n");
	
	// Generate Filter
	GFunction << <LXigama, LGama >> >(dev_G, GamaInt, dev_Xigamadomain, Distance_s2d);

    // Preweight and filtration

	PreWeightFiltration << <block_cubic, thread_cubic >> >(dev_Rcov, dev_R, dev_G, dev_Gamadomain,
		dev_BetaScanRange, dev_Xigamadomain, betaStartAngle, MaxGama, Distance, Distance_s2d, GamaInt);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "PreWeight and Filtration failed: %s\n", hipGetErrorString(cudaStatus));
		mexPrintf("PreWeight and Filtration failed\n");
		goto Error1;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error1;
	}

Error1:
	hipFree(dev_R);
	hipFree(dev_G);

	cudaStatus = hipMalloc((void**)&dev_Display, LDisplay * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_Display hipMalloc failed!\n");
		mexPrintf("dev_Display hipMalloc failed!\n");
		goto Error;
	}
	hipMemset(dev_Display, 0, sizeof(double));

	cudaStatus = hipMalloc((void**)&dev_Size, 3 * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_Size hipMalloc failed!\n");
		mexPrintf("dev_Size hipMalloc failed!\n");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_Size, Size, 3 * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy Size failed!\n");
		mexPrintf("hipMemcpy Size failed!\n");
		goto Error;
	}

	//Backprojection
	for (int betaIndex = 0; betaIndex < LBeta; betaIndex++)
	{
		//int betaIndex = 0;
		BackProjection << <block_cubic_Bp, thread_cubic_Bp >> > (dev_Rcov, dev_Display, dev_Size, z_length,
			BetaScanRange[betaIndex], Distance, Distance_s2d, dev_Gamadomain, dev_Xigamadomain, GamaInt, XigamaInt,
			BetaScanInt, minGama, maxGama, minXigama, maxXigama, betaIndex, LGama, LXigama);
	}
		
    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
		fprintf(stderr, "BackProjection launch failed: %s\n", hipGetErrorString(cudaStatus));
		mexPrintf("BackProjection launch failed\n");
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(Display, dev_Display, LDisplay * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!\n");
		mexPrintf("hipMemcpy dev_Display failed!\n");
        goto Error;
    }


Error:    
	hipFree(dev_Gamadomain);
	hipFree(dev_BetaScanRange);
	hipFree(dev_Xigamadomain);
	hipFree(dev_Rcov);
	hipFree(dev_Display);
	hipFree(dev_Size);

	mexPrintf("Exit FDKUI\n");
  return cudaStatus;
}
